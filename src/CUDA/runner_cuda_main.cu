#include "hip/hip_runtime.h"
/* Hacky method to make c++ compilers not die. */
#ifdef WITH_CUDA
#ifndef static
#define static
#endif
#ifndef restrict
#define restrict __restrict__
#endif
#endif

extern "C" {
#include <string.h>
}
#include "runner_cuda_main.h"
#include "queue_cuda.h"
#include "task_cuda.h"
#include "cell_cuda.h"
#include "../kernel_hydro.h"
#include "../dimension.h"
#include "../cell.h"
#include "../hydro_properties.h"
#include "../engine.h"
#include "../scheduler.h"
#include "../space.h"
#include "../adiabatic_index.h"

/*Definition of particle structures (SoA on GPU) */
struct particle_arrays {
  /* Device array containing the particle ID */
  long long int *id;
  /*Device array containing the particle positions. */
  double *x_x, *x_y, *x_z;
  /*Device array containing the particle predicted velocity */
  float3 *v;
  /*Device array containing the particle acceleration. */
  float3 *a_hydro;
  /* Device array contining the particle cutoff radius */
  float *h;
  /* Device array containing particle masses */
  float *mass;
  /* Device array containing particle densities. */
  float *rho;
  /* Device array containing the particle entropies. */
  float *entropy;
  /* Device array containing the particle entropy_dt */
  float *entropy_dt;

  /* Unclear if unions work on the GPU, so unrolling the union for now */
  /* DENSITY */
  /* Device array containing the number of neighbours. */
  float *wcount;
  /* Device array containing the number of neighbours spatial derivative */
  float *wcount_dh;
  /* Device array containing the derative of the density w.r.t. h */
  float *rho_dh;
  /* Device array containing the particle velocity curl. */
  float3 *rot_v;
  /* Device array containing the particle velocity divergence */
  float *div_v;

  /* FLOAT */
  /* Device array containing the balsara switch */
  float *balsara;
  /* Device array containing the "Grad h" term */
  float *f;
  /* Device array containing the pressure over density squared */
  float *P_over_rho2;
  /* Device array containing the particle sound speeds */
  float *soundspeed;
  /* Device array containing the signal velocities */
  volatile float *v_sig;
  /* Device array containing the time derivative of the smoothing lengths */
  float *h_dt;

  /* Device array containing time step length */
  timebin_t *time_bin;
};

__device__ struct particle_arrays cuda_parts;
__constant__ int cuda_nr_parts;

/* Cell array and array of CPU cell pointers.*/
__device__ struct cell_cuda *cells_cuda;
__device__ struct cell **cpu_cells;

/* Queue variables*/
__device__ struct queue_cuda cuda_queues[cuda_numqueues];
__device__ struct queue_cuda load_queue;
__device__ struct queue_cuda unload_queue;

/* Array of tasks on the device */
__device__ struct task_cuda *tasks;

/* Array of cuda tasks on the host */
struct task_cuda *tasks_host;

/* Array of unlocks on the device*/
__device__ int *cuda_unlocks;
__device__ int cuda_nr_unlocks;

/* Array of unlocks on the host. */
int *host_unlocks;

/* Simulation constants */
__device__ __constant__ integertime_t ti_current;
__device__ __constant__ double dim[3];
__device__ __constant__ timebin_t max_active_bin;
__device__ __constant__ float delta_neighbours;
__device__ __constant__ float target_neighbours;
__device__ __constant__ float hydro_h_max;
__device__ __constant__ float cuda_h_tolerance;
__device__ __constant__ float cuda_eta_neighbours;

/* Queue function to search for a task index from a specific queue. */
__device__ int cuda_queue_gettask(struct queue_cuda *q) {

  int ind, tid = -1;

  /* Don't try if queue is empty. */
  if (q->rec_count == q->count) return -1;

  /* Get index of the next task. */
  ind = atomicAdd(&q->first, 1);

  ind %= cuda_queue_size;
  /* Loop until there is a valid task at that index. */
  while (q->rec_count < q->count && (tid = q->data[ind]) < 0)
    ;

  /* Remove the task from the queue. */
  if (tid >= 0) {
    q->data[ind] = -1;
    atomicAdd((int *)&tot_num_tasks, -1);
  }

  /* Return the acquired task ID */
  return tid;
}

/* Queue function to add task index tid to the queue*/
__device__ void cuda_queue_puttask(struct queue_cuda *q, int tid) {

  int ind;

  /* Get the index of the next task. */
  ind = atomicAdd(&q->last, 1) % cuda_queue_size;

  /* Wait for the slot in the queue to be empty. */
  while (q->data[ind] != -1)
    ;

  /* Write the task back to the queue. */
  q->data[ind] = tid;

  atomicAdd((int *)&q->nr_avail_tasks, 1);
}

/* Density kernel function */
__constant__ float cuda_kernel_coeffs[(kernel_degree + 1) * (kernel_ivals + 1)];

#define cuda_kernel_root                        \
  ((float)(cuda_kernel_coeffs[kernel_degree]) * \
   kernel_constant *kernel_gamma_inv_dim)

__device__ int cuda_cell_is_active(struct cell_cuda *c) {
  return (c->ti_end_min == ti_current);
}

__device__ float cuda_pow_dimension(float x) {

#if defined(HYDRO_DIMENSION_3D)
  return x * x * x;
#elif defined(HYDRO_DIMENSION_2D)
  return x * x;
#elif defined(HYDRO_DIMENSION_1D)
  return x;
#else
  printf("The dimension is not defined!");
  return 0.f;
#endif
}

__device__ float cuda_pow_dimension_plus_one(float x) {
#if defined(HYDRO_DIMENSION_3D)
  const float x2 = x * x;
  return x2 * x2;
#elif defined(HYDRO_DIMENSION_2D)
  return x * x * x;
#elif defined(HYDRO_DIMENSION_1D)
  return x * x;
#else
  printf("The dimension is not defined!");
  return 0.f;
#endif
}

__device__ float cuda_pow_dimension_minus_one(float x) {
#if defined(HYDRO_DIMENSION_3D)
  return x * x;
#elif defined(HYDRO_DIMENSION_2D)
  return x;
#elif defined(HYDRO_DIMENSION_1D)
  return 1.f;
#else
  printf("The dimension is not defined!");
  return 0.f;
#endif
}

__device__ __inline__ void cuda_kernel_deval(float u, float *restrict W,
                                             float *restrict dW_dx) {
  /* Go to the range [0,1] from [0,H] */
  const float x = u * kernel_gamma_inv;

  /* Pick the correct branch of the kernel */
  const int temp = (int)(x * kernel_ivals_f);
  const int ind = temp > kernel_ivals ? kernel_ivals : temp;
  const float *coeffs = &cuda_kernel_coeffs[ind * (kernel_degree + 1)];

  /* First two terms of the polynomial*/
  float w = coeffs[0] * x + coeffs[1];
  float dw_dx = coeffs[0];
  /* and the rest of them*/
  for (int k = 2; k <= kernel_degree; k++) {
    dw_dx = dw_dx * x + w;
    w = x * w + coeffs[k];
  }

  /* Return the results */
  *W = w * kernel_constant * kernel_gamma_inv_dim;
  *dW_dx = dw_dx * kernel_constant * kernel_gamma_inv_dim_plus_one;
}
__device__ __inline__ int cuda_part_is_active(int pid) {

  return (cuda_parts.time_bin[pid] <= max_active_bin);
}

__device__ void load_cell(int cell_index) {
  /* Get the pointer to the relevant cells. */
  struct cell *cpu_cell = cpu_cells[cell_index];
  struct cell_cuda *cell = &cells_cuda[cell_index];
  struct part *parts = cpu_cell->parts;
  int i;
  /* Get the index to copy the data for the 0th particle in this cell to.*/
  int start = cell->first_part;
//  if(cell_index == 1) asm("trap;");
  __syncthreads();
  for (i = threadIdx.x; i < cell->part_count; i += blockDim.x) {
    struct part *current = &parts[i];
    /* Load the ID and position*/
    long long local_id = current->id;
    double local_x[3];
    local_x[0] = current->x[0];
    local_x[1] = current->x[1];
    local_x[2] = current->x[2];

    /* Unpack the ID and position and put into the device arrays. */
    cuda_parts.id[start + i] = local_id;
    cuda_parts.x_x[start + i] = local_x[0];
    cuda_parts.x_y[start + i] = local_x[1];
    cuda_parts.x_z[start + i] = local_x[2];

    /* Load the predicted velocity */
    float local_tempv[3];
    local_tempv[0] = current->v[0];
    local_tempv[1] = current->v[1];
    local_tempv[2] = current->v[2];

    /* Repack the predicted velocity */
    float3 local_v;
    local_v.x = local_tempv[0];
    local_v.y = local_tempv[1];
    local_v.z = local_tempv[2];

    /* Copy the predicted velocity to the particle array. */
    cuda_parts.v[start + i] = local_v;

    /* Load the acceleration */
    float local_tempa[3];
    local_tempa[0] = current->a_hydro[0];
    local_tempa[1] = current->a_hydro[1];
    local_tempa[2] = current->a_hydro[2];

    /* Repack the acceleration */
    float3 local_a;
    local_a.x = local_tempa[0];
    local_a.y = local_tempa[1];
    local_a.z = local_tempa[2];

    /* Copy the predicted velocity to the particle array. */
    cuda_parts.a_hydro[start + i] = local_a;

    /* Copy the cutoff, mass, density, entropy and entropy derivative*/
    float local_h, local_mass, local_rho, local_entropy, local_entropy_dt;
    local_h = current->h;
    local_mass = current->mass;
    local_rho = current->rho;
    local_entropy = current->entropy;
    local_entropy_dt = current->entropy_dt;

    cuda_parts.h[start + i] = local_h;
    cuda_parts.mass[start + i] = local_mass;
    cuda_parts.rho[start + i] = local_rho;
    cuda_parts.entropy[start + i] = local_entropy;
    cuda_parts.entropy_dt[start + i] = local_entropy_dt;

    /* Copy the density union to the device */
    float local_wcount, local_wcount_dh, local_rho_dh, local_div_v;
    float3 local_rot_v;
    local_wcount = current->density.wcount;
    local_wcount_dh = current->density.wcount_dh;
    local_rho_dh = current->density.rho_dh;
    local_rot_v.x = current->density.rot_v[0];
    local_rot_v.y = current->density.rot_v[1];
    local_rot_v.z = current->density.rot_v[2];
    local_div_v = current->density.div_v;

    cuda_parts.wcount[start + i] = local_wcount;
    cuda_parts.wcount_dh[start + i] = local_wcount_dh;
    cuda_parts.rho_dh[start + i] = local_rho_dh;
    cuda_parts.rot_v[start + i] = local_rot_v;
    cuda_parts.div_v[start + i] = local_div_v;

    /* Copy the force union to the device. TODO Do we need this?*/
    float local_balsara, local_f, local_P_over_rho2, local_soundspeed,
        local_vsig, local_h_dt;

    local_balsara = current->force.balsara;
    local_f = current->force.f;
    local_P_over_rho2 = current->force.P_over_rho2;
    local_soundspeed = current->force.soundspeed;
    local_vsig = current->force.v_sig;
    local_h_dt = current->force.h_dt;

    cuda_parts.balsara[start + i] = local_balsara;
    cuda_parts.f[start + i] = local_f;
    cuda_parts.P_over_rho2[start + i] = local_P_over_rho2;
    cuda_parts.soundspeed[start + i] = local_soundspeed;
    cuda_parts.v_sig[start + i] = local_vsig;
    cuda_parts.h_dt[start + i] = local_h_dt;

    cuda_parts.time_bin[start + i] = current->time_bin;
  }
}

/* Task function to unload a specific cell. */
/* TODO: Note that this copies back certain data that I expect is not
 * modified.*/
__device__ void unload_cell(int cell_index) {

  /* Get the pointer to the relevant cells. */
  struct cell *cpu_cell = cpu_cells[cell_index];
  const struct cell_cuda *cell = &cells_cuda[cell_index];
  struct part *parts = cpu_cell->parts;
  int i;
  /* Get the index to copy the data for the 0th particle in this cell to.*/
  int start = cell->first_part;

  for (i = threadIdx.x; i < cell->part_count; i += blockDim.x) {
    struct part *current = &parts[i];

    /* Copy back the ID and position.*/
    current->id = cuda_parts.id[start + i];
    current->x[0] = cuda_parts.x_x[start + i];
    current->x[1] = cuda_parts.x_y[start + i];
    current->x[2] = cuda_parts.x_z[start + i];

    /* Copy back the velocity*/
    float3 local_v = cuda_parts.v[start + i];
    current->v[0] = local_v.x;
    current->v[1] = local_v.y;
    current->v[2] = local_v.z;

    /* Copy back the acceleration */
    float3 local_a_hydro = cuda_parts.a_hydro[start + i];
    current->a_hydro[0] = local_a_hydro.x;
    current->a_hydro[1] = local_a_hydro.y;
    current->a_hydro[2] = local_a_hydro.z;

    /* Copy back the cutoff, mass, density, entropy and entropy_dt*/
    current->h = cuda_parts.h[start + i];
    current->mass = cuda_parts.mass[start + i];
    current->rho = cuda_parts.rho[start + i];
    current->entropy = cuda_parts.entropy[start + i];
    current->entropy_dt = cuda_parts.entropy_dt[start + i];

    /* Copy back the force union.*/
    current->force.balsara = cuda_parts.balsara[start + i];
    current->force.f = cuda_parts.f[start + i];
    current->force.P_over_rho2 = cuda_parts.P_over_rho2[start + i];
    current->force.soundspeed = cuda_parts.soundspeed[start + i];
    current->force.v_sig = cuda_parts.v_sig[start + i];
    current->force.h_dt = cuda_parts.h_dt[start + i];

    /* Copy back the timebin. */
    current->time_bin = cuda_parts.time_bin[start + i];
  }
}

/* Task function to execute a self-density task. */
__device__ void doself_density(struct cell_cuda *ci) {

  /* Is the cell active? */
  if (!cuda_cell_is_active(ci)) {
    printf(
        "Cell isn't active..., ti_end_min=%i, ti_current=%i, "
        "max_active_bin=%i\n",
        ci->ti_end_min, ti_current, max_active_bin);
    return;
  }

  const int count_i = ci->part_count;
  int part_i = ci->first_part;
  float rho, rho_dh, div_v, wcount, wcount_dh;
  float3 rot_v;

  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {
    double pix[3];
    pix[0] = cuda_parts.x_x[pid];
    pix[1] = cuda_parts.x_y[pid];
    pix[2] = cuda_parts.x_z[pid];
    const float hi = cuda_parts.h[pid];
    const float hig2 = hi * hi * kernel_gamma2;

    /* Reset local values. */
    rho = 0.0f;
    rho_dh = 0.0f;
    div_v = 0.0f;
    wcount = 0.0f;
    wcount_dh = 0.0f;
    rot_v.x = 0.0f;
    rot_v.y = 0.0f;
    rot_v.z = 0.0f;

    /* If the particle isn't active skip it. */
    if (!cuda_part_is_active(pid)) {
      printf("Particle %i isn't active\n", pid);
      continue;
    }

    /* Search for the neighbours! */
    for (int pjd = part_i; pjd < part_i + count_i; pjd++) {
      /* Particles don't interact with themselves */
      if (pid == pjd) continue;
      float dx[3], r2 = 0.0f;
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];

      /* If in range then interact. */
      if (r2 < hig2) {
        float w, dw_dx;
        float dv[3], curlvr[3];
        /* Load mass on particle pj. */
        const float mj = cuda_parts.mass[pjd];

        /* Get r and 1/r */
        const float r = sqrtf(r2);
        const float ri = 1.0f / r;

        /* Compute the kernel function */
        const float hi_inv = 1.0f / hi;
        const float ui = r * hi_inv;

        cuda_kernel_deval(ui, &w, &dw_dx);
        /* Compute contribution to the density. */
        rho += mj * w;
        rho_dh -= mj * (hydro_dimension * w + ui * dw_dx);

        /* Compute contribution to the number of neighbours */
        wcount += w;
        wcount_dh -= (hydro_dimension * w + ui * dw_dx);

        const float fac = mj * dw_dx * ri;

        /* Compute dv dot r */
        float3 piv, pjv;
        piv = cuda_parts.v[pid];
        pjv = cuda_parts.v[pjd];
        dv[0] = piv.x - pjv.x;
        dv[1] = piv.y - pjv.y;
        dv[2] = piv.z - pjv.z;
        const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

        div_v -= fac * dvdr;

        curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
        curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
        curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];

        rot_v.x += fac * curlvr[0];
        rot_v.y += fac * curlvr[1];
        rot_v.z += fac * curlvr[2];
      }
    }  // Loop over cj.
    /* Write data for particle pid back to global stores. */
    atomicAdd(&cuda_parts.rho[pid], rho);
    atomicAdd(&cuda_parts.rho_dh[pid], rho_dh);
    atomicAdd(&cuda_parts.wcount[pid], wcount);
    atomicAdd(&cuda_parts.wcount_dh[pid], wcount_dh);
    atomicAdd(&cuda_parts.div_v[pid], div_v);
    atomicAdd(&cuda_parts.rot_v[pid].x, rot_v.x);
    atomicAdd(&cuda_parts.rot_v[pid].y, rot_v.y);
    atomicAdd(&cuda_parts.rot_v[pid].z, rot_v.z);
  }
}

/* Task function to execute a density task. Uses naive n^2 algorithm without
 * symmetry*/
/* To do density between Cell i and cell j this needs to be called twice. */
__device__ void dopair_density(struct cell_cuda *ci, struct cell_cuda *cj) {

  /* Are these cells active? */
  if (!cuda_cell_is_active(ci) && !cuda_cell_is_active(cj)) return;

  const int count_i = ci->part_count;
  const int count_j = cj->part_count;
  int part_i = ci->first_part;
  int part_j = cj->first_part;
  float rho, rho_dh, div_v, wcount, wcount_dh;
  float3 rot_v;
  double shift[3] = {0.0, 0.0, 0.0};

  /* Deal with periodicity concerns. */
  for (int k = 0; k < 3; k++) {
    if (cj->loc[k] - ci->loc[k] < -dim[k] / 2)
      shift[k] = dim[k];
    else if (cj->loc[k] - ci->loc[k] > dim[k] / 2)
      shift[k] = -dim[k];
  }

  /* Loop over the parts in cell ci */
  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {

    const float hi = cuda_parts.h[pid];

    double pix[3];
    pix[0] = cuda_parts.x_x[pid] - shift[0];
    pix[1] = cuda_parts.x_y[pid] - shift[1];
    pix[2] = cuda_parts.x_z[pid] - shift[2];
    const float hig2 = hi * hi * kernel_gamma2;

    if (!cuda_part_is_active(pid)) continue;

    /* Reset local values. */
    rho = 0.0f;
    rho_dh = 0.0f;
    div_v = 0.0f;
    wcount = 0.0f;
    wcount_dh = 0.0f;
    rot_v.x = 0.0f;
    rot_v.y = 0.0f;
    rot_v.z = 0.0f;

    /* Loop over the parts in cj. */
    /* TODO May be possible to optimize this loop ordering.*/
    for (int pjd = part_j; pjd < part_j + count_j; pjd++) {

      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];
      /* If in range then interact. */
      if (r2 < hig2) {
        float w, dw_dx;
        float dv[3], curlvr[3];

        /* Load mass on particle pj. */
        const float mj = cuda_parts.mass[pjd];

        /* Get r and 1/r */
        const float r = sqrtf(r2);
        const float ri = 1.0f / r;

        /* Compute the kernel function */
        const float hi_inv = 1.0f / hi;
        const float ui = r * hi_inv;

        cuda_kernel_deval(ui, &w, &dw_dx);

        /* Compute contribution to the density. */
        rho += mj * w;
        rho_dh -= mj * (hydro_dimension * w + ui * dw_dx);

        /* Compute contribution to the number of neighbours */
        wcount += w;
        wcount_dh -= (hydro_dimension * w + ui * dw_dx);

        const float fac = mj * dw_dx * ri;

        /* Compute dv dot r */
        float3 piv, pjv;
        piv = cuda_parts.v[pid];
        pjv = cuda_parts.v[pjd];
        dv[0] = piv.x - pjv.x;
        dv[1] = piv.y - pjv.y;
        dv[2] = piv.z - pjv.z;
        const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

        div_v -= fac * dvdr;

        curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
        curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
        curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];
        
        rot_v.x += fac * curlvr[0];
        rot_v.y += fac * curlvr[1];
        rot_v.z += fac * curlvr[2];
        // w);
      }
    }  // Loop over cj.
    /* Write data for particle pid back to global stores. */
    atomicAdd(&cuda_parts.rho[pid], rho);
    atomicAdd(&cuda_parts.rho_dh[pid], rho_dh);
    atomicAdd(&cuda_parts.wcount[pid], wcount);
    atomicAdd(&cuda_parts.wcount_dh[pid], wcount_dh);
    atomicAdd(&cuda_parts.div_v[pid], div_v);
    atomicAdd(&cuda_parts.rot_v[pid].x, rot_v.x);
    atomicAdd(&cuda_parts.rot_v[pid].y, rot_v.y);
    atomicAdd(&cuda_parts.rot_v[pid].z, rot_v.z);

  }  // Loop over ci.
}

/* Task function to perform a self force task. No symmetry */
__device__ void doself_force(struct cell_cuda *ci) {
  /* Is the cell active? */
  if (!cuda_cell_is_active(ci)) return;

  const int count_i = ci->part_count;
  int part_i = ci->first_part;

  float3 a_hydro;
  float h_dt, v_sig_stor, entropy_dt;

  /* Loop over the particles */
  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {

    const float hi = cuda_parts.h[pid];
    if (!cuda_part_is_active(pid)) continue;
    /* Reset the values. */
    a_hydro.x = 0.0f;
    a_hydro.y = 0.0f;
    a_hydro.z = 0.0f;
    h_dt = 0.0f;
    v_sig_stor = cuda_parts.v_sig[pid];
    entropy_dt = 0.0f;

    double pix[3];
    pix[0] = cuda_parts.x_x[pid];
    pix[1] = cuda_parts.x_y[pid];
    pix[2] = cuda_parts.x_z[pid];

    const float hig2 = hi * hi * kernel_gamma2;

    /* Loop over the particles in cj. */
    for (int pjd = part_i; pjd < part_i + count_i; pjd++) {

      if (pid == pjd) continue;
      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];
      const float hj = cuda_parts.h[pjd];
      if (r2 < hig2 || r2 < hj * hj * kernel_gamma2) {
        float wi, wj, wi_dx, wj_dx;
        const float fac_mu = 1.f;

        const float r = sqrtf(r2);
        const float r_inv = 1.0f / r;

        /* Load some data.*/
        const float mj = cuda_parts.mass[pjd];
        const float rhoi = cuda_parts.rho[pid];
        const float rhoj = cuda_parts.rho[pjd];

        /* Get the kernel for hi. */
        const float hi_inv = 1.0f / hi;
        const float hid_inv = cuda_pow_dimension_plus_one(hi_inv);
        const float ui = r * hi_inv;
        cuda_kernel_deval(ui, &wi, &wi_dx);
        const float wi_dr = hid_inv * wi_dx;

        /* Get the kernel for hj. */
        const float hj_inv = 1.0f / hj;
        const float hjd_inv = cuda_pow_dimension_plus_one(hj_inv);
        const float xj = r * hj_inv;
        cuda_kernel_deval(xj, &wj, &wj_dx);
        const float wj_dr = hjd_inv * wj_dx;

        /* Compute h-gradient terms */
        const float f_i = cuda_parts.f[pid];
        const float f_j = cuda_parts.f[pjd];

        /* Compute pressure terms */
        const float P_over_rho2_i = cuda_parts.P_over_rho2[pid];
        const float P_over_rho2_j = cuda_parts.P_over_rho2[pjd];

        /* Compute sound speeds*/
        const float ci = cuda_parts.soundspeed[pid];
        const float cj = cuda_parts.soundspeed[pjd];

        /* Compute dv dot r. */
        const float dvdr = (cuda_parts.v[pid].x - cuda_parts.v[pjd].x) * dx[0] +
                           (cuda_parts.v[pid].y - cuda_parts.v[pjd].y) * dx[1] +
                           (cuda_parts.v[pid].z - cuda_parts.v[pjd].z) * dx[2];

        /* Balsara term */
        const float balsara_i = cuda_parts.balsara[pid];
        const float balsara_j = cuda_parts.balsara[pjd];

        /* Are the particles moving towards each other? */
        const float omega_ij = (dvdr < 0.f) ? dvdr : 0.f;
        const float mu_ij = fac_mu * r_inv * omega_ij;

        /* Signal velocity */
        const float v_sig = ci + cj - 3.f * mu_ij;
        const float rho_ij = 0.5f * (rhoi + rhoj);
        const float visc = -0.25 * const_viscosity_alpha * v_sig * mu_ij *
                           (balsara_i + balsara_j) / rho_ij;

        /* Now convolce with the kernel */
        const float visc_term = 0.5f * visc * (wi_dr + wj_dr) * r_inv;
        const float sph_term =
            (f_i * P_over_rho2_i * wi_dr + f_j * P_over_rho2_j * wj_dr) * r_inv;

        /* Compute the acceleration */
        const float acc = visc_term + sph_term;

        /* Compute the force */
        a_hydro.x -= mj * acc * dx[0];
        a_hydro.y -= mj * acc * dx[1];
        a_hydro.z -= mj * acc * dx[2];


        /* Get the time derivative for h. */
        h_dt -= mj * dvdr * r_inv / rhoj * wi_dr;

        /* Update the signal velocity. */
        v_sig_stor = (v_sig_stor > v_sig) ? v_sig_stor : v_sig;

        /* Change in entropy */
        entropy_dt += mj * visc_term * dvdr;
      }

    }  // Inner loop

    /* Flush to global stores.*/
    atomicAdd(&cuda_parts.a_hydro[pid].x, a_hydro.x);
    atomicAdd(&cuda_parts.a_hydro[pid].y, a_hydro.y);
    atomicAdd(&cuda_parts.a_hydro[pid].z, a_hydro.z);
    atomicAdd(&cuda_parts.h_dt[pid], h_dt);
    atomicAdd(&cuda_parts.entropy_dt[pid], entropy_dt);

    /* Update the signal velocity */
    float global_vsig = cuda_parts.v_sig[pid];
    int *address_as_int = (int *)&cuda_parts.v_sig[pid];
    int old = *address_as_int;
    int assumed;
    do {
      global_vsig = cuda_parts.v_sig[pid];  // Scary line.
      assumed = old;
      if (v_sig_stor > global_vsig)
        old = atomicCAS(address_as_int, assumed, __float_as_int(v_sig_stor));
    } while (assumed != old && v_sig_stor > global_vsig);

  }  // Outer loop
}

/* Task function to execute a force task. Uses naive n^2 algorithm without
 * symmetry*/
/* To do force between Cell i and cell j this needs to be called twice. */
__device__ void dopair_force(struct cell_cuda *ci, struct cell_cuda *cj) {

  /* Are these cells active? */
  if (!cuda_cell_is_active(ci) && !cuda_cell_is_active(cj)) return;

  const int count_i = ci->part_count;
  const int count_j = cj->part_count;
  int part_i = ci->first_part;
  int part_j = cj->first_part;

  float3 a_hydro;
  float h_dt, v_sig_stor, entropy_dt;

  double shift[3] = {0.0, 0.0, 0.0};
  /* Deal with periodicity concerns. */
  for (int k = 0; k < 3; k++) {
    if (cj->loc[k] - ci->loc[k] < -dim[k] / 2)
      shift[k] = dim[k];
    else if (cj->loc[k] - ci->loc[k] > dim[k] / 2)
      shift[k] = -dim[k];
  }

  /* Loop over the parts in cell ci */
  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {

    const float hi = cuda_parts.h[pid];
    if (!cuda_part_is_active(pid)) continue;
    /* Reset the values. */
    a_hydro.x = 0.0f;
    a_hydro.y = 0.0f;
    a_hydro.z = 0.0f;
    h_dt = 0.0f;
    v_sig_stor = cuda_parts.v_sig[pid];
    entropy_dt = 0.0f;

    double pix[3];
    pix[0] = cuda_parts.x_x[pid] - shift[0];
    pix[1] = cuda_parts.x_y[pid] - shift[1];
    pix[2] = cuda_parts.x_z[pid] - shift[2];

    const float hig2 = hi * hi * kernel_gamma2;

    /* Loop over the particles in cj. */
    for (int pjd = part_j; pjd < part_j + count_j; pjd++) {

      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];

      const float hj = cuda_parts.h[pjd];
      if (r2 < hig2 || r2 < hj * hj * kernel_gamma2) {
        float wi, wj, wi_dx, wj_dx;
        const float fac_mu = 1.f;

        const float r = sqrtf(r2);
        const float r_inv = 1.0f / r;

        /* Load some data.*/
        const float mj = cuda_parts.mass[pjd];
        const float rhoi = cuda_parts.rho[pid];
        const float rhoj = cuda_parts.rho[pjd];

        /* Get the kernel for hi. */
        const float hi_inv = 1.0f / hi;
        const float hid_inv = cuda_pow_dimension_plus_one(hi_inv);
        const float ui = r * hi_inv;
        cuda_kernel_deval(ui, &wi, &wi_dx);
        const float wi_dr = hid_inv * wi_dx;

        /* Get the kernel for hj. */
        const float hj_inv = 1.0f / hj;
        const float hjd_inv = cuda_pow_dimension_plus_one(hj_inv);
        const float xj = r * hj_inv;
        cuda_kernel_deval(xj, &wj, &wj_dx);
        const float wj_dr = hjd_inv * wj_dx;

        /* Compute h-gradient terms */
        const float f_i = cuda_parts.f[pid];
        const float f_j = cuda_parts.f[pjd];

        /* Compute pressure terms */
        const float P_over_rho2_i = cuda_parts.P_over_rho2[pid];
        const float P_over_rho2_j = cuda_parts.P_over_rho2[pjd];

        /* Compute sound speeds*/
        const float ci = cuda_parts.soundspeed[pid];
        const float cj = cuda_parts.soundspeed[pjd];

        /* Compute dv dot r. */
        const float dvdr = (cuda_parts.v[pid].x - cuda_parts.v[pjd].x) * dx[0] +
                           (cuda_parts.v[pid].y - cuda_parts.v[pjd].y) * dx[1] +
                           (cuda_parts.v[pid].z - cuda_parts.v[pjd].z) * dx[2];

        /* Balsara term */
        const float balsara_i = cuda_parts.balsara[pid];
        const float balsara_j = cuda_parts.balsara[pjd];

        /* Are the particles moving towards each other? */
        const float omega_ij = (dvdr < 0.f) ? dvdr : 0.f;
        const float mu_ij = fac_mu * r_inv * omega_ij;

        /* Signal velocity */
        const float v_sig = ci + cj - 3.f * mu_ij;

        /* Now construct the full viscosity term */
        const float rho_ij = 0.5f * (rhoi + rhoj);
        const float visc = -0.25 * const_viscosity_alpha * v_sig * mu_ij *
                           (balsara_i + balsara_j) / rho_ij;

        /* Now convolce with the kernel */
        const float visc_term = 0.5f * visc * (wi_dr + wj_dr) * r_inv;
        const float sph_term =
            (f_i * P_over_rho2_i * wi_dr + f_j * P_over_rho2_j * wj_dr) * r_inv;

        /* Compute the acceleration */
        const float acc = visc_term + sph_term;

        /* Compute the force */
        a_hydro.x -= mj * acc * dx[0];
        a_hydro.y -= mj * acc * dx[1];
        a_hydro.z -= mj * acc * dx[2];

        /* Get the time derivative for h. */
        h_dt -= mj * dvdr * r_inv / rhoj * wi_dr;

        /* Update the signal velocity. */
        v_sig_stor = (v_sig_stor > v_sig) ? v_sig_stor : v_sig;

        /* Change in entropy */
        entropy_dt += mj * visc_term * dvdr;
        // INTERACT
      }

    }  // Loop over cell cj.

    /* Flush to global stores.*/
    atomicAdd(&cuda_parts.a_hydro[pid].x, a_hydro.x);
    atomicAdd(&cuda_parts.a_hydro[pid].y, a_hydro.y);
    atomicAdd(&cuda_parts.a_hydro[pid].z, a_hydro.z);
    atomicAdd(&cuda_parts.h_dt[pid], h_dt);
    atomicAdd(&cuda_parts.entropy_dt[pid], entropy_dt);

    /* Update the signal velocity */
    float global_vsig = cuda_parts.v_sig[pid];
    int *address_as_int = (int *)&cuda_parts.v_sig[pid];
    int old = *address_as_int;
    int assumed;
    do {
      global_vsig = cuda_parts.v_sig[pid];
      assumed = old;
      if (v_sig_stor > global_vsig)
        old = atomicCAS(address_as_int, assumed, __float_as_int(v_sig_stor));
    } while (assumed != old && v_sig_stor > global_vsig);

  }  // Loop over cell ci.
}

/* Device function to finish the density calculation. */
__device__ void hydro_end_density(int pid) {

  const float h = cuda_parts.h[pid];
  const float h_inv = 1.0f / h;
  const float h_inv_dim = cuda_pow_dimension(h_inv);
  const float h_inv_dim_plus_one = h_inv_dim * h_inv;

  /* Final operation on the density (self-contribution) */
  float temp = cuda_parts.mass[pid] * cuda_kernel_root;
  cuda_parts.rho[pid] += temp;
  cuda_parts.rho_dh[pid] -= hydro_dimension * temp;
  cuda_parts.wcount[pid] += cuda_kernel_root;
  cuda_parts.wcount_dh[pid] -= hydro_dimension * cuda_kernel_root;

  /* Finish the calculation by inser4ting the missing h-factors */
  cuda_parts.rho[pid] *= h_inv_dim;
  cuda_parts.rho_dh[pid] *= h_inv_dim_plus_one;
  cuda_parts.wcount[pid] *= kernel_norm;
  cuda_parts.wcount_dh[pid] *= h_inv_dim_plus_one;

  const float rho_inv = 1.0 / cuda_parts.rho[pid];

  /* Finish the calculation of the velocity curl components. */
  cuda_parts.rot_v[pid].x *= h_inv_dim_plus_one * rho_inv;
  cuda_parts.rot_v[pid].y *= h_inv_dim_plus_one * rho_inv;
  cuda_parts.rot_v[pid].z *= h_inv_dim_plus_one * rho_inv;

  /* Finish calaculation fo the velocity divergence. */
  cuda_parts.div_v[pid] *= h_inv_dim_plus_one * rho_inv;
}

__device__ void cuda_hydro_part_has_no_neighbours(int pid) {
  const float h = cuda_parts.h[pid];
  const float h_inv = 1.0f / h;
  const float h_inv_dim = cuda_pow_dimension(h_inv);

  cuda_parts.rho[pid] = cuda_parts.mass[pid] * cuda_kernel_root * h_inv_dim;
  cuda_parts.wcount[pid] = cuda_kernel_root * kernel_norm * h_inv_dim;
  cuda_parts.rho_dh[pid] = 0.f;
  cuda_parts.wcount_dh[pid] = 0.f;
  cuda_parts.div_v[pid] = 0.f;
  cuda_parts.rot_v[pid].x = 0.f;
  cuda_parts.rot_v[pid].y = 0.f;
  cuda_parts.rot_v[pid].z = 0.f;
}

__device__ float cuda_pow_gamma(float x) {
#if defined(HYDRO_GAMMA_5_3)
  const float cbrt = cbrtf(x);
  return cbrt * cbrt * x;
#elif defined(HYDRO_GAMMA_7_5)
  return powf(x, 1.4f);
#elif defined(HYDRO_GAMMA_4_3)
  return cbrtf(x) * x;
#elif defined(HYDRO_GAMMA_2_1)
  return x * x;
#else
  printf("The adiabatic index is not defined!");
  return 0.f;
#endif
}

/* USES IDEAL GAS ONLY */
__device__ float cuda_gas_pressure_from_entropy(float density, float entropy) {

  return entropy * cuda_pow_gamma(density);
}

__device__ float cuda_gas_soundspeed_from_pressure(float density, float P) {

  const float density_inv = 1.f / density;
  return sqrtf(hydro_gamma * P * density_inv);
}

__device__ void cuda_hydro_prepare_force(int pid) {
  const float fac_mu = 1.f; /* Will change with cosmological integration. */

  /* Compute the norm of the curl */
  const float curl_v = sqrtf(cuda_parts.rot_v[pid].x * cuda_parts.rot_v[pid].x +
                             cuda_parts.rot_v[pid].y * cuda_parts.rot_v[pid].y +
                             cuda_parts.rot_v[pid].z * cuda_parts.rot_v[pid].z);

  /* Compute the norm of div v */
  const float abs_div_v = fabsf(cuda_parts.div_v[pid]);

  /* Compute the pressure */
  const float pressure = cuda_gas_pressure_from_entropy(
      cuda_parts.rho[pid], cuda_parts.entropy[pid]);

  /* Compute the soundspeed */
  const float soundspeed =
      cuda_gas_soundspeed_from_pressure(cuda_parts.rho[pid], pressure);

  const float rho_inv = 1.f / cuda_parts.rho[pid];
  const float P_over_rho2 = pressure * rho_inv * rho_inv;

  const float balsara =
      abs_div_v /
      (abs_div_v + curl_v + 0.0001f * soundspeed / fac_mu / cuda_parts.h[pid]);

  /* Compute the grah h term*/
  const float grad_h_term =
      1.f / (1.f + hydro_dimension_inv * cuda_parts.h[pid] *
                       cuda_parts.rho_dh[pid] * rho_inv);

  /* Update variables */
  cuda_parts.f[pid] = grad_h_term;
  cuda_parts.P_over_rho2[pid] = P_over_rho2;
  cuda_parts.soundspeed[pid] = soundspeed;
  cuda_parts.balsara[pid] = balsara;
}

__device__ void cuda_doself_subset_density(int pid, int cell) {

  /* Loop over the particles in the cell and interact them with pid. */
  /* This is an inner loop. */
  int parts = cells_cuda[cell].first_part;
  int count = cells_cuda[cell].part_count;
  double pix[3];
  pix[0] = cuda_parts.x_x[pid];
  pix[1] = cuda_parts.x_y[pid];
  pix[2] = cuda_parts.x_z[pid];
  const float hi = cuda_parts.h[pid];
  const float hig2 = hi * hi * kernel_gamma2;
  float rho = 0.0f, rho_dh = 0.0f, div_v = 0.0f, wcount = 0.0f,
        wcount_dh = 0.0f;
  float3 rot_v;
  rot_v.x = 0.0f;
  rot_v.y = 0.0f;
  rot_v.z = 0.0f;
  for (int j = parts; j < parts + count; j++) {
    if (j == pid) continue;
    float r2 = 0.0f;
    float dx[3];
    dx[0] = pix[0] - cuda_parts.x_x[j];
    r2 += dx[0] * dx[0];
    dx[1] = pix[1] - cuda_parts.x_y[j];
    r2 += dx[1] * dx[1];
    dx[2] = pix[2] - cuda_parts.x_z[j];
    r2 += dx[2] * dx[2];
    if (r2 < hig2) {
      float w, dw_dx;
      float dv[3], curlvr[3];

      /* Load mass for particle pj. */
      const float mj = cuda_parts.mass[j];

      /* Get r and 1/r */
      const float r = sqrtf(r2);
      const float ri = 1.0f / r;

      /* Compute the kernel function */
      const float hi_inv = 1.0f / hi;
      const float ui = r * hi_inv;

      cuda_kernel_deval(ui, &w, &dw_dx);

      /* Compute contribution to the density */
      rho += mj * w;
      rho_dh -= mj * (hydro_dimension * w + ui * dw_dx);

      /* Compute condtribution to the number of neighbours */
      wcount += w;
      wcount_dh -= (hydro_dimension * w + ui * dw_dx);

      const float fac = mj * dw_dx * ri;

      float3 piv, pjv;
      piv = cuda_parts.v[pid];
      pjv = cuda_parts.v[j];
      dv[0] = piv.x - pjv.x;
      dx[1] = piv.y - pjv.y;
      dx[2] = piv.z - pjv.z;
      const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

      div_v -= fac * dvdr;

      curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
      curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
      curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];

      rot_v.x += fac * curlvr[0];
      rot_v.y += fac * curlvr[1];
      rot_v.z += fac * curlvr[2];
    }
  }

  /* Write the data for particle pid */
  atomicAdd(&cuda_parts.rho[pid], rho);
  atomicAdd(&cuda_parts.rho_dh[pid], rho_dh);
  atomicAdd(&cuda_parts.wcount[pid], wcount);
  atomicAdd(&cuda_parts.wcount_dh[pid], wcount_dh);
  atomicAdd(&cuda_parts.div_v[pid], div_v);
  atomicAdd(&cuda_parts.rot_v[pid].x, rot_v.x);
  atomicAdd(&cuda_parts.rot_v[pid].y, rot_v.y);
  atomicAdd(&cuda_parts.rot_v[pid].z, rot_v.z);
}

__device__ void cuda_dopair_subset_density(int pid, int cell, int pid_cell) {

  /* Loop over the particles in the cell and interact them with pid. */
  /* This is an inner loop. */
  int parts = cells_cuda[cell].first_part;
  int count = cells_cuda[cell].part_count;
  double pix[3];
  const float hi = cuda_parts.h[pid];
  const float hig2 = hi * hi * kernel_gamma2;
  float rho = 0.0f, rho_dh = 0.0f, div_v = 0.0f, wcount = 0.0f,
        wcount_dh = 0.0f;
  float3 rot_v;
  rot_v.x = 0.0f, rot_v.y = 0.0f, rot_v.z = 0.0f;
  double shift[3] = {0.0, 0.0, 0.0};

  /* Deal with periodicity concerns. */
  for (int k = 0; k < 3; k++) {
    if (cells_cuda[cell].loc[k] - cells_cuda[pid_cell].loc[k] < -dim[k] / 2)
      shift[k] = dim[k];
    else if (cells_cuda[cell].loc[k] - cells_cuda[pid_cell].loc[k] > dim[k] / 2)
      shift[k] = -dim[k];
  }
  pix[0] = cuda_parts.x_x[pid] - shift[0];
  pix[1] = cuda_parts.x_y[pid] - shift[1];
  pix[2] = cuda_parts.x_z[pid] - shift[2];
  for (int j = parts; j < parts + count; j++) {
    float r2 = 0.0f;
    float dx[3];
    dx[0] = pix[0] - cuda_parts.x_x[j];
    r2 += dx[0] * dx[0];
    dx[1] = pix[1] - cuda_parts.x_y[j];
    r2 += dx[1] * dx[1];
    dx[2] = pix[2] - cuda_parts.x_z[j];
    r2 += dx[2] * dx[2];
    if (r2 < hig2) {
      float w, dw_dx;
      float dv[3], curlvr[3];

      /* Load mass for particle pj. */
      const float mj = cuda_parts.mass[j];

      /* Get r and 1/r */
      const float r = sqrtf(r2);
      const float ri = 1.0f / r;

      /* Compute the kernel function */
      const float hi_inv = 1.0f / hi;
      const float ui = r * hi_inv;

      cuda_kernel_deval(ui, &w, &dw_dx);

      /* Compute contribution to the density */
      rho += mj * w;
      rho_dh -= mj * (hydro_dimension * w + ui * dw_dx);

      /* Compute condtribution to the number of neighbours */
      wcount += w;
      wcount_dh -= (hydro_dimension * w + ui * dw_dx);

      const float fac = mj * dw_dx * ri;

      float3 piv, pjv;
      piv = cuda_parts.v[pid];
      pjv = cuda_parts.v[j];
      dv[0] = piv.x - pjv.x;
      dx[1] = piv.y - pjv.y;
      dx[2] = piv.z - pjv.z;
      const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

      div_v -= fac * dvdr;

      curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
      curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
      curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];

      rot_v.x += fac * curlvr[0];
      rot_v.y += fac * curlvr[1];
      rot_v.z += fac * curlvr[2];
    }
  }

  /* Write the data for particle pid */
  atomicAdd(&cuda_parts.rho[pid], rho);
  atomicAdd(&cuda_parts.rho_dh[pid], rho_dh);
  atomicAdd(&cuda_parts.wcount[pid], wcount);
  atomicAdd(&cuda_parts.wcount_dh[pid], wcount_dh);
  atomicAdd(&cuda_parts.div_v[pid], div_v);
  atomicAdd(&cuda_parts.rot_v[pid].x, rot_v.x);
}

/* Device function to fix a single particle that was messed up in the density
 * loop. */
/* This interacts a single particle because the method used on the CPU will not
 * work here*/
/* Its probably super inefficient but it should work at least...*/
__device__ void cuda_hydro_fix_particle(int pid, struct cell_cuda *c) {

  /* First we need to reset everything.. */
  cuda_parts.rho[pid] = 0.f;
  cuda_parts.wcount[pid] = 0.f;
  cuda_parts.wcount_dh[pid] = 0.f;
  cuda_parts.rho_dh[pid] = 0.f;
  cuda_parts.div_v[pid] = 0.f;
  cuda_parts.rot_v[pid].x = 0.f;
  cuda_parts.rot_v[pid].y = 0.f;
  cuda_parts.rot_v[pid].z = 0.f;

  /* Climb up the cell hierarchy. */
  struct cell_cuda *c2 = c;
  for (int cell = (c - cells_cuda); cell >= 0; cell = c2->parent) {
    c2 = &cells_cuda[cell];

    for (int l = 0; l < c->nr_links; l++) {
      if (tasks[l].type == task_type_self ||
          tasks[l].type == task_type_sub_self)
        cuda_doself_subset_density(pid, cell);
      else if (tasks[l].type == task_type_pair ||
               tasks[l].type == task_type_sub_pair) {
        if (tasks[l].ci == cell) {
          cuda_dopair_subset_density(pid, tasks[l].cj, cell);
        } else {
          cuda_dopair_subset_density(pid, tasks[l].ci, cell);
        }
      }
    }
  }
}

/* During ghost work only this block will be accessing the thread, no need for
 * atomics. */
__device__ void do_ghost(struct cell_cuda *c) {

  int part_i = c->first_part;
  int count_i = c->part_count;
  //  const float target_wcount = target_neighbours;
  //  const float max_wcount = target_wcount + delta_neighbours;
  // const float min_wcount = target_wcount - delta_neighbours;
  const float hydro_eta_dim = cuda_eta_neighbours;
  const float eps = cuda_h_tolerance;

  /* Is the cell active? */
  if (!cuda_cell_is_active(c)) return;

  /* Recurse... */
  if (c->split) {
    for (int k = 0; k < 8; k++) {
      if (c->progeny[k] >= 0) do_ghost(&cells_cuda[k]);
    }
  } else {

    /* Loop over the particles in the cell. */
    for (int i = part_i + threadIdx.x; i < part_i + count_i; i+= blockDim.x) {
      float h_new;
      const float h_old = cuda_parts.h[i];
      const float h_old_dim = cuda_pow_dimension(h_old);
      const float h_old_dim_minus_one = cuda_pow_dimension_minus_one(h_old);

      if (!cuda_part_is_active(i)) continue;
      if (cuda_parts.wcount[i] == 0.f) {
        h_new = 2.f * h_old;
      } else {
        /* Finish the density calculation. */
        hydro_end_density(i);

        /* Compute a step of the Newton-Raphson scheme */
        const float n_sum = cuda_parts.wcount[i] * h_old_dim;
        const float n_target = hydro_eta_dim;
        const float f = n_sum - n_target;
        const float f_prime =
            cuda_parts.wcount_dh[i] * h_old_dim +
            hydro_dimension * cuda_parts.wcount[i] * h_old_dim_minus_one;

        h_new = h_old - f / f_prime;

        /* Safety check: truncate to the range [ h_old/2 , 2h_old ]. */
        h_new = min(h_new, 2.f * h_old);
        h_new = max(h_new, 0.5f * h_old);
      }
      /* Did we get the right number of neighbours? */
      if (fabsf(h_new - h_old) > eps * h_old) {

        cuda_parts.h[i] = h_new;

        // If below absolute max try again
        // else give up...
        if (cuda_parts.h[i] < hydro_h_max) {
          cuda_hydro_fix_particle(i, c);
        } else {
          /* The particle is a lost cause */
          cuda_parts.h[i] = hydro_h_max;

          /* Do some damage control if no neighbours were found */
          if (cuda_parts.wcount[i] == cuda_kernel_root * kernel_norm)
            cuda_hydro_part_has_no_neighbours(i);
        }

      } /* correct number of neighbours */

      /* We now have a particle whose smoothing length has convegered */
      /* Time to set the force variables */
      /* Compute variables required for the force loop */
      cuda_hydro_prepare_force(i);

      /* The particle force values are now set. */
      /* Prepare the particle for the force loop over neighbours */
      cuda_parts.a_hydro[i].x = 0.0f;
      cuda_parts.a_hydro[i].y = 0.0f;
      cuda_parts.a_hydro[i].z = 0.0f;
      cuda_parts.entropy_dt[i] = 0.0f;
      cuda_parts.h_dt[i] = 0.0f;
      cuda_parts.v_sig[i] = cuda_parts.soundspeed[i];
    }
  }
}

/* Runner function to retrieve a task index from a queue. */
__device__ int runner_cuda_gettask(struct queue_cuda *q) {

  int tid = -1;
  if (atomicAdd((int *)&q->nr_avail_tasks, -1) <= 0) {
    atomicAdd((int *)&q->nr_avail_tasks, 1);
    return -1;
  }

  /* Main loop */
  while ((tid = cuda_queue_gettask(q)) >= 0) {
    // TODO Do we need to lock anything here? Probably no.
    // TODO Does this need to be a while?
    break;
  }
  if (tid >= 0) {
    q->rec_data[atomicAdd((int *)&q->rec_count, 1)] =
        tid;  // TODO Are we keeping rec_data
  }

  return tid;
}

/* The main kernel. */
__global__ void swift_device_kernel() {
  __shared__ volatile int tid;
  __shared__ volatile int done;
  int i;

  /* Main loop */
  while (1) {
    __syncthreads();
    /* Get a task from the queue. */
    if (threadIdx.x == 0) {
      tid = -1;
      /* Highest Priority queue, unload tasks. */
      if (unload_queue.nr_avail_tasks > 0) {
        tid = runner_cuda_gettask(&unload_queue);
      }

      /* Next highest priority queue, load tasks. Only some blocks look in here
       */
      if (tid < 0 && load_queue.nr_avail_tasks > 0 &&
          blockIdx.x < cuda_numloaders) {
        tid = runner_cuda_gettask(&load_queue);
      }

      /* Finally loop through work queues in priority order. queue 0 is highest
       * priority*/
      for (i = 0; i < cuda_numqueues && tid < 0; i++) {
        if (cuda_queues[i].nr_avail_tasks > 0) {
          tid = runner_cuda_gettask(&cuda_queues[i]);
        }
      }

    }  // Get task from queue

    /* Threads need to wait until they have work to do */
    __syncthreads();

    /* If the tasks are all complete and we don't have anything to do exit*/
    if (tid < 0 && tot_num_tasks == 0) break;

    if (tid < 0) continue;

    int type = tasks[tid].type;
    int subtype = tasks[tid].subtype;

    if (type == type_load) {
      load_cell(tasks[tid].ci);
    } else if (type == type_unload) {
      unload_cell(tasks[tid].ci);
    } else if (type == task_type_pair || type == task_type_sub_pair) {
      if (subtype == task_subtype_density) {
        struct cell_cuda *ci = &cells_cuda[tasks[tid].ci];
        struct cell_cuda *cj = &cells_cuda[tasks[tid].cj];
        dopair_density(ci, cj);
      } else if (subtype == task_subtype_force) {
        struct cell_cuda *ci = &cells_cuda[tasks[tid].ci];
        struct cell_cuda *cj = &cells_cuda[tasks[tid].cj];
        dopair_force(ci, cj);
      }
    } else if (type == task_type_self || type == task_type_sub_self) {
      if (subtype == task_subtype_density) {
        struct cell_cuda *ci = &cells_cuda[tasks[tid].ci];
        doself_density(ci);
      } else if (subtype == task_subtype_force) {
        struct cell_cuda *ci = &cells_cuda[tasks[tid].ci];
        doself_force(ci);
      }
    } else if (type == task_type_ghost) {
      struct cell_cuda *ci = &cells_cuda[tasks[tid].ci];
      do_ghost(ci);
    }

    __syncthreads();

    /* Unlock dependencies*/
    for (i = threadIdx.x; i < tasks[tid].nr_unlock_tasks; i+=blockDim.x) {
      int dependant = tasks[tid].unlocks[i];
      if (atomicSub(&tasks[dependant].wait, 1) == 1 && !tasks[dependant].skip) {
        if (tasks[dependant].type <= type_unload &&
            tasks[dependant].type >= type_implicit_unload) {
          cuda_queue_puttask(&unload_queue, dependant);
        } else {
          if (tasks[dependant].weight >= median_cost) {
            cuda_queue_puttask(&cuda_queues[0], dependant);
          } else {
            cuda_queue_puttask(&cuda_queues[1], dependant);
          }
        }
      }
    }

  }  // End of main loop

  /* Don't need to do any cleanup, all the dependencies and skips and queues are
   * set by CPU. */
}

/* Task function to unload a specific cell with density data instead of force.
 */
__device__ void test_27_unload_cell(int cell_index) {

  /* Get the pointer to the relevant cells. */
  struct cell *cpu_cell = cpu_cells[cell_index];
  struct cell_cuda *cell = &cells_cuda[cell_index];
  struct part *parts = cpu_cell->parts;
  int i;
  /* Get the index to copy the data for the 0th particle in this cell to.*/
  int start = cell->first_part;

  for (i = threadIdx.x; i < cell->part_count; i += blockDim.x) {
    struct part *current = &parts[i];

    /* Copy back the ID and position.*/
    current->id = cuda_parts.id[start + i];
    current->x[0] = cuda_parts.x_x[start + i];
    current->x[1] = cuda_parts.x_y[start + i];
    current->x[2] = cuda_parts.x_z[start + i];

    /* Copy back the velocity*/
    float3 local_v = cuda_parts.v[start + i];
    current->v[0] = local_v.x;
    current->v[1] = local_v.y;
    current->v[2] = local_v.z;

    /* Copy back the acceleration */
    float3 local_a_hydro = cuda_parts.a_hydro[start + i];
    current->a_hydro[0] = local_a_hydro.x;
    current->a_hydro[1] = local_a_hydro.y;
    current->a_hydro[2] = local_a_hydro.z;

    /* Copy back the cutoff, mass, density, entropy and entropy_dt*/
    current->h = cuda_parts.h[start + i];
    current->mass = cuda_parts.mass[start + i];
    current->rho = cuda_parts.rho[start + i];
    current->entropy = cuda_parts.entropy[start + i];
    current->entropy_dt = cuda_parts.entropy_dt[start + i];

    /* Copy back the density union (needed for tests) */
    current->density.wcount = cuda_parts.wcount[start + i];
    current->density.wcount_dh = cuda_parts.wcount_dh[start + i];
    current->density.rho_dh = cuda_parts.rho_dh[start + i];
    current->density.rot_v[0] = cuda_parts.rot_v[start + i].x;
    current->density.rot_v[1] = cuda_parts.rot_v[start + i].y;
    current->density.rot_v[2] = cuda_parts.rot_v[start + i].z;
    current->density.div_v = cuda_parts.div_v[start + i];

    /* Copy back the timebin. */
    current->time_bin = cuda_parts.time_bin[start + i];
  }
}
/*
 _____          _
|  ___|        | |
| |__ _ __   __| |
|  __| '_ \ / _` |
| |__| | | | (_| |
\____/_| |_|\__,_|


 _____  __
|  _  |/ _|
| | | | |_
| | | |  _|
\ \_/ / |
 \___/|_|


 _____
|  __ \
| |  \/_ __  _   _
| | __| '_ \| | | |
| |_\ \ |_) | |_| |
 \____/ .__/ \__,_|
      | |
      |_|
               _
              | |
  ___ ___   __| | ___
 / __/ _ \ / _` |/ _ \
| (_| (_) | (_| |  __/
 \___\___/ \__,_|\___|


  */

/* Host function to check cuda functions don't return errors */
__host__ inline void cudaErrCheck(hipError_t status) {
  if (status != hipSuccess) {
    printf("%s\n", hipGetErrorString(status));
  }
}

/* Host function to give all the cells the IDs required for CUDA order and
 * tasks. */
__host__ void cell_IDs(struct cell *c, int *k) {
  int i;
  c->cuda_ID = *k;
  *k = *k + 1;
  if (c->split) {
    for (i = 0; i < 8; i++) {
      if (c->progeny[i] != NULL) {
        cell_IDs(c->progeny[i], k);
      }
    }
  }
}

/* Host function to create the load, unload and implicit tasks */
__host__ void create_transfer_tasks(struct cell *c, int *k,
                                    int parent_load_task,
                                    int parent_unload_task) {
  tasks_host[*k].unlocks =
      (int *)malloc(sizeof(int) * 9);  // For now we create CPU storage for
                                       // these unlocks for each task. No task
                                       // should have more than 9 unlocks to
                                       // other transfer tasks.
  tasks_host[*k].nr_unlock_tasks = 0;
  tasks_host[*k].size_unlocks = 9;
  if (c->split) {

    /* If the task is split we create implicit tasks to deal with dependencies.
     */
    tasks_host[*k].type = type_implicit_load;
    tasks_host[*k].ci = c->cuda_ID;
    tasks_host[*k].cj = -1;  // These tasks operate on a single cell.
    tasks_host[*k].weight = c->count;
    tasks_host[*k].wait = 0;
    tasks_host[*k].subtype = task_subtype_none;
    tasks_host[*k].skip = 0;
    tasks_host[*k].implicit = 0;
    tasks_host[*k].task = NULL;
    /* The load implicit tasks unlocks the parent's task */
    if (parent_load_task >= 0) {
      tasks_host[*k].unlocks[tasks_host[*k].nr_unlock_tasks++] =
          parent_load_task;
    }
    c->load_task = *k;
    *k = *k + 1;

    /* Create the implicit unload task. */
    tasks_host[*k].unlocks =
        (int *)malloc(sizeof(int) * 9);  // For now we create CPU storage for
                                         // these unlocks for each task. No task
                                         // should have more than 9 unlocks to
                                         // other transfer tasks.
    tasks_host[*k].nr_unlock_tasks = 0;
    tasks_host[*k].size_unlocks = 9;
    tasks_host[*k].type = type_implicit_unload;
    tasks_host[*k].ci = c->cuda_ID;
    tasks_host[*k].cj = -1;  // These tasks operate on a single cell.
    tasks_host[*k].weight = c->count;
    tasks_host[*k].wait = 0;
    tasks_host[*k].subtype = task_subtype_none;
    tasks_host[*k].skip = 0;
    tasks_host[*k].implicit = 0;
    tasks_host[*k].task = NULL;

    /* The unload implicit task is unlocked by the parent task */
    if (parent_unload_task >= 0) {
      tasks_host[parent_unload_task]
          .unlocks[tasks_host[parent_unload_task].nr_unlock_tasks++] = *k;
    }
    c->unload_task = *k;
    *k = *k + 1;

    /* Recurse down the tree. */
    int load = *k - 2;
    int unload = *k - 1;
    for (int i = 0; i < 8; i++) {
      if (c->progeny[i] != NULL)
        create_transfer_tasks(c->progeny[i], k, load, unload);
    }

  } else {
    /* Create the load task*/
    tasks_host[*k].type = type_load;
    tasks_host[*k].ci = c->cuda_ID;
    tasks_host[*k].cj = -1;  // These tasks operate on a single cell.
    tasks_host[*k].weight = c->count;
    tasks_host[*k].wait = 0;
    tasks_host[*k].subtype = task_subtype_none;
    tasks_host[*k].skip = 0;
    tasks_host[*k].implicit = 0;
    tasks_host[*k].task = NULL;
    /* This load task unlocks the parent's task. */
    if (parent_load_task >= 0) {
      tasks_host[*k].unlocks[tasks_host[*k].nr_unlock_tasks++] =
          parent_load_task;
    }
    c->load_task = *k;
    *k = *k + 1;

    /* Create the unload task */
    tasks_host[*k].unlocks =
        NULL;  // unload tasks never unlock anything, end of tree.
    tasks_host[*k].nr_unlock_tasks = 0;
    tasks_host[*k].type = type_unload;
    tasks_host[*k].ci = c->cuda_ID;
    tasks_host[*k].cj = -1;  // These tasks operate on a single cell.
    tasks_host[*k].weight = c->count;
    tasks_host[*k].wait = 0;
    tasks_host[*k].subtype = task_subtype_none;
    tasks_host[*k].skip = 0;
    tasks_host[*k].implicit = 0;
    tasks_host[*k].task = NULL;
    /* The unload task is unlocked by the parent task */
    if (parent_unload_task >= 0) {
      tasks_host[parent_unload_task]
          .unlocks[tasks_host[parent_unload_task].nr_unlock_tasks++] = *k;
    }
    c->unload_task = *k;
    *k = *k + 1;
  }
}

/* Recursive function to initialise the links required for the ghosts.  */
__host__ void init_links(struct cell *c, struct cell_cuda *cell_host) {

  struct link *l = c->density;
  struct cell_cuda *c2 = &cell_host[c->cuda_ID];
  while (l != NULL) {
    c2->links[c2->nr_links++] = l->t->cuda_task;
    l = l->next;
  }

  for (int i = 0; i < 8; i++) {
    /* Recurse */
    if (c->progeny[i] != NULL) {
      init_links(c->progeny[i], cell_host);
    }
  }
}

/* Recursive function to create the cell structures require for the GPU.*/
__host__ void create_cells(struct cell *c, struct cell_cuda *cell_host,
                           struct cell **host_pointers, struct part *parts) {

  /* Set the host pointer. */
  host_pointers[c->cuda_ID] = c;
  struct cell_cuda *c2 = &cell_host[c->cuda_ID];

  c2->loc[0] = c->loc[0];
  c2->loc[1] = c->loc[1];
  c2->loc[2] = c->loc[2];
  c2->width[0] = c->width[0];
  c2->width[1] = c->width[1];
  c2->width[2] = c->width[2];
  c2->h_max = c->h_max;
  c2->first_part = c->parts - parts;
  c2->part_count = c->count;
  if (c->parent != NULL) {
    c2->parent = c->parent->cuda_ID;
  } else {
    c2->parent = -1;
  }
  if (c->super != NULL) {
    c2->super = c->super->cuda_ID;
  } else {
    c2->super = -1;
  }
  c2->ti_end_min = c->ti_end_min;
  c2->ti_end_max = c->ti_end_max;
  c2->dmin = c->dmin;
  c2->nr_links = 0;
  c2->split = c->split;

  for (int i = 0; i < 8; i++) {
    /* Set progeny and recurse. */
    if (c->progeny[i] != NULL) {
      c2->progeny[i] = c->progeny[i]->cuda_ID;
      create_cells(c->progeny[i], cell_host, host_pointers, parts);
    }
  }
}

__host__ int partition(int *list, int left, int right, int pivotIndex) {
  int pivotValue = list[pivotIndex];
  int temp = list[right];
  int i;
  list[right] = list[pivotIndex];
  list[pivotIndex] = list[right];
  int storeIndex = left;
  for (i = left; i < right - 1; i++) {
    if (list[i] < pivotValue) {
      temp = list[storeIndex];
      list[storeIndex] = list[i];
      list[i] = temp;
      storeIndex++;
    }
  }
  temp = list[right];
  list[right] = list[storeIndex];
  list[storeIndex] = temp;
  return storeIndex;
}

__host__ double r2() { return (double)rand() / (double)RAND_MAX; }

/* Host function to perform quickselect */
__host__ int select(int *list, int left, int right, int n) {
  static int value = -1;
  if (value < 0) value = right;
  if (left == right) return list[left];
  int pivotIndex = left + floor(r2() * (right - left + 1));
  pivotIndex = partition(list, left, right, pivotIndex);
  if (n == pivotIndex)
    return list[n];
  else if (n < pivotIndex) {
    return select(list, left, pivotIndex - 1, n);
  } else {
    return select(list, pivotIndex + 1, right, n);
  }
}

/* Host function used for priority cutoff */
#define PERCENTILE 0.8f
__host__ int find_priority_cutoff(struct task_cuda *tasks, int count) {

  int nr_work_tasks = 0;
  int *costs = (int *)malloc(sizeof(int) * count);
  for (int i = 0; i < count; i++) {
    if (tasks[i].type >= type_load) {
      costs[nr_work_tasks++] = tasks[i].weight;
    }
  }
  int result = select(costs, 0, nr_work_tasks - 1,
                      (int)((float)(nr_work_tasks - 1) * PERCENTILE));
  free(costs);
  return result;
}

/* Host function to update the GPU tasks and set skips and dependencies. */
__host__ void update_tasks(struct engine *e) {

  int nr_gpu_tasks;
  int nr_tasks;
  /* Download the cuda_tasks from the GPU. */
  cudaErrCheck(hipMemcpyFromSymbol(&nr_gpu_tasks, HIP_SYMBOL(cuda_numtasks), sizeof(int)));
  cudaErrCheck(hipMemcpyFromSymbol(&nr_tasks, HIP_SYMBOL(tot_num_tasks), sizeof(int)));
  struct task_cuda *gpu_pointer = NULL;
  cudaErrCheck(hipMemcpyFromSymbol(
       &gpu_pointer, HIP_SYMBOL(tasks), sizeof(struct task_cuda *)));  // TODO check.
  struct task_cuda *host_tasks = NULL;
  host_tasks =
      (struct task_cuda *)malloc(sizeof(struct task_cuda) * nr_gpu_tasks);
  cudaErrCheck(hipMemcpy(host_tasks, gpu_pointer,
                          sizeof(struct task_cuda ) * nr_gpu_tasks,
                          hipMemcpyDeviceToHost));
  int cuda_unlock_count;
  cudaErrCheck(hipMemcpyFromSymbol( &cuda_unlock_count, HIP_SYMBOL(cuda_nr_unlocks), sizeof(int) ));
  int *host_unlock_copy = (int*) malloc(sizeof(int) * cuda_unlock_count);
  int *host_unlock_pointer = NULL;
  cudaErrCheck( hipMemcpyFromSymbol( &host_unlock_pointer, HIP_SYMBOL(cuda_unlocks), sizeof(int*) ) );
  cudaErrCheck( hipDeviceSynchronize());
  cudaErrCheck( hipMemcpy( host_unlock_copy, host_unlock_pointer, sizeof(int) * cuda_unlock_count, hipMemcpyDeviceToHost) );

  int task_count=0;

  for (int i = 0; i < nr_gpu_tasks; i++) {
    // Update the skip flag and reset the wait to 0.
    host_tasks[i].wait = 0;
    if (host_tasks[i].type > type_load)
      host_tasks[i].skip = host_tasks[i].task->skip;
    else
      host_tasks[i].skip = 0;
  }

  /* Reset the waits. */
  for (int i = 0; i < nr_gpu_tasks; i++) {
    if (!host_tasks[i].skip) {
      task_count++;
      struct task_cuda *temp_t = &host_tasks[i];
      int *unlocks = host_unlock_copy + (temp_t->unlocks - host_unlock_pointer);
      for (int ii = 0; ii < temp_t->nr_unlock_tasks; ii++) {
          if(!host_tasks[unlocks[ii]].skip)
            host_tasks[unlocks[ii]].wait++;
      }
    }
  }

  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(tot_num_tasks), &task_count, sizeof(int)));
  /* Reset the queue data.*/
  int qsize;
  cudaErrCheck(hipMemcpyFromSymbol(&qsize, HIP_SYMBOL(cuda_queue_size), sizeof(int)));

  /* Remake the data array for the unload q and copy it*/
  /* Download the unload queue. */
  struct queue_cuda unload_host;
  cudaErrCheck(hipMemcpyFromSymbol(&unload_host, HIP_SYMBOL(unload_queue),
                                    sizeof(struct queue_cuda)));

  int *data = (int *)malloc(sizeof(int) * qsize);
  int nr_unload;
  unload_host.count = 0;
  for (int i = 0; i < nr_gpu_tasks; i++) {
    if (host_tasks[i].type <= type_unload &&
        host_tasks[i].type >= type_implicit_unload && !host_tasks[i].skip) {
      if (host_tasks[i].wait == 0) {
        data[unload_host.count++] = i;
      }
      nr_unload++;
    }
  }
  for (int i = unload_host.count; i < qsize; i++) {
    data[i] = -1;
  }
  /* Allocate and copy the data to the device. */
  cudaErrCheck(hipMemcpy((void *)unload_host.data, data, sizeof(int) * qsize,
                          hipMemcpyHostToDevice));
  unload_host.first = 0;
  unload_host.last = unload_host.count;
  unload_host.rec_count = 0;
  unload_host.nr_avail_tasks = unload_host.count;
  unload_host.count = nr_unload;

  /* Copy the queue to the device */
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(unload_queue), &unload_host,
                                  sizeof(struct queue_cuda)));

  /* Download the load queue. */
  struct queue_cuda load_host;
  cudaErrCheck(
      hipMemcpyFromSymbol(&load_host, HIP_SYMBOL(load_queue), sizeof(struct queue_cuda)));
  int nr_load;
  load_host.count = 0;
  for (int i = 0; i < nr_gpu_tasks; i++) {
    if (host_tasks[i].type == type_load && !host_tasks[i].skip) {
      if (host_tasks[i].wait == 0) {
        data[load_host.count++] = i;
      }
      nr_load++;
    }
  }
  for (int i = load_host.count; i < qsize; i++) {
    data[i] = -1;
  }

  /* Allocate and copy the data to the device. */
  cudaErrCheck(hipMemcpy((void *)load_host.data, data, sizeof(int) * qsize,
                          hipMemcpyHostToDevice));
  load_host.first = 0;
  load_host.last = load_host.count;
  load_host.rec_count = 0;
  load_host.nr_avail_tasks = load_host.count;
  load_host.count = nr_load;

  /* Copy the queue to the device */
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(load_queue), &load_host, sizeof(struct queue_cuda)));

  /* Remake the data array for queue[0] and copy it */
  /* Download the work queues. */
  struct queue_cuda work_host[cuda_numqueues];
  cudaErrCheck(hipMemcpyFromSymbol(&work_host, HIP_SYMBOL(cuda_queues),
                                    sizeof(struct queue_cuda) * 2));

  /* Download the priority. */
  int median_host;
  cudaErrCheck(hipMemcpyFromSymbol(&median_host, HIP_SYMBOL(median_cost), sizeof(int)));

  work_host[0].count = 0;
  int nr_work = 0;
  for (int i = 0; i < nr_gpu_tasks; i++) {
    if (host_tasks[i].type > type_load && !host_tasks[i].skip &&
        host_tasks[i].weight >= median_host) {
      if (host_tasks[i].wait == 0) {
        data[work_host[0].count++] = i;
      }
      nr_work++;
    }
  }
  for (int i = work_host[0].count; i < qsize; i++) {
    data[i] = -1;
  }

  cudaErrCheck(hipMemcpy((void *)work_host[0].data, data, sizeof(int) * qsize,
                          hipMemcpyHostToDevice));
  work_host[0].first = 0;
  work_host[0].last = work_host[0].count;
  work_host[0].rec_count = 0;
  work_host[0].nr_avail_tasks = work_host[0].count;
  work_host[0].count = nr_work;

  /* Remake the data array for queue[1] and copy it */
  work_host[1].count = 0;
  nr_work = 0;
  for (int i = 0; i < nr_gpu_tasks; i++) {
    if (host_tasks[i].type > type_load && !host_tasks[i].skip &&
        host_tasks[i].weight < median_host) {
      if (host_tasks[i].wait == 0) {
        data[work_host[1].count++] = i;
      }
      nr_work++;
    }
  }
  for (int i = work_host[1].count; i < qsize; i++) {
    data[i] = -1;
  }

  cudaErrCheck(hipMemcpy((void *)work_host[1].data, data, sizeof(int) * qsize,
                          hipMemcpyHostToDevice));
  work_host[1].first = 0;
  work_host[1].last = work_host[1].count;
  work_host[1].rec_count = 0;
  work_host[1].nr_avail_tasks = work_host[1].count;
  work_host[1].count = nr_work;

  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_queues), &work_host,
                                  sizeof(struct queue_cuda) * 2));

  /* Copy the tasks back to the GPU. */
  cudaErrCheck(hipMemcpy(gpu_pointer, host_tasks,
                          sizeof(struct task_cuda) * nr_gpu_tasks,
                          hipMemcpyHostToDevice));

  /* Update simulation constants
__device__ __constant__ integertime_t ti_current;
__device__ __constant__ double dim[3];
__device__ __constant__ timebin_t max_active_bin; */
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(ti_current), &e->ti_current, sizeof(integertime_t)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(dim), &e->s->dim, sizeof(double) * 3));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(max_active_bin), &e->max_active_bin,
                                  sizeof(timebin_t)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
      delta_neighbours), &e->hydro_properties->delta_neighbours, sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(target_neighbours),
                                  &e->hydro_properties->target_neighbours,
                                  sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(hydro_h_max), &e->hydro_properties->h_max,
                                  sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
      cuda_h_tolerance), &e->hydro_properties->h_tolerance, sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_eta_neighbours),
                                  &e->hydro_properties->eta_neighbours,
                                  sizeof(float)));
  /* Clean up */
  free(host_tasks);
  free(data);
}

/* Host function to check if the supplied task should have a GPU version
 * created. */
__host__ int is_gpu_task(struct task *t) {

  int result = 0;
  for (int i = 0; i < num_gpu_types; i++) {
    if (t->type == gpu_work_task_array[i]) result = 1;
  }
  return result;
}

/* Host function to create the GPU tasks. Should be called whenever the tasks
 * are recreated */
/* This function ignores skips but should work. Call update_tasks to ensure
 * skips are set and */
/* waits are set correctly for the skips. */
__host__ void create_tasks(struct engine *e) {

  struct scheduler *sched = &e->sched;
  struct space *s = e->s;
  int num_gpu_tasks = 0;
  int i, k;
  struct cell *c;
  static int firstrun = 0;

  /* We only create density, ghost and force tasks on the device at current. */
  for (i = 0; i < sched->nr_tasks; i++) {
    if (is_gpu_task(&sched->tasks[i])) num_gpu_tasks++;
  }

  /* We also create a load and unload task for every cell in the system */
  num_gpu_tasks += s->tot_cells * 2;

  /* Allocate page-locked memory for the host version of the GPU tasks. */
  cudaErrCheck(hipHostMalloc((void **)&tasks_host,
                              num_gpu_tasks * sizeof(struct task_cuda)));

  k = 0;
  /* Loop through the cells and give them all an ID. */
  for (i = 0; i < s->cdim[0] * s->cdim[1] * s->cdim[2]; i++) {
    c = &s->cells_top[i];
    cell_IDs(c, &k);
  }

  k = 0;
  /* Create the tasks. */
  for (i = 0; i < sched->nr_tasks; i++) {

    if (is_gpu_task(&sched->tasks[i])) {
      /* Copy the data to the CUDA task. */
      struct task *t = &sched->tasks[i];
      tasks_host[k].flags = t->flags;
      tasks_host[k].rank = t->rank;
      tasks_host[k].weight = t->weight;
      tasks_host[k].nr_unlock_tasks = (int)t->nr_unlock_tasks;
      tasks_host[k].type = t->type;
      tasks_host[k].subtype = t->subtype;
      tasks_host[k].skip = t->skip;
      tasks_host[k].implicit = t->implicit;
      tasks_host[k].size_unlocks = 0;

      tasks_host[k].ci = t->ci->cuda_ID;
      if(t->cj != NULL)
        tasks_host[k].cj = t->cj->cuda_ID;

      /* We have a double linked structure because its easier to create. */
      tasks_host[k].task = t;
      t->cuda_task = k;
      k++;
    }
  }

  /* Create the data transfer tasks. */
  for (i = 0; i < s->cdim[0] * s->cdim[1] * s->cdim[2]; i++) {
    c = &s->cells_top[i];
    create_transfer_tasks(c, &k, -1, -1);
  }

  /* Check we got this right initially.. */
  if (k != num_gpu_tasks) {
    error("We created a different number of GPU tasks than expected");
  }

  /* Now we have the tasks, time to start working on the dependencies. */

  /* Loop through the tasks */
  for (i = 0; i < num_gpu_tasks; i++) {
    /* The transfer tasks dependencies are done anyway so skip them. */
    if (tasks_host[i].type == type_load || tasks_host[i].type == type_unload ||
        tasks_host[i].type == type_implicit_load ||
        tasks_host[i].type == type_implicit_unload)
      continue;

    /* Get the task. */
    struct task_cuda *t = &tasks_host[i];

    /* How many dependencies did the CPU task have. */
    int deps = t->task->nr_unlock_tasks;

    /* If it is a force task then it also unlocks the unload tasks. */
    if (t->subtype == task_subtype_force) {
      deps++;
      /* If its a pair force task then it needs 2 unlocks. */
      if (t->type == task_type_pair) {
        deps++;
      }
    }

    /* Allocate some CPU memory for the unlocks. */
    t->unlocks = (int *)malloc(sizeof(int) * deps);
    t->size_unlocks = deps;
    t->nr_unlock_tasks = 0;

    /* Copy the dependencies */
    for (int j = 0; j < t->task->nr_unlock_tasks; j++) {
      if(t->task->unlock_tasks[j]->cuda_task >= 0)
        t->unlocks[t->nr_unlock_tasks++] = t->task->unlock_tasks[j]->cuda_task;
    }

    /* If it is a force task then add the unload tasks.*/
    if (t->subtype == task_subtype_force) {
      t->unlocks[t->nr_unlock_tasks++] = t->task->ci->unload_task;
      if (t->type == task_type_pair) {
        t->unlocks[t->nr_unlock_tasks++] = t->task->cj->unload_task;
      }
    }

    /* If it is a density task then it is unlocked by the load task. */
    if (t->subtype == task_subtype_density) {
      /* We may need to stretch the load task's unlocks */
      if (tasks_host[t->task->ci->load_task].nr_unlock_tasks ==
          tasks_host[t->task->ci->load_task].size_unlocks) {

        int *temp = (int *)malloc(
            sizeof(int) * tasks_host[t->task->ci->load_task].size_unlocks * 2);
        memcpy(
            temp, tasks_host[t->task->ci->load_task].unlocks,
            sizeof(int) * tasks_host[t->task->ci->load_task].nr_unlock_tasks);
        tasks_host[t->task->ci->load_task].size_unlocks *= 2;
        free(tasks_host[t->task->ci->load_task].unlocks);
        tasks_host[t->task->ci->load_task].unlocks = temp;
      }
      tasks_host[t->task->ci->load_task]
          .unlocks[tasks_host[t->task->ci->load_task].nr_unlock_tasks++] = i;

      if (t->type == task_type_pair) {
        /* We may need to stretch the load task's unlocks */
        if (tasks_host[t->task->cj->load_task].nr_unlock_tasks ==
            tasks_host[t->task->cj->load_task].size_unlocks) {

          int *temp = (int *)malloc(
              sizeof(int) * tasks_host[t->task->cj->load_task].size_unlocks *
              2);
          memcpy(
              temp, tasks_host[t->task->cj->load_task].unlocks,
              sizeof(int) * tasks_host[t->task->cj->load_task].nr_unlock_tasks);
          tasks_host[t->task->cj->load_task].size_unlocks *= 2;
          free(tasks_host[t->task->cj->load_task].unlocks);
          tasks_host[t->task->cj->load_task].unlocks = temp;
        }
        tasks_host[t->task->cj->load_task]
            .unlocks[tasks_host[t->task->cj->load_task].nr_unlock_tasks++] = i;

      }  // If is pair task.

    }  // Load to density task dependencies.

  }  // Loop over the tasks.

  /* Now we have the dependencies we need to squash them into a single array. */

  /* First we count how many there are.*/
  int num_deps = 0;
  for (i = 0; i < num_gpu_tasks; i++) {
    num_deps += tasks_host[i].nr_unlock_tasks;
  }

  /* Create a storage location for the dependency array. */
  int *host_dependencies = (int *)malloc(sizeof(int) * num_deps);
  int deps_filled = 0;

  /* Add the arrays, update the pointers, remove the small arrays. */
  for (i = 0; i < num_gpu_tasks; i++) {
    memcpy(&host_dependencies[deps_filled], tasks_host[i].unlocks,
           tasks_host[i].nr_unlock_tasks * sizeof(int));
    free(tasks_host[i].unlocks);
    tasks_host[i].unlocks = &host_dependencies[deps_filled];
    deps_filled += tasks_host[i].nr_unlock_tasks;
  }

  /* Set the waits! */
  for (i = 0; i < num_gpu_tasks; i++) {
    tasks_host[i].wait = 0;
  }
  for (i = 0; i < num_gpu_tasks; i++) {
      struct task_cuda *temp_t = &tasks_host[i];
      for (int ii = 0; ii < temp_t->nr_unlock_tasks; ii++) {
          tasks_host[temp_t->unlocks[ii]].wait++;
      }
  }

  /* Allocate storage for the dependencies on the GPU.*/
  int *gpu_dependencies = NULL;
  if (firstrun) {
    /* If we already have an array for this we need to remove it. */
    cudaErrCheck(
        hipMemcpyFromSymbol(gpu_dependencies, HIP_SYMBOL(&cuda_unlocks), sizeof(int *)));
    hipFree(gpu_dependencies);
    gpu_dependencies = NULL;
  }
  cudaErrCheck(hipMalloc((void **)&gpu_dependencies, sizeof(int) * num_deps));
  /* Start copying the dependency array to the device */
  cudaErrCheck(hipMemcpy(gpu_dependencies, host_dependencies,
                               sizeof(int) * num_deps, hipMemcpyHostToDevice));

  /* We need the task's unlock pointers to point at the device stuff, which we
   * do with pointer maths */
  for (i = 0; i < num_gpu_tasks; i++) {
    int *temp_p =
        gpu_dependencies + (tasks_host[i].unlocks - host_dependencies);
    tasks_host[i].unlocks = temp_p;
  }

  /* Wait for the transfer to complete.*/
  cudaErrCheck(hipDeviceSynchronize());

  /* Copy the new device array to where it will be visible. */
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(cuda_unlocks), &gpu_dependencies, sizeof(int *)));
  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL(cuda_nr_unlocks), &num_deps, sizeof(int)));

  /* Copy the tasks to the device. */
  struct task_cuda *gpu_tasks = NULL;
  if (firstrun) {
    cudaErrCheck(
        hipMemcpyFromSymbol(gpu_tasks, HIP_SYMBOL(&tasks), sizeof(struct task_cuda *)));
    hipFree(gpu_tasks);
    gpu_tasks = NULL;
  }
  cudaErrCheck(hipMalloc((void **)&gpu_tasks,
                          sizeof(struct task_cuda) * num_gpu_tasks));

  cudaErrCheck(hipMemcpy(gpu_tasks, tasks_host,
                          sizeof(struct task_cuda) * num_gpu_tasks,
                          hipMemcpyHostToDevice));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(tasks), &gpu_tasks, sizeof(struct task_cuda *)));

  /* Create the cuda_cells on the CPU. */
  struct cell_cuda *cell_host =
      (struct cell_cuda *)malloc(sizeof(struct cell_cuda) * s->tot_cells);
  struct cell **host_pointers =
      (struct cell **)malloc(sizeof(struct cell *) * s->tot_cells);
  k = 0;
  for (int i = 0; i < s->nr_cells; i++) {
    c = &s->cells_top[i];
    /*Create cells recursively. */
    create_cells(c, cell_host, host_pointers, s->parts);
  }

  /* Need to setup the links */
  for (int i = 0; i < s->nr_cells; i++) {
    c = &s->cells_top[i];
    init_links(c, cell_host);
  }

  /* Allocate space on the device for the cells. */
  struct cell_cuda *cell_device = NULL;
  struct cell *pointers_device = NULL;
  if (firstrun) {
    /* If we already have an array for this we need to remove it. */
    cudaErrCheck(hipMemcpyFromSymbol(cell_device, HIP_SYMBOL(&cells_cuda),
                                      sizeof(struct cell_cuda *)));
    hipFree(cell_device);
    cudaErrCheck(hipMemcpyFromSymbol(pointers_device, HIP_SYMBOL(&cpu_cells),
                                      sizeof(struct cell **)));
    hipFree(pointers_device);
    cell_device = NULL;
    pointers_device = NULL;
  }
  cudaErrCheck(hipMalloc((void **)&cell_device,
                          sizeof(struct cell_cuda) * s->tot_cells));
  cudaErrCheck(hipMalloc((void **)&pointers_device,
                          sizeof(struct cell *) * s->tot_cells));

  /* Copy the cells and pointers to the device and set up the symbol. */
  cudaErrCheck(hipMemcpy(cell_device, cell_host,
                          sizeof(struct cell_cuda) * s->tot_cells,
                          hipMemcpyHostToDevice));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(cells_cuda), &cell_device, sizeof(struct cell_cuda *)));

  cudaErrCheck(hipMemcpy(pointers_device, host_pointers,
                          sizeof(struct cell *) * s->tot_cells,
                          hipMemcpyHostToDevice));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(cpu_cells), &pointers_device, sizeof(struct cell **)));

  /* Setup the queues. */
  /* We have 4 queues, one containing unload & implicit tasks. */
  /* One containing load tasks only. */
  /* One containing high priority work tasks.*/
  /* Last one containing all other tasks. */
  struct queue_cuda load_host;
  struct queue_cuda unload_host;
  struct queue_cuda work_host[cuda_numqueues];
  int nr_load = 0, nr_unload = 0, nr_high = 0, nr_low = 0;

  /* Compute the 80th percentile for the work priorities.*/
  int cut = find_priority_cutoff(tasks_host, num_gpu_tasks);

  /* cuda_queue_size is lazily set to fix all of the tasks in for now. If this
     becomes an issue
     it can be reduced */
  int qsize = max(num_gpu_tasks, 256);
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_queue_size), &qsize, sizeof(int)));


  /* Create the queues */

  /* Create the buffers used to initialised the data and rec_data arrays. */
  int *data, *data2;

  if ((data = (int *)malloc(sizeof(int) * qsize)) == NULL)
    error("Failed to allocate the data buffer on the host.");
  if ((data2 = (int *)malloc(sizeof(int) * qsize)) == NULL)
    error("Failed to allocate the rec_data buffer on the host.");

  load_host.count = 0;
  /* Find the load tasks */
  for (i = 0; i < num_gpu_tasks; i++) {
    if (tasks_host[i].type == type_load) {
      if (tasks_host[i].wait == 0) {
        data[load_host.count] = i;
        data2[load_host.count++] = -1;
      }
      nr_load++;
    }
  }

  for (i = load_host.count; i < qsize; i++) {
    data[i] = -1;
    data2[i] = -1;
  }

  /* Allocate and copy the data to the device. */
  cudaErrCheck(hipMalloc(&load_host.data, sizeof(int) * qsize));
  cudaErrCheck(hipMemcpy((void *)load_host.data, data, sizeof(int) * qsize,
                          hipMemcpyHostToDevice));
  cudaErrCheck(hipMalloc(&load_host.rec_data, sizeof(int) * qsize));
  cudaErrCheck(hipMemcpy((void *)load_host.rec_data, data2,
                          sizeof(int) * qsize, hipMemcpyHostToDevice));
  load_host.first = 0;
  load_host.last = load_host.count;
  load_host.rec_count = 0;
  load_host.nr_avail_tasks = load_host.count;
  load_host.count = nr_load;

  /* Copy the queue to the device */
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(load_queue), &load_host, sizeof(struct queue_cuda)));

  /* Create the unload queue. */
  unload_host.count = 0;
  for (i = 0; i < num_gpu_tasks; i++) {
    if (tasks_host[i].type <= type_unload &&
        tasks_host[i].type >= type_implicit_unload) {
      if (tasks_host[i].wait == 0) {
        data[unload_host.count] = i;
        data2[unload_host.count++] = -1;
      }
      nr_unload++;
    }
  }
  for (i = unload_host.count; i < qsize; i++) {
    data[i] = -1;
    data2[i] = -1;
  }

  /* Allocate and copy the data to the device. */
  cudaErrCheck(hipMalloc(&unload_host.data, sizeof(int) * qsize));
  cudaErrCheck(hipMemcpy((void *)unload_host.data, data, sizeof(int) * qsize,
                          hipMemcpyHostToDevice));
  cudaErrCheck(hipMalloc(&unload_host.rec_data, sizeof(int) * qsize));
  cudaErrCheck(hipMemcpy((void *)unload_host.rec_data, data2,
                          sizeof(int) * qsize, hipMemcpyHostToDevice));
  unload_host.first = 0;
  unload_host.last = unload_host.count;
  unload_host.rec_count = 0;
  unload_host.nr_avail_tasks = unload_host.count;
  unload_host.count = nr_unload;

  /* Copy the queue to the device */
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(unload_queue), &unload_host,
                                  sizeof(struct queue_cuda)));

  /* Create the high priority queue. */

  work_host[0].count = 0;
  for (i = 0; i < num_gpu_tasks; i++) {
    if (tasks_host[i].type > type_load && tasks_host[i].weight >= cut) {
      if (tasks_host[i].wait == 0) {
        data[work_host[0].count] = i;
        data2[work_host[0].count++] = -1;
      }
      nr_high++;
    }
  }
  for (i = work_host[0].count; i < qsize; i++) {
    data[i] = -1;
    data2[i] = -1;
  }

  /* Allocate and copy the data to the device. */
  cudaErrCheck(hipMalloc(&work_host[0].data, sizeof(int) * qsize));
  cudaErrCheck(hipMemcpy((void *)work_host[0].data, data, sizeof(int) * qsize,
                          hipMemcpyHostToDevice));
  cudaErrCheck(hipMalloc(&work_host[0].rec_data, sizeof(int) * qsize));
  cudaErrCheck(hipMemcpy((void *)work_host[0].rec_data, data2,
                          sizeof(int) * qsize, hipMemcpyHostToDevice));
  work_host[0].first = 0;
  work_host[0].last = work_host[0].count;
  work_host[0].rec_count = 0;
  work_host[0].nr_avail_tasks = work_host[0].count;
  work_host[0].count = nr_high;

  /* Create the low priority queue. */
  work_host[1].count = 0;
  for (i = 0; i < num_gpu_tasks; i++) {
    if (tasks_host[i].type > type_load && tasks_host[i].weight < cut) {
      if (tasks_host[i].wait == 0) {
        data[work_host[0].count] = i;
        data2[work_host[0].count++] = -1;
      }
      nr_low++;
    }
  }
  work_host[1].first = 0;
  work_host[1].last = work_host[0].count;
  work_host[1].rec_count = 0;
  work_host[1].nr_avail_tasks = work_host[1].count;
  work_host[1].count = nr_low;
  /* Allocate and copy the data to the device. */
  cudaErrCheck(hipMalloc(&work_host[1].data, sizeof(int) * qsize));
  cudaErrCheck(hipMemcpy((void *)work_host[1].data, data, sizeof(int) * qsize,
                          hipMemcpyHostToDevice));
  cudaErrCheck(hipMalloc(&work_host[1].rec_data, sizeof(int) * qsize));
  cudaErrCheck(hipMemcpy((void *)work_host[1].rec_data, data2,
                          sizeof(int) * qsize, hipMemcpyHostToDevice));

  /* Copy the work queues to the GPU */
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_queues), &work_host,
                                  sizeof(struct queue_cuda) * 2));

  /* Set some other values needed for scheduling. */
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_queue_size), &qsize, sizeof(int)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_numtasks), &num_gpu_tasks, sizeof(int)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(tot_num_tasks), &num_gpu_tasks, sizeof(int)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(median_cost), &cut, sizeof(int)));

  /* Allocate particle arrays on the GPU */
  struct particle_arrays host_particles;
  if (firstrun) {
    cudaErrCheck(hipMemcpyFromSymbol(&host_particles, HIP_SYMBOL(cuda_parts),
                                      sizeof(struct particle_arrays)));

    cudaErrCheck(hipFree(host_particles.id));
    cudaErrCheck(hipFree(host_particles.x_x));
    cudaErrCheck(hipFree(host_particles.x_y));
    cudaErrCheck(hipFree(host_particles.x_z));
    cudaErrCheck(hipFree(host_particles.v));
    cudaErrCheck(hipFree(host_particles.a_hydro));
    cudaErrCheck(hipFree(host_particles.h));
    cudaErrCheck(hipFree(host_particles.mass));
    cudaErrCheck(hipFree(host_particles.rho));
    cudaErrCheck(hipFree(host_particles.entropy));

    cudaErrCheck(hipFree(host_particles.wcount));
    cudaErrCheck(hipFree(host_particles.wcount_dh));
    cudaErrCheck(hipFree(host_particles.rho_dh));
    cudaErrCheck(hipFree(host_particles.rot_v));
    cudaErrCheck(hipFree(host_particles.div_v));

    cudaErrCheck(hipFree(host_particles.balsara));
    cudaErrCheck(hipFree(host_particles.f));
    cudaErrCheck(hipFree(host_particles.P_over_rho2));
    cudaErrCheck(hipFree(host_particles.soundspeed));
    cudaErrCheck(hipFree((void*) host_particles.v_sig));
    cudaErrCheck(hipFree(host_particles.h_dt));
    cudaErrCheck(hipFree(host_particles.time_bin));
  }

  cudaErrCheck(hipMalloc(&host_particles.id,
                          sizeof(long long int) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.x_x, sizeof(double) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.x_y, sizeof(double) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.x_z, sizeof(double) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.v, sizeof(float3) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.a_hydro, sizeof(float3) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.h, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.mass, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.rho, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.entropy, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(hipMalloc(&host_particles.entropy_dt,
                          sizeof(float) * e->total_nr_parts));

  cudaErrCheck(
      hipMalloc(&host_particles.wcount, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.wcount_dh, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.rho_dh, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.rot_v, sizeof(float3) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.div_v, sizeof(float) * e->total_nr_parts));

  cudaErrCheck(
      hipMalloc(&host_particles.balsara, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.f, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(hipMalloc(&host_particles.P_over_rho2,
                          sizeof(float) * e->total_nr_parts));
  cudaErrCheck(hipMalloc(&host_particles.soundspeed,
                          sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.v_sig, sizeof(float) * e->total_nr_parts));
  cudaErrCheck(
      hipMalloc(&host_particles.h_dt, sizeof(float) * e->total_nr_parts));

  cudaErrCheck(hipMalloc(&host_particles.time_bin,
                          sizeof(timebin_t) * e->total_nr_parts));

  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_parts), &host_particles,
                                  sizeof(struct particle_arrays)));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(cuda_nr_parts), &e->total_nr_parts, sizeof(int)));

  if (!firstrun) {
    float host_kernel_coeffs[(kernel_degree + 1) * (kernel_ivals + 1)];
    for (int a = 0; a < (kernel_degree + 1) * (kernel_ivals + 1); a++) {
      host_kernel_coeffs[a] = kernel_coeffs[a];
    }
    cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
        cuda_kernel_coeffs), &host_kernel_coeffs,
        sizeof(float) * (kernel_degree + 1) * (kernel_ivals + 1)));
  }

  /* This is no longer the first run. */
  firstrun = 1;
  /* Make sure we free everything we made here otherwise leaky code. */
  /* Free the tasks_host array. */
  cudaErrCheck(hipHostFree(tasks_host));
  tasks_host = NULL;
  /* Free the data and data 2 arrays.*/
  free(data);
  data = NULL;
  free(data2);
  data2 = NULL;
  /* Host dependency array has been copied now, so time to remove it.*/
  free(host_dependencies);
  host_dependencies = NULL;
  /* Free cell_host and host_pointers */
  free(cell_host);
  cell_host = NULL;
  free(host_pointers);
  host_pointers = NULL;
}

__host__ void run_cuda() {
  printf("running cuda\n");
  swift_device_kernel << <num_blocks, num_cuda_threads>>> ();
  cudaErrCheck(hipDeviceSynchronize());
}

/* Make the tests! */

__global__ void test_27_kernel() {

  /* Load the particle data. */
  for (int i = 0; i < 27; i++) {
    load_cell(i);
  }
__syncthreads();
  /* Compute the density pair tasks*/
  for (int i = 0; i < 27; i++) {
    if (i == 13) continue;
    dopair_density(&cells_cuda[13], &cells_cuda[i]);
    //      dopair_density(&cells_cuda[i], &cells_cuda[13]);
  }
  /* Compute the self task. */
  doself_density(&cells_cuda[13]);

  __syncthreads();
  /* Unload the particle data. */
  for (int i = 0; i < 27; i++) {
    test_27_unload_cell(i);
  }
}

__global__ void test_125_kernel() {

  /* Load the particle data. */
  for (int i = 0; i < 125; i++) {
    load_cell(i);
  }
__syncthreads();
  /* Run all the pairs (only once !)*/
  for (int i = 0; i < 5; i++) {
    for (int j = 0; j < 5; j++) {
      for (int k = 0; k < 5; k++) {

        struct cell_cuda *ci = &cells_cuda[i * 25 + j * 5 + k];

        for (int ii = -1; ii < 2; ii++) {
          int iii = i + ii;
          if (iii < 0 || iii >= 5) continue;
          iii = (iii + 5) % 5;
          for (int jj = -1; jj < 2; jj++) {
            int jjj = j + jj;
            if (jjj < 0 || jjj >= 5) continue;
            jjj = (jjj + 5) % 5;
            for (int kk = -1; kk < 2; kk++) {
              int kkk = k + kk;
              if (kkk < 0 || kkk >= 5) continue;
              kkk = (kkk + 5) % 5;

              struct cell_cuda *cj = &cells_cuda[iii * 25 + jjj * 5 + kkk];

              if (cj > ci) dopair_density(ci, cj);
              if (cj > ci) dopair_density(cj, ci);
            }
          }
        }
      }
    }
  }
  /* And now the self-interaction for the central cells*/
  for (int i = 1; i < 4; i++) {
    for (int j = 1; j < 4; j++) {
      for (int k = 1; k < 4; k++) {
        doself_density(&cells_cuda[i * 25 + j * 5 + k]);
      }
    }
  }
__syncthreads();

  /* And now the ghost interaction for the central cells*/
  for (int i = 1; i < 4; i++) {
    for (int j = 1; j < 4; j++) {
      for (int k = 1; k < 4; k++) {
        do_ghost(&cells_cuda[i * 25 + j * 5 + k]);
      }
    }
  }
__syncthreads();

  /* And now the force pair interaction for the central cells*/
  for (int i = 1; i < 4; i++) {
    for (int j = 1; j < 4; j++) {
      for (int k = 1; k < 4; k++) {
        if((i*25 + j*5 + k) == 62) continue;
        dopair_force(&cells_cuda[62], &cells_cuda[i * 25 + j * 5 + k]);
        dopair_force(&cells_cuda[i * 25 + j * 5 + k], &cells_cuda[62]);
      }
    }
  }

  /* Force self interaction for the central cell */
  doself_force(&cells_cuda[62]);
__syncthreads();
  for (int i = 0; i < 125; i++) {
    unload_cell(i);
  }
}

__host__ void test_125_cells(struct cell **cells, struct cell *main_cell,
                             struct part *parts, struct engine *e) {
  /* Compute the particle count. */
  int num_part_host = 0;
  for (int i = 0; i < 125; i++) {
    num_part_host += cells[i]->count;
    cells[i]->cuda_ID = i;
  }
  /* Allocate particle arrays on the device. */
  struct particle_arrays host_particles;
  cudaErrCheck(
      hipMalloc(&host_particles.id, sizeof(long long int) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.x_x, sizeof(double) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.x_y, sizeof(double) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.x_z, sizeof(double) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.v, sizeof(float3) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.a_hydro, sizeof(float3) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.h, sizeof(float) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.mass, sizeof(float) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.rho, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.entropy, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.entropy_dt, sizeof(float) * num_part_host));

  cudaErrCheck(
      hipMalloc(&host_particles.wcount, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.wcount_dh, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.rho_dh, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.rot_v, sizeof(float3) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.div_v, sizeof(float) * num_part_host));

  cudaErrCheck(
      hipMalloc(&host_particles.balsara, sizeof(float) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.f, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.P_over_rho2, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.soundspeed, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.v_sig, sizeof(float) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.h_dt, sizeof(float) * num_part_host));

  cudaErrCheck(
      hipMalloc(&host_particles.time_bin, sizeof(timebin_t) * num_part_host));
  float host_kernel_coeffs[(kernel_degree + 1) * (kernel_ivals + 1)];
  for (int a = 0; a < (kernel_degree + 1) * (kernel_ivals + 1); a++) {
    host_kernel_coeffs[a] = kernel_coeffs[a];
  }
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
      cuda_kernel_coeffs), &host_kernel_coeffs,
      sizeof(float) * (kernel_degree + 1) * (kernel_ivals + 1)));
  integertime_t current = 8;
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(ti_current), &current, sizeof(integertime_t)));
  timebin_t current2 = 56;
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(max_active_bin), &current2, sizeof(timebin_t)));

  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_parts), &host_particles,
                                  sizeof(struct particle_arrays)));

  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_nr_parts), &num_part_host, sizeof(int)));

  /* Create the cells for the device. */
  struct cell_cuda *cell_host;
  cudaErrCheck(
      hipHostMalloc((void **)&cell_host, sizeof(struct cell_cuda) * 125));
  struct cell **host_pointers =
      (struct cell **)malloc(sizeof(struct cell *) * 125);
  for (int i = 0; i < 125; i++) {
    struct cell *c = cells[i];
    /*Create cells recursively. */
    create_cells(c, cell_host, host_pointers, parts);
  }
  /* Allocate space on the device for the cells. */
  struct cell_cuda *cell_device = NULL;
  struct cell *pointers_device = NULL;
  cudaErrCheck(
      hipMalloc((void **)&cell_device, sizeof(struct cell_cuda) * 125));
  cudaErrCheck(
      hipMalloc((void **)&pointers_device, sizeof(struct cell *) * 125));

  /* Copy the cells and pointers to the device and set up the symbol. */
  cudaErrCheck(hipMemcpy(cell_device, cell_host,
                          sizeof(struct cell_cuda) * 125,
                          hipMemcpyHostToDevice));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(cells_cuda), &cell_device, sizeof(struct cell_cuda *)));

  cudaErrCheck(hipMemcpy(pointers_device, host_pointers,
                          sizeof(struct cell *) * 125, hipMemcpyHostToDevice));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(cpu_cells), &pointers_device, sizeof(struct cell **)));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(ti_current), &e->ti_current, sizeof(integertime_t)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(dim), &e->s->dim, sizeof(double) * 3));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(max_active_bin), &e->max_active_bin,
                                  sizeof(timebin_t)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
      delta_neighbours), &e->hydro_properties->delta_neighbours, sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(target_neighbours),
                                  &e->hydro_properties->target_neighbours,
                                  sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(hydro_h_max), &e->hydro_properties->h_max,
                                  sizeof(float)));
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(ti_current), &e->ti_current, sizeof(integertime_t)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(dim), &e->s->dim, sizeof(double) * 3));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(max_active_bin), &e->max_active_bin,
                                  sizeof(timebin_t)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
      delta_neighbours), &e->hydro_properties->delta_neighbours, sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(target_neighbours),
                                  &e->hydro_properties->target_neighbours,
                                  sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(hydro_h_max), &e->hydro_properties->h_max,
                                  sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
      cuda_h_tolerance), &e->hydro_properties->h_tolerance, sizeof(float)));
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_eta_neighbours),
                                  &e->hydro_properties->eta_neighbours,
                                  sizeof(float)));
  /* Clean up */
  /* We copied the cells and cpu pointers to the GPU, setup the cells and create
   * the particle arrays. */
  /* Time to launch the kernel. */
  test_125_kernel << <1, 128>>> ();  // Single block.
  hipDeviceSynchronize();
  // Clean up
  cudaErrCheck(hipHostFree(cell_host));
  free(host_pointers);
}

__host__ void test_27_cells(struct cell **cells, struct cell *main_cell,
                            struct part *parts) {

  /* Compute the particle count. */
  int num_part_host = 0;
  for (int i = 0; i < 27; i++) {
    num_part_host += cells[i]->count;
    cells[i]->cuda_ID = i;
  }

  /* Allocate particle arrays on the device. */
  struct particle_arrays host_particles;
  cudaErrCheck(
      hipMalloc(&host_particles.id, sizeof(long long int) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.x_x, sizeof(double) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.x_y, sizeof(double) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.x_z, sizeof(double) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.v, sizeof(float3) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.a_hydro, sizeof(float3) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.h, sizeof(float) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.mass, sizeof(float) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.rho, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.entropy, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.entropy_dt, sizeof(float) * num_part_host));

  cudaErrCheck(
      hipMalloc(&host_particles.wcount, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.wcount_dh, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.rho_dh, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.rot_v, sizeof(float3) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.div_v, sizeof(float) * num_part_host));

  cudaErrCheck(
      hipMalloc(&host_particles.balsara, sizeof(float) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.f, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.P_over_rho2, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.soundspeed, sizeof(float) * num_part_host));
  cudaErrCheck(
      hipMalloc(&host_particles.v_sig, sizeof(float) * num_part_host));
  cudaErrCheck(hipMalloc(&host_particles.h_dt, sizeof(float) * num_part_host));

  cudaErrCheck(
      hipMalloc(&host_particles.time_bin, sizeof(timebin_t) * num_part_host));

  float host_kernel_coeffs[(kernel_degree + 1) * (kernel_ivals + 1)];
  for (int a = 0; a < (kernel_degree + 1) * (kernel_ivals + 1); a++) {
    host_kernel_coeffs[a] = kernel_coeffs[a];
  }
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
      cuda_kernel_coeffs), &host_kernel_coeffs,
      sizeof(float) * (kernel_degree + 1) * (kernel_ivals + 1)));
  integertime_t current = 8;
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(ti_current), &current, sizeof(integertime_t)));
  timebin_t current2 = 56;
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(max_active_bin), &current2, sizeof(timebin_t)));

  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_parts), &host_particles,
                                  sizeof(struct particle_arrays)));

  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_nr_parts), &num_part_host, sizeof(int)));

  /* Create the cells for the device. */
  struct cell_cuda *cell_host;
  cudaErrCheck(
      hipHostMalloc((void **)&cell_host, sizeof(struct cell_cuda) * 27));
  struct cell **host_pointers =
      (struct cell **)malloc(sizeof(struct cell *) * 27);
  for (int i = 0; i < 27; i++) {
    struct cell *c = cells[i];
    /*Create cells recursively. */
    create_cells(c, cell_host, host_pointers, parts);
  }
  /* Allocate space on the device for the cells. */
  struct cell_cuda *cell_device = NULL;
  struct cell *pointers_device = NULL;
  cudaErrCheck(
      hipMalloc((void **)&cell_device, sizeof(struct cell_cuda) * 27));
  cudaErrCheck(
      hipMalloc((void **)&pointers_device, sizeof(struct cell *) * 27));

  /* Copy the cells and pointers to the device and set up the symbol. */
  cudaErrCheck(hipMemcpy(cell_device, cell_host, sizeof(struct cell_cuda) * 27,
                          hipMemcpyHostToDevice));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(cells_cuda), &cell_device, sizeof(struct cell_cuda *)));

  cudaErrCheck(hipMemcpy(pointers_device, host_pointers,
                          sizeof(struct cell *) * 27, hipMemcpyHostToDevice));

  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(cpu_cells), &pointers_device, sizeof(struct cell **)));

  /* We copied the cells and cpu pointers to the GPU, setup the cells and create
   * the particle arrays. */
  /* Time to launch the kernel. */
  test_27_kernel << <1, 128>>> ();  // Single block.
  hipDeviceSynchronize();
  // Clean up
  cudaErrCheck(hipHostFree(cell_host));
  free(host_pointers);
}

__host__ void allocate_cells(void **parts, int particles, int cells) {
  if (hipHostMalloc((void **)parts, particles * particles * particles * cells *
                                         sizeof(struct part)) != hipSuccess) {
    error("couldn't allocate particles, no. of particles: %d",
          (int)particles * particles * particles);
  }
}

__host__ void allocate_cell(void **cell) {
  cudaErrCheck(hipHostMalloc(cell, sizeof(struct cell)));
}

__host__ void free_parts(void *parts) { cudaErrCheck(hipHostFree(parts)); }

__host__ void free_cell(void *cell) { cudaErrCheck(hipHostFree(cell)); }

#ifdef WITH_CUDA
#undef static
#undef restrict
#endif
