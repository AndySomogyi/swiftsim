#include "hip/hip_runtime.h"

#include "runner_cuda_main.h"
#include "queue_cuda.h"
#include "task_cuda.h"
#include "../kernel_hydro.h"
#include "../dimension.h"

/*Definition of particle structures (SoA on GPU) */
__device__ struct particle_arrays{
/* Device array containing the particle ID */
__device__ long long int *id;
/*Device array containing the particle positions. */
__device__ double *x_x, *x_y, *x_z;
/*Device array containing the particle predicted velocity */
__device__ float3 *v;
/*Device array containing the particle acceleration. */
__device__ float3 *a_hydro;
/* Device array contining the particle cutoff radius */
__device__ float *h;
/* Device array containing particle masses */
__device__ float *mass;
/* Device array containing particle densities. */
__device__ float *rho;
/* Device array containing the particle entropies. */
__device__ float *entropy;

/* Unclear if unions work on the GPU, so unrolling the union for now */
/* DENSITY */
/* Device array containing the number of neighbours. */
__device__ float *wcount;
/* Device array containing the number of neighbours spatial derivative */
__device__ float *wcount_dh;
/* Device array containing the derative of the density w.r.t. h */
__device__ float *rho_dh;
/* Device array containing the particle velocity curl. */
__device__ float3 *rot_v;
/* Device array containing the particle velocity divergence */
__device__ float *div_v;

/* FLOAT */
/* Device array containing the balsara switch */
__device__ float *balsara;
/* Device array containing the "Grad h" term */
__device__ float f;
/* Device array containing the pressure over density squared */
__device__ float *P_over_rho2;
/* Device array containing the particle sound speeds */
__device__ float *soundspeed;
/* Device array containing the signal velocities */
__device__ float *v_sig;
/* Device array containing the time derivative of the smoothing lengths */
__device__ float *h_dt;

/* Device array containing time step length */
__device__ timebin_t *time_bin;
} cuda_parts;

/* Cell array and array of CPU cell pointers.*/
__device__ struct cell_cuda *cells;
__device__ struct cell *cpu_cells;

/* Queue variables*/
__device__ struct queue_cuda cuda_queues[ cuda_numqueues ];
__device__ struct load_queue;
__device__ struct unload_queue;

/* Array of tasks on the device */
__device__ struct task_cuda *tasks;

/* Array of cuda tasks on the host */
struct task_cuda *tasks_host;

/* Array of unlocks on the device*/
__device__ int *cuda_unlocks;

/* Array of unlocks on the host. */
int *host_unlocks;

/* Simulation constants */
__device__ __constant__ integertime_t ti_current;
__device__ __constant__ double dim[3];

/* Density kernel function */
__constant__ cuda_kernel_coeffs[(kernel_degree+1)*(kernel_ivals+1)];

__device__ int cuda_cell_is_active( struct cell_cuda *c ) {
  return (c->ti_end_min == e->ti_current);
}

__device__ float cuda_pow_dimension_plus_one( float x ) {
#if defined(HYDRO_DIMENSION_3D)
  const float x2 = x*x;
  return x2*x2;
#elif defined(HYDRO_DIMENSION_2D)
  return x*x*x;
#elif defined(HYDRO_DIMENSION_3D)
  return x*x;
#else
  printf("The dimension is not defined!");
  return 0.f;
#endif
}


__device__ void load_cell(int cell_index)
{
  /* Get the pointer to the relevant cells. */
  struct cell *cpu_cell = &cpu_cells[cell_index];
  struct cell *cell = &cells[cell_index];
  struct part *parts = cpu_cell->parts;
  int i;
  /* Get the index to copy the data for the 0th particle in this cell to.*/
  int start = cell->first_part;

  for( i = threadIdx.x; i < cell->part_count; i+= blockDim.x ){
    struct part *current = &parts[i];
    /* Load the ID and position*/
    long long local_id = current->id;
    double local_x[3];
    local_x[0] = current->x[0];
    local_x[1] = current->x[1];
    local_x[2] = current->x[2];

    /* Unpack the ID and position and put into the device arrays. */
    cuda_parts.id[start+i] = local_id;
    cuda_parts.x_x[start+i] = local_x[0];
    cuda_parts.x_y[start+i] = local_x[1];
    cuda_parts.x_z[start+i] = local_x[2];

    /* Load the predicted velocity */ 
    float local_tempv[3];
    local_tempv[0] = current->v[0];
    local_tempv[1] = current->v[1];
    local_tempv[2] = current->v[2];

    /* Repack the predicted velocity */
    float3 local_v;
    local_v.x = local_tempv[0];
    local_v.y = local_tempv[1];
    local_v.z = local_tempv[2];

    /* Copy the predicted velocity to the particle array. */
    cuda_parts.v[start+i] = local_v;
   
    /* Load the acceleration */ 
    float local_tempa[3];
    local_tempa[0] = current->a_hydro[0];
    local_tempa[1] = current->a_hydro[1];
    local_tempa[2] = current->a_hydro[2];

    /* Repack the acceleration */
    float3 local_a;
    local_a.x = local_tempa[0];
    local_a.y = local_tempa[1];
    local_a.z = local_tempa[2];

    /* Copy the predicted velocity to the particle array. */
    cuda_parts.a_hydro[start+i] = local_a;

    /* Copy the cutoff, mass, density, entropy and entropy derivative*/
    float local_h, local_mass, local_rho, local_entropy, local_entropy_dt;
    local_h = current->h;
    local_mass = current->mass;
    local_rho = current->rho;
    local_entropy = current->entropy;
    local_entropy_dt = current->entropy_dt;

    cuda_parts.h[start+i] = local_h;
    cuda_parts.mass[start+i] = local_mass;
    cuda_parts.rho[start+i] = local_rho;
    cuda_parts.entropy[start+i] = local_entropy;
    cuda_parts.entropy_dt[start+i] = local_entropy_dt;

    /* Copy the density union to the device */
    float local_wcount, local_wcount_dh, local_rho_dh, local_div_v;
    float3 local_rot_v;
    local_wcount = current->density.wcount;
    local_wcount_dh = current->density.wcount_dh;
    local_rho_dh = current->density.rho_dh;
    local_rot_v.x = current->density.rot_v[0];
    local_rot_v.y = current->density.rot_v[1];
    local_rot_v.z = current->density.rot_v[2];
    local_div_v = current->density.div_v;

    cuda_parts.wcount[start+i] = local_wcount;
    cuda_parts.wcount_dh[start+i] = local_wcount_dh;
    cuda_parts.rho_dh[start+i] = local_rho_dh;
    cuda_parts.rot_v[start+i] = local_rot_v;
    cuda_parts.div_v = local_div_v;

    /* Copy the force union to the device. TODO Do we need this?*/
    float local_balsara, local_f, local_P_over_rho2, local_soundspeed, local_vsig, local_h_dt;

    local_balsara = current->force.balsara;
    local_f = current->force.f;
    local_P_over_rho2 = current->force.P_over_rho2;
    local_soundspeed = current->force.soundspeed;
    local_v_sig = current->force.v_sig;
    local_h_dt = current->force.h_dt;

    cuda_parts.balsara[start+i] = local_balsara;
    cuda_parts.f[start+i] = local_f;
    cuda_parts.P_over_rho2[start+i] = local_P_over_rho2;
    cuda_parts.soundspeed[start+i] = local_soundspeed;
    cuda_parts.v_sig[start+i] = local_v_sig;
    cuda_parts.h_dt[start+i] = local_h_dt;

    cuda_parts.time_bin[start+i] = current->time_bin;
  }
}

/* Task function to unload a specific cell. */
/* TODO: Note that this copies back certain data that I expect is not modified.*/
__device__ void unload_cell( int cell_index ) {

  /* Get the pointer to the relevant cells. */
  struct cell *cpu_cell = &cpu_cells[cell_index];
  struct cell *cell = &cells[cell_index];
  struct part *parts = cpu_cell->parts;
  int i;
  /* Get the index to copy the data for the 0th particle in this cell to.*/
  int start = cell->first_part;

  for(i = threadIdx.x; i < cell->part_count; i+= blockDim.x )
  { 
    struct part *current = &parts[i];

    /* Copy back the ID and position.*/
    current->id = cuda_parts.id[start+i]
    current->x[0] = cuda_parts.x_x[start+i];
    current->x[1] = cuda_parts.x_y[start+i];
    current->x[2] = cuda_parts.x_z[start+i];

    /* Copy back the velocity*/
    float3 local_v = cuda_parts.v[start+i];
    current->v[0] = local_v.x;
    current->v[1] = local_v.y;
    current->v[2] = local_v.z;

    /* Copy back the acceleration */
    float3 local_a_hydro = cuda_parts.a_hydro[start+i];
    current->a_hydro[0] = local_a_hydro.x;
    current->a_hydro[1] = local_a_hydro.y;
    current->a_hydro[2] = local_a_hydro.z;

    /* Copy back the cutoff, mass, density, entropy and entropy_dt*/
    current->h = cuda_parts.h[start+i];
    current->mass = cuda_parts.mass[start+i];
    current->rho = cuda_parts.rho[start+i];
    current->entropy = cuda_parts.entropy[start+i];
    current->entropy_dt = cuda_parts.entropy_dt[start+i];

    /* Copy back the force union.*/
    current->force.balsara = cuda_parts.balsara[start+i];
    current->force.f = cuda_parts.f[start+i];
    current->force.P_over_rho2 = cuda_parts.P_over_rho2[start+i];
    current->force.soundspeed = cuda_parts.soundspeed[start+i];
    current->force.v_sig = cuda_parts.v_sig[start+i];
    current->force.h_dt = cuda_parts.h_dt[start+i];

    /* Copy back the timebin. */
    current->time_bin = cuda_parts.time_bin[start+i];
  }

}

/* Task function to execute a density task. Uses naive n^2 algorithm without symmetry*/
/* To do density between Cell i and cell j this needs to be called twice. */
__device__ void dopair_density( struct cell_cuda *ci, struct cell_cuda *cj ) {
 
 /* Are these cells active? */
  if( !cell_is_active(ci) && !cell_is_active(cj) ) return;

  const int count_i = ci->count;
  const int count_j = cj->count;
  int part_i = ci->first_part;
  int part_j = cj->first_part;
  float rho, rho_dh, div_v, wcount, wcount_dh;
  float3 rot_v;
  double shift[3] = {0.0, 0.0, 0.0};

  /* Deal with periodicity concerns. */
  for(int k = 0; k < 3; k++ ) {
    if( cj->loc[k] - ci->loc[k] < dim[k] / 2)
      shift[k] = dim[k];
    else if( cj->loc[k] - ci->loc[k] > dim[k] / 2 )
      shift[k] = dim[k]
  }

  /* Loop over the parts in cell ci */
  for( int pid = part_i + threadidx.x; pid < part_i+count_i; pid += blockDim.x ){

    const float hi = cuda_parts.h[pid];
    
    double pix[3];
    pix[0] = cuda_parts.x_x[pid] - shift[0];
    pix[1] = cuda_parts.x_y[pid] - shift[1];
    pix[2] = cuda_parts.x_z[pid] - shift[2];
    const float hig2 = hi * hi * kernel_gamma2;

    /* Reset local values. */
    rho = 0.0f; rho_dh = 0.0f; div_v = 0.0f; wcount = 0.0f; wcount_dh = 0.0f;
    rot_v.x = 0.0f; rot_v.y = 0.0f; rot_v;z = 0.0f;


    /* Loop over the parts in cj. */
    /* TODO May be possible to optimize this loop ordering.*/
    for( int pjd = part_j; pjd < part_j + count_j; pjd++ ) {

      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];

      /* If in range then interact. */
      if( r2 < hig2 ) {
        float wi, wi_dx;
        float dv[3], curlvr[3];

        /* Load mass on particle pj. */
        const float mj = cuda_parts.mass[pjd];

        /* Get r and 1/r */
        const float r = sqrtf(r2);
        const float ri = 1.0f / r;

        /* Compute the kernel function */
        const float hi_inv = 1.0f / hi;
        const float ui = r * hi_inv;
        
        cuda_kernel_deval(ui, &w, &dw_dx);

        /* Compute contribution to the density. */
        rho += mj * w;
        rho_dh -= mj * (hydro_dimension * w + ui * dw_dx);

        /* Compute contribution to the number of neighbours */
        wcount += w;
        wcount_dh -= ui * dw_dx

        const float fac = mj * dw_dx * ri;

        /* Compute dv dot r */
        float3 piv, pjv;
        piv = cuda_parts.v[pid];
        pjv = cuda_parts.v[pjd];
        dv[0] = piv.x - pjv.x;
        dv[1] = piv.y - pjv.y;
        dv[2] = piv.z - pjv.z;
        const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

        div_v -= fac*dvdr;

        curlvr[0] = dx[1] * dx[2] - dv[2] * dx[1];
        curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
        curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];

        rot_v.x += fac*curlvr[0];
        rot_v.y += fac*curlvr[1];
        rot_v.z += fac*curlvr[2];
        
      } 
    }//Loop over cj.
    /* Write data for particle pid back to global stores. */
    atomicAdd( &cuda_parts.rho[pid], rho);
    atomicAdd( &cuda_parts.rho_dh[pid], rho_dh);
    atomicAdd( &cuda_parts.wcount[pid], wcount);
    atomicAdd( &cuda_parts.wcount_dh[pid], wcount_dh);
    atomicAdd( &cuda_parts.div_v[pid], div_v);
    atomic_add( &cuda_parts.rot_v[pid].x, rot_v.x);
    atomic_add( &cuda_parts.rot_v[pid].y, rot_v.y);
    atomic_add( &cuda_parts.rot_v[pid].z, rot_v.z);

  }//Loop over ci.
}

__device__ __inline__ void cuda_kernel_deval( float u, float *restrict W, float *restrict dW_dx) {
  /* Go to the range [0,1] from [0,H] */
  const float x = u * kernel_gamma_inv;

  /* Pick the correct branch of the kernel */
  const int temp = (int)(x*kernel_ivals_f);
  const int ind = temp > kernel_ivals ? kernel_ivals : temp;
  const float *coeffs = &kernel_coeffs[ind*(kernel_degree+1)];

  /* First two terms of the polynomial*/
  float w = coeffs[0] * x + coeffs[1];
  float dw_dx = coeffs[0];

  /* and the rest of them*/
  for(int k = 2; k <= kernel_degree; k++)
  {
    dw_dx = dw_dx * x + w;
    w = x * w + coeffs[k];
  }

  /* Return the results */
  *W = w * kernel_constant * kernel_gamma_inv_dim;
  *dW_dx = dw_dx * kernel_constant * kernel_gamma_inv_dim_plus_one;
}

/* Task function to execute a force task. Uses naive n^2 algorithm without symmetry*/
/* To do force between Cell i and cell j this needs to be called twice. */
__device__ void dopair_force( struct cell_cuda *ci, struct cell_cuda *cj) {

 /* Are these cells active? */
  if( !cell_is_active(ci) && !cell_is_active(cj) ) return;

  const int count_i = ci->count;
  const int count_j = cj->count;
  int part_i = ci->first_part;
  int part_j = cj->first_part;

  float3 a_hydro;
  float h_dt, v_sig_stor, entropy_dt;

  /* Deal with periodicity concerns. */
  for(int k = 0; k < 3; k++ ) {
    if( cj->loc[k] - ci->loc[k] < dim[k] / 2)
      shift[k] = dim[k];
    else if( cj->loc[k] - ci->loc[k] > dim[k] / 2 )
      shift[k] = dim[k]
  }

  /* Loop over the parts in cell ci */
  for( int pid = part_i + threadidx.x; pid < part_i+count_i; pid += blockDim.x ){

    const float hi = cuda_parts.h[pid];

    /* Reset the values. */
    a_hydro.x = 0.0f; a_hydro.y = 0.0f; a_hydro.z = 0.0f;
    h_dt = 0.0f; v_sig_stor = cuda_parts.v_sig[pid]; entropy_dt = 0.0f;

    double pix[3];
    pix[0] = cuda_parts.x_x[pid] - shift[0];
    pix[1] = cuda_parts.x_y[pid] - shift[1];
    pix[2] = cuda_parts.x_z[pid] - shift[2];

    const float hig2 = hi * hi * kernel_gamma2;

    /* Loop over the particles in cj. */
    for( int pjd = part_j; pjd < part_j + count_j; pjd++ ) {
      
      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];

      const float hj = cuda_parts.h[pjd];
      if(r2 < hig2 || r2 < hj*hj*kernel_gamma2) {
        float wi, wj, wi_dx, wj_dx;
        const float fac_mu = 1.f; 

        const float r = sqrtf(r2);
        const float r_inv = 1.0f / r;

        /* Load some data.*/
        const float mj = cuda_parts.mass[pjd];
        const float rhoi = cuda_parts.rho[pid];
        const float rhoj = cuda_parts.rho[pjd];

        /* Get the kernel for hi. */
        const float hi_inv = 1.0f/hi;
        const float hid_inv = cuda_pow_dimension_plus_one(hi_inv);
        const float ui = r * hi_inv;
        cuda_kernel_deval(ui, &wi, &wi_dx)
        const float wi_dr = hid_inv * wi_dx;

        /* Get the kernel for hj. */
        const float hj_inv = 1.0f/hj;
        const float hjd_inv = cuda_pow_dimension_plus_one(hj_inv);
        const float xj = r * hj_inv;
        cuda_kernel_deval(xj, &wj, &wj_dx);
        const float wj_dr = hjd_inv * wj_dx;

        /* Compute h-gradient terms */
        const float f_i = cuda_parts.f[pid];
        const float f_j = cuda_parts.f[pjd];

        /* Compute pressure terms */
        const float P_over_rho2_i = cuda_parts.P_over_rho2[pid];
        const float P_over_rho2_j = cuda_parts.P_over_rho2[pjd];

        /* Compute sound speeds*/
        const float ci = cuda_parts.soundspeed[pid];
        const float cj = cuda_parts.soundspeed[pjd];

        /* Compute dv dot r. */
        const float dvdr = (cuda_parts.v[pid].x - cuda_parts.v[pjd].x) * dx[0] + 
                            cuda_parts.v[pid].y - cuda_parts.v[pjd].y) * dx[1] + 
                            cuda_parts.v[pid].z - cuda_parts.v[pjd].z) * dx[2];

        /* Balsara term */
        const float balsara_i = cuda_parts.balsara[pid];
        const float balsara_j = cuda_parts.balsara[pjd];

        /* Are the particles moving towards each other? */
        const float omega_ij = (dvdr < 0.f) ? dvdr : 0.f;
        const float mu_ij = fac_mu * r_inv * omega_id;

        /* Signal velocity */
        const float v_sig = ci + cj - 3.f * mu_ij;

        /* Now construct the full viscosity term */
        const float rho_ij = 0.5f* (rhoi+rhoj);
        const float visc = -0.25 * const_viscosity_alpha * v_sig * mu_id * 
                           (balsara_i + balsara_j) / rho_ij;

        /* Now convolce with the kernel */
        const float visc_term = 0.5f* visc * (wi_dr + wj_dr) * r_inv;
        const float sph_term = 
          (f_i * P_over_rho2_i * wi_dr + f_j * P_over_rho2_j * wj_dr) * r_inv;

        /* Compute the acceleration */
        const float acc = visc_term + sph_term;

        /* Compute the force */
        a_hydro.x -= mj * acc * dx[0];
        a_hydro.y -= mj * acc * dx[1];
        a_hydro.z -= mj * acc * dx[2];

        /* Get the time derivative for h. */
        h_dt -= mj * dvdr * r_inv / rhoj * wi_dr;

        /* Update the signal velocity. */
        v_sig_stor = (v_sig_stor > v_sig ) ? v_sig_stor : v_sig;

        /* Change in entropy */
        entropy_dt += mj * visc_term * dvdr;
        //INTERACT
      }

    } //Loop over cell cj.

    /* Flush to global stores.*/
    atomicAdd(&cuda_parts.a_hydro[pid].x, a_hydro.x);
    atomicAdd(&cuda_parts.a_hydro[pid].y, a_hydro.y);
    atomicAdd(&cuda_parts.a_hydro[pid].z, a_hydro.z);
    atomicAdd(&cuda_parts.h_dt[pid], h_dt);
    atomicAdd(&cuda_parts.entropy_dt[pid], entropy_dt);

    /* Update the signal velocity */
    float global_vsig = cuda_parts.v_sig[pid];
    

  }//Loop over cell ci.
}

/* Runner function to retrieve a task index from a queue. */
__device__ int runner_cuda_gettask( struct queue_cuda *q ) {

  int tid = -1;
  if( atomicAdd((int*)&q->nr_avail_tasks, -1) <= 0)
  {
    atomicAdd((int*)&q->nr_avail_tasks, 1);
    return -1;
  }

  /* Main loop */
  while( ( tid = cuda_queue_gettask( q ) ) >= 0 ){
    //TODO Do we need to lock anything here? Probably no.
    //TODO Does this need to be a while?
    break;
  }
  if( tid >= 0 ) {
    q->rec_data[atomicAdd( (int *)&q->rec_count, 1 ) ] = tid; //TODO Are we keeping rec_data
  }

  return tid;
}


/* The main kernel. */
__global__ void swift_device_kernel( )
{
  __shared__ volatile int tid;
  __shared__ volatile int done;
  int *src, *dest;
  int i;

  /* Main loop */
  while( 1 ){
    __syncthreads();
    /* Get a task from the queue. */
    if(threadIdx.x == 0){
      tid = -1;
      /* Highest Priority queue, unload tasks. */
      if(unload_queue.nr_avail_tasks > 0 ) {
        tid = runner_cuda_gettask( &unload_queue);
      }

      /* Next highest priority queue, load tasks. Only some blocks look in here */
      if(tid < 0 && load_queue.nr_avail_tasks > 0 && blockIdx.x < cuda_numloaders ) {
        tid = runner_cuda_gettask( &load_queue );
      }

      /* Finally loop through work queues in priority order. queue 0 is highest priority*/
      for( i = 0; i < cuda_numqueues && tid < 0; i++ ) {
        if( cuda_queues[i].nr_avail_tasks > 0 ){
          tid = runner_cuda_gettask( &cuda_queues[i] );
        }
      }

    }//Get task from queue

    /* Threads need to wait until they have work to do */
    __syncthreads();

    /* If the tasks are all complete and we don't have anything to do exit*/
    if(tid < 0 && tot_num_tasks == 0)
      break;

    if( tid < 0 )
      continue;

    //TODO Do work here!


  } // End of main loop

}

/* Host function to check cuda functions don't return errors */
__host__ inline void cudaErrCheck( hipError_t status)
{
  if(status != hipSuccess)
  {
    error(hipGetErrorString(status);
  }
}

/* Host function to give all the cells the IDs required for CUDA order and tasks. */
__host__ void cell_IDs(struct cell *c, int *k){
  int i;
  c->cudaID = *k;
  *k = *k + 1;
  if(c->split){
    for(i = 0; i < 8; i++)
    {
      if(c->progeny[i] != NULL){
        cell_IDs(&c->progeny[i], k);
      }
    }
  }

}

/* Host function to create the load, unload and implicit tasks */
__host__ void create_transfer_tasks(struct cell *c, int *k, int parent_load_task, int parent_unload_task)
{
  tasks_host[*k].unlocks = malloc(sizeof(int) * 9); //For now we create CPU storage for these unlocks for each task. No task should have more than 9 unlocks to other transfer tasks.
  tasks_host[*k].nr_unlock_tasks = 0;
  tasks_host[*k].size_unlocks = 9;
  if(c->split){

    /* If the task is split we create implicit tasks to deal with dependencies. */
    tasks_host[*k].type = type_implicit_load;
    tasks_host[*k].ci = c->cuda_ID;
    tasks_host[*k].cj = -1; //These tasks operate on a single cell.
    tasks_host[*k].weight = c->count;
    tasks_host[*k].wait = 0;
    tasks_host[*k].subtype = task_subtype_none;
    tasks_host[*k].skip = 0;
    tasks_host[*k].implicit = 0;
    tasks_host[*k].task = NULL;
    /* The load implicit tasks unlocks the parent's task */
    if(parent_load_task >= 0){
      tasks_host[*k].unlocks[tasks_host[*k].nr_unlock_tasks++] = parent_load_task;
    }
    c->load_task = *k;
    *k = *k + 1;

    /* Create the implicit unload task. */
    tasks_host[*k].unlocks = malloc(sizeof(int) * 9); //For now we create CPU storage for these unlocks for each task. No task should have more than 9 unlocks to other transfer tasks.
    tasks_host[*k].nr_unlock_tasks = 0;
    tasks_host[*k].size_unlocks = 9;
    tasks_host[*k].type = type_implicit_unload;
    tasks_host[*k].ci = c->cuda_ID;
    tasks_host[*k].cj = -1; //These tasks operate on a single cell.
    tasks_host[*k].weight = c->count;
    tasks_host[*k].wait = 0;
    tasks_host[*k].subtype = task_subtype_none;
    tasks_host[*k].skip = 0;
    tasks_host[*k].implicit = 0;
    tasks_host[*k].task = NULL;
    
    /* The unload implicit task is unlocked by the parent task */
    if( parent_unload_task >= 0 ){
      tasks_host[parent_unload_task].unlocks[tasks_host[parent_unload_task].nr_unlock_tasks++] = *k;
    }
    c->unload_task = *k;
    *k = *k + 1;

    /* Recurse down the tree. */
    int load = *k-2;
    int unload = *k-1;
    for(int i = 0; i < 8; i++ )
    {
      if(c->progeny[i] != NULL)
        create_transfer_tasks( &c->progeny[i] , k, load, unload);
    }

  }else{
    /* Create the load task*/
    tasks_host[*k].type = type_load;
    tasks_host[*k].ci = c->cuda_ID;
    tasks_host[*k].cj = -1; //These tasks operate on a single cell.
    tasks_host[*k].weight = c->count;
    tasks_host[*k].wait = 0;
    tasks_host[*k].subtype = task_subtype_none;
    tasks_host[*k].skip = 0;
    tasks_host[*k].implicit = 0;
    tasks_host[*k].task = NULL;
    /* This load task unlocks the parent's task. */
    if(parent_load_task >= 0){
      tasks_host[*k].unlocks[tasks_host[*k].nr_unlock_tasks++] = parent_load_task;
    }
    c->load_task = *k;
    *k = *k + 1;

    /* Create the unload task */
    tasks_host[*k].unlocks = NULL; //unload tasks never unlock anything, end of tree.
    tasks_host[*k].nr_unlock_tasks = 0;
    tasks_host[*k].type = type_unload;
    tasks_host[*k].ci = c->cuda_ID;
    tasks_host[*k].cj = -1; //These tasks operate on a single cell.
    tasks_host[*k].weight = c->count;
    tasks_host[*k].wait = 0;
    tasks_host[*k].subtype = task_subtype_none;
    tasks_host[*k].skip = 0;
    tasks_host[*k].implicit = 0;
    tasks_host[*k].task = NULL;
    /* The unload task is unlocked by the parent task */
    if(parent_unload_task >= 0){
      tasks_host[parent_unload_task].unlocks[tasks_host[parent_unload_task].nr_unlock_tasks++] =  *k;
    }
    c->unload_task = *k;
    *k = *k + 1;
  }
}

/* Recursive function to create the cell structures require for the GPU.*/
__host__ void create_cells( struct cell *c, struct cell_cuda *cell_host, struct cell **host_pointers, struct part *parts){

  /* Set the host pointer. */
  host_ponters[c->cuda_ID] = c;
  struct cell_cuda *c2 = &cell_host[c->cuda_ID];

  c2->loc[0] = c->loc[0];
  c2->loc[1] = c->loc[1];
  c2->loc[2] = c->loc[2];
  c2->width[0] = c->width[0];
  c2->width[1] = c->width[1];
  c2->width[2] = c->width[2];
  c2->h_max = c->h_max;
  c2->first_part = c->first_part - parts;
  c2->part_count = c->part_count;
  c2->parent = c->parent->cuda_ID;
  c2->super = c->super->cuda_ID;
  c2->ti_end_min = c->ti_end_min;
  c2->ti_end_max = c->ti_end_max;
  c2->dmin = c->dmin;

  for(int i = 0; i < 8; i++)
  {
    /* Set progeny and recurse. */
    c2->progeny[i] = c->progeny[i]->cuda_ID;
    create_cells(c->progeny[i], cell_host, host_pointers, parts);
  }

}

/* Host function to perform quickselect */
__host__ int select(int* list, int left, int right, int n){
    static int value = -1;
    if(value < 0)
        value = right;
    if(left == right)
        return list[left];
    int pivotIndex = left + floor(r2() * (right-left+1));
    pivotIndex = partition(list, left, right, pivotIndex);
    if(n == pivotIndex)
        return list[n];
    else if ( n < pivotIndex){
        return select(list, left, pivotIndex-1, n);
    }else{
        return select(list, pivotIndex+1, right, n );
    }
}

/* Host function used for priority cutoff */
#define PERCENTILE 0.8
__host__ int find_priority_cutoff( struct task_cuda *tasks, int count ){

  int nr_work_tasks = 0;
  int *costs = malloc( sizeof(int) * count);
  for(i = 0; i < count; i++){
    if(tasks[i].type >= type_load){
      costs[nr_work_tasks++] = tasks[i].weight;
    }
  }
  int result = select(costs, 0, nr_work_tasks-1, (int) ((float)( nr_work_tasks-1 ) * 0.8f  ) );
  free(costs);
  return result;
}

/* Host function to create the GPU tasks. Should be called whenever the tasks are recreated */
__host__ void create_tasks(struct engine *e){

  struct scheduler *sched = &e->sched;
  struct space *s = &e->s;
  int num_gpu_tasks = 0;
  int i, k;
  struct cell *c;
  static int firstrun = 0;

  /* We only create density, ghost and force tasks on the device at current. */
  for(i = 0; i < sched->nr_tasks; i++)
  {
    if(sched->tasks[i].subtype == task_subtype_density || sched->tasks[i].subtype == task_subtype_force ||
        sched->tasks[i].type == task_type_ghost )
        num_gpu_tasks++;
  }

  /* We also create a load and unload task for every cell in the system */
  num_gpu_tasks += s->tot_cells * 2;

  /* Allocate page-locked memory for the host version of the GPU tasks. */
  cudaErrCheck( hipHostMalloc( (void**) &tasks_host, num_gpu_tasks * sizeof(struct task_cuda ) ) );

  k = 0;
  /* Loop through the cells and give them all an ID. */
  for(i = 0; i < cdim[0]*cdim[1]*cdim[2]; i++)
  {
    c = &s->cells_top[i];
    cell_IDs(c, &k);
  } 

  k = 0
  /* Create the tasks. */
  for(i = 0; i < sched->nr_tasks; i++){

    if(sched->tasks[i].subtype == task_subtype_density || sched->tasks[i].subtype == task_subtype_force ||
        sched->tasks[i].type == task_type_ghost ){
      /* Copy the data to the CUDA task. */
      struct task *t = &sched->tasks[i];
      tasks_host[k].flags = t->flags;
      tasks_host[k].rank = t->rank;
      tasks_host[k].weight = t->weight;
      tasks_host[k].nr_unlock_tasks = (int) t->nr_unlock_tasks;
      tasks_host[k].type = t->type;
      tasks_host[k].subtype = t->subtype;
      tasks_host[k].skip = t->skip;
      tasks_host[k].implicit = t->implicit;
      tasks_host[*k].size_unlocks = 0;

      tasks_host[k].ci = t->ci->cuda_ID;
      tasks_host[k].cj = t->cj->cuda_ID;

      /* We have a double linked structure because its easier to create. */
      tasks_host[k].task = t;
      t->cuda_task = k;
      k++;
    }
  }

  /* Create the data transfer tasks. */
  for(i = 0; i < cdim[0]*cdim[1]*cdim[2]; i++)
  {
    c = &s->cells_top[i];
    create_transfer_tasks( c , k, -1, -1);
  } 

  /* Check we got this right initially.. */
  if(k != num_gpu_tasks){
    error("We created a different number of GPU tasks than expected");
  }

  /* Now we have the tasks, time to start working on the dependencies. */
  
  /* Loop through the tasks */
  for(i = 0; i < num_gpu_tasks; i++)
  {
    /* The transfer tasks dependencies are done anyway so skip them. */
    if(tasks_host[i].type == type_load || tasks_host[i].type == type_unload || 
       tasks_host[i].type == type_implicit_load || tasks_host[i].type == type_implicit_unload )
        continue;

    /* Get the task. */
    struct task_cuda t = &tasks_host[i];

    /* How many dependencies did the CPU task have. */
    int deps = t->task->nr_unlock_tasks;

    /* If it is a force task then it also unlocks the unload tasks. */ 
    if(t->subtype == task_subtype_force){
      deps++;
      /* If its a pair force task then it needs 2 unlocks. */
      if( t->type == task_type_pair ){
        deps++;
      }
    }

    /* Allocate some CPU memory for the unlocks. */
    t->unlocks = malloc( sizeof(int) * deps );
    t->size_unlocks = deps;

    /* Copy the dependencies */
    for(int j = 0; j < t->task->nr_unlock_tasks; j++){
      t->unlocks[t->nr_unlock_tasks++] = t->task->unlocks[j]->cuda_task;
    }

    /* If it is a force task then add the unload tasks.*/
    if(t->subtype == task_subtype_force){
      t->unlocks[t->nr_unlock_tasks++] = t->task->ci->unload_task;
      if(t->type == task_type_pair ){
        t->unlocks[t->nr_unlock_tasks++] = t->task->cj->unload_task;
      }
    }

    /* If it is a density task then it is unlocked by the load task. */
    if(t->subtype == task_subtype_density){
      /* We may need to stretch the load task's unlocks */
      if(tasks_host[t->ci->load_task].nr_unlock_tasks == 
        tasks_host[t->ci->load_task].size_unlocks){
      
        int *temp = malloc(sizeof(int) * tasks_host[t->ci->lock_task].size_unlocks * 2);
        memcpy(temp, tasks_host[t->ci->load_task].unlocks,
           sizeof(int) * tasks_host[t->ci->load_task].nr_unlock_tasks );
        free(tasks_host[t->ci->load_task].unlocks);
        tasks_host[t->ci->load_task].unlocks = temp;

      }
      tasks_host[t->ci->load_task].unlock[tasks_host[t->ci->load_task].nr_unlock_tasks++] = i;

      if( t->type == task_type_pair ){
        /* We may need to stretch the load task's unlocks */
        if(tasks_host[t->cj->load_task].nr_unlock_tasks == 
          tasks_host[t->cj->load_task].size_unlocks){
        
          int *temp = malloc(sizeof(int) * tasks_host[t->cj->lock_task].size_unlocks * 2);
          memcpy(temp, tasks_host[t->cj->load_task].unlocks,
             sizeof(int) * tasks_host[t->cj->load_task].nr_unlock_tasks );
          free(tasks_host[t->cj->load_task].unlocks);
          tasks_host[t->cj->load_task].unlocks = temp;
  
        }
        tasks_host[t->cj->load_task].unlock[tasks_host[t->cj->load_task].nr_unlock_tasks++] = i;

      }//If is pair task.

    }//Load to density task dependencies.

  } //Loop over the tasks.


  /* Now we have the dependencies we need to squash them into a single array. */

  /* First we count how many there are.*/
  int num_deps = 0;
  for(i = 0; i < num_gpu_tasks; i++){
    num_deps += tasks_host[i].nr_unlock_tasks;
  }

  /* Create a storage location for the dependency array. */
  int *host_dependencies = malloc(sizeof(int) * num_deps);
  int deps_filled = 0;

  /* Add the arrays, update the pointers, remove the small arrays. */ 
  for(i = 0; i < num_gpu_tasks; i++){
    memcpy(&host_dependencies[deps_filled], tasks_host[i].unlocks, tasks_host[i].nr_unlock_tasks);
    free(tasks_host[i].unlocks);
    tasks_host[i].unlocks = &host_dependencies[deps_filled];
    deps_filled += tasks_host[i].nr_unlock_tasks;
  }

  /* Allocate storage for the dependencies on the GPU.*/
  int *gpu_dependencies = NULL;
  if(firstrun){
    /* If we already have an array for this we need to remove it. */
    cudaErrCheck(hipMemcpyFromSymbol( gpu_dependencies, HIP_SYMBOL(&cuda_unlocks), sizeof(int *) ) );
    hipFree(gpu_dependencies);
    gpu_dependencies = NULL;
  }
  cudaErrCheck( hipMalloc((void**) &gpu_dependencies, sizeof(int) * num_deps ));
  
  /* Start copying the dependency array to the device */ 
  cudaErrCheck( hipMemcpyAsync(gpu_dependencies, host_dependencies, sizeof(int) * num_deps,
       hipMemcpyHostToDevice ) );

  /* We need the task's unlock pointers to point at the device stuff, which we do with pointer maths */
  for(i = 0; i < num_gpu_tasks; i++){
    int *temp_p = gpu_dependencies + (tasks_host[i].unlocks - host_dependencies);
    tasks_host[i].unlocks = temp_p;
  }

  /* Wait for the transfer to complete.*/
  cudaErrCheck(hipDeviceSynchronize());

  /* Copy the new device array to where it will be visible. */
  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL( cuda_unlocks), gpu_dependencies, sizeof(int *) ) );
  
  /* Copy the tasks to the device. */
  struct task_cuda *gpu_tasks= NULL;
  if(firstrun){
    cudaErrCheck( hipMemcpyFromSymbol(gpu_tasks, HIP_SYMBOL(&tasks), sizeof(struct task_cuda *) ) );
    hipFree(gpu_tasks);
    gpu_tasks = NULL;
  }
  cudaErrCheck( hipMalloc((void**) &gpu_tasks, sizeof(struct task_cuda) * num_gpu_tasks) );
  
  cudaErrCheck( hipMemcpy(gpu_tasks, tasks_host, sizeof(struct task_cuda) * num_gpu_tasks,
        hipMemcpyHostToDevice ) );

  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL( tasks), gpu_tasks, sizeof(struct task_cuda *) ) );


  /* Create the cuda_cells on the CPU. */
  struct cell_cuda *cell_host = malloc(sizeof(struct cell_cuda) * s->tot_cells); 
  struct cell **host_pointers = malloc(sizeof(struct *cell) * s->tot_cells);
  k = 0;
  for(int i = 0; i < nr_cells; i++)
  {
    c = &s->cells_top[i];
    /*Create cells recursively. */
    create_cells(c, cell_host, host_pointers, s->parts);
  }

  /* Allocate space on the device for the cells. */
  struct cell_cuda *cell_device = NULL;
  struct cell **pointers_device = NULL;
  if(firstrun){
    /* If we already have an array for this we need to remove it. */
    cudaErrCheck(hipMemcpyFromSymbol( cell_device, HIP_SYMBOL(&cells), sizeof(struct cell_cuda *) ) );
    hipFree(cell_device);
    cudaErrCheck(hipMemcpyFromSymbol( pointers_device, HIP_SYMBOL(&cpu_cells), sizeof(struct cell **) ) );
    hipFree(pointers_device);
    cell_device = NULL;
    pointers_device = NULL;
  }
  cudaErrCheck( hipMalloc((void**) &cell_device, sizeof(struct cell_cuda ) * s->tot_cells ) );
  cudaErrCheck( hipMalloc((void**)) &pointers_device, sizof(struct cell *) * s->tot_cells ) );
 
  /* Copy the cells and pointers to the device and set up the symbol. */
  cudaErrCheck( hipMemcpy(cell_device, cell_host, sizeof(struct cell_cuda) * s->tot_cells,
        hipMemcpyHostToDevice ) );

  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL( cells), cell_device, sizeof(struct cell_cuda *) ) );

  cudaErrCheck( hipMemcpy( pointers_device, host_pointers, sizeof(struct cell *) * s->tot_cells, 
       hipMemcpyHostToDevice );

  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL( cpu_cells), pointers_device, sizeof(struct cell **) ) );

  /* Setup the queues. */
  /* We have 4 queues, one containing unload & implicit tasks. */
  /* One containing load tasks only. */
  /* One containing high priority work tasks.*/
  /* Last one containing all other tasks. */
  struct queue_cuda load_host;
  struct queue_cuda unload_host;
  struct queue_cuda work_host[cuda_numqueues];
  int nr_load=0, nr_unload=0, nr_high=0, nr_low=0;

  /* Compute the 80th percentile for the work priorities.*/
  int cut = priority_cutoff( tasks_host,  num_gpu_tasks );

  /* cuda_queue_size is lazily set to fix all of the tasks in for now. If this becomes an issue
     it can be reduced */
  int qsize = max( num_gpu_tasks, 256 );
  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL( cuda_queue_size), &qsize, sizeof(int) ) );

  /* TODO Set the waits! */
  for(i = 0; i < num_gpu_tasks; i++ ) {
    tasks_host[i].wait = 0;
  }
  for(i = 0; i < num_gpu_tasks; i++) {
    struct task_cuda *temp_t = &tasks_host[i];
    for(int ii = 0; ii < temp_t->nr_unlock_tasks; ii++)
    {
      tasks_host[ temp_t->unlocks[ii]].wait++;
    }
  }

  /* Create the queues */

  /* Create the buffers used to initialised the data and rec_data arrays. */
  int *data, data2;

  if( ( data = (int*) malloc(sizeof(int) * qsize ) ) == NULL )
    error("Failed to allocate the data buffer on the host.");
  if( ( data2 = (int*) malloc(sizeof(int) * qsize ) ) == NULL )
    error("Failed to allocate the rec_data buffer on the host.");

  load_host.count = 0;
  /* Find the load tasks */
  for( i = 0; i < num_gpu_tasks; i++ ) {
    if( tasks_host[i].type == type_load ) {
      if( tasks_host[i].wait == 0) {
        data[load_host.count] = i;
        data2[load_host.count++] = -1;
      }
      nr_load++;
    }
  }

  for(i = load_host.count; i < qsize; i++){
    data[i] = -1;
    data2[i] = -1;
  }

  /* Allocate and copy the data to the device. */
  cudaErrCheck( hipMalloc( &load_host.data, sizeof(int) * qsize ) );
  cudaErrCheck( hipMemcpy( (void*) load_host.data, data, sizeof(int) * qsize, 
       hipMemcpyHostToDevice ) );
  cudaErrCheck( hipMalloc( &load_host.rec_data, sizeof(int) * qsize ) );
  cudaErrCheck( hipMemcpy( (void*) load_host.rec_data, data2, sizeof(int) * qsize, 
       hipMemcpyHostToDevice ) );
  load_host.first = 0;
  load_host.last = load_host.count;
  load_host.rec_count = 0;
  load_host.nr_avail_tasks = load_host.count;
  load_host.count = nr_load;

  /* Copy the queue to the device */
  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL( load_queue), &load_host , sizeof(struct queue_cuda) ) );
 
 /* Create the unload queue. */
  unload_host.count = 0;
  for( i = 0; i < num_gpu_tasks; i++ ){
    if( tasks_host[i].type <= type_unload && tasks_host[i].type >= type_implicit_unload ){
      if( tasks_host[i].wait == 0 ) {
        data[unload_host.count] = i;
        data2[unload_host.count++] = -1;
      }
      nr_unload++;
    }
  }
  for(i = unload_host.count; i < qsize; i++){
    data[i] = -1;
    data2[i] = -1;
  }

  /* Allocate and copy the data to the device. */
  cudaErrCheck( hipMalloc( &unload_host.data, sizeof(int) * qsize ) );
  cudaErrCheck( hipMemcpy( (void*) unload_host.data, data, sizeof(int) * qsize, 
       hipMemcpyHostToDevice ) );
  cudaErrCheck( hipMalloc( &unload_host.rec_data, sizeof(int) * qsize ) );
  cudaErrCheck( hipMemcpy( (void*) unload_host.rec_data, data2, sizeof(int) * qsize, 
       hipMemcpyHostToDevice ) );
  unload_host.first = 0;
  unload_host.last = unload_host.count;
  unload_host.rec_count = 0;
  unload_host.nr_avail_tasks = unload_host.count;
  unload_host.count = nr_unload;

  /* Copy the queue to the device */
  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL( unload_queue), &unload_host , sizeof(struct queue_cuda) ) );


  /* Create the high priority queue. */

  work_host[0].count = 0;
  for( i = 0; i < num_gpu_tasks; i++ )
  {
    if(tasks_host[i].type > type_load && tasks_host[i].weight >= cut){ 
      if(tasks_host[i].wait == 0) {
        data[work_host[0].count] = i;
        data2[work_host[0].count++] = -1;
      }
      nr_high++;
    }
  }
  for(i = work_host[0].count; i < qsize; i++){
    data[i] = -1;
    data2[i] = -1;
  }

  /* Allocate and copy the data to the device. */
  cudaErrCheck( hipMalloc( &work_host[0].data, sizeof(int) * qsize ) );
  cudaErrCheck( hipMemcpy( (void*) work_host[0].data, data, sizeof(int) * qsize, 
       hipMemcpyHostToDevice ) );
  cudaErrCheck( hipMalloc( &work_host[0].rec_data, sizeof(int) * qsize ) );
  cudaErrCheck( hipMemcpy( (void*) work_host[0].rec_data, data2, sizeof(int) * qsize, 
       hipMemcpyHostToDevice ) );
  work_host[0].first = 0;
  work_host[0].last = work_host[0].count;
  work_host[0].rec_count = 0;
  work_host[0].nr_avail_tasks = work_host[0].count;
  work_host[0].count = nr_high;

  /* Create the low priority queue. */
  work_host[1].count = 0;
  for(i = 0; i < num_gpu_tasks; i++ )
  {
    if(tasks_host[i].type > type_load && tasks_host[i].weight < cut){
      if(tasks_host[i].wait == 0) {
        data[work_host[0].count] = i;
        data2[work_host[0].count++] = -1;
      }
      nr_low++;
    }
  }
  work_host[1].first = 0;
  work_host[1].last = work_host[0].count;
  work_host[1].rec_count = 0;
  work_host[1].nr_avail_tasks = work_host[1].count;
  work_host[1].count = nr_low;

  /* Copy the work queues to the GPU */
  cudaErrCheck( hipMemcpyToSymbol(HIP_SYMBOL( cuda_queues), &work_host , sizeof(struct queue_cuda) * 2 ) ); 


  /* This is no longer the first run. */
  firstrun = 1;
  /* TODO Make sure we free everything we made here otherwise leaky code. */
  /* Free the tasks_host array. */
  cudaErrCheck( hipHostFree( tasks_host ) );
  tasks_host = NULL;
  /* Free the data and data 2 arrays.*/
  free(data); data = NULL;
  free(data2); data2=NULL;
  /* Host dependency array has been copied now, so time to remove it.*/
  free(host_dependencies);
  host_dependencies = NULL;
  /* Free cell_host and host_pointers */
  free(cell_host); cell_host = NULL;
  free(host_pointers); host_pointers = NULL;
}
