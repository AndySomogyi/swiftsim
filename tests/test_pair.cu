#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of SWIFT.
 * Copyright (C) 2015 Matthieu Schaller (matthieu.schaller@durham.ac.uk).
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ******************************************************************************/

// NEED TO USE
//   gpuErrchk( hipPeekAtLastError() );
//  gpuErrchk( hipDeviceSynchronize() );

#ifndef static
#define static
#endif
#ifndef restrict
#define restrict __restrict__
#endif
//#define FULL_COMP
extern "C" {
#include "testcuda.h"
/* Some standard headers. */
#include <fenv.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

}

#include <hip/hip_runtime.h>


/* Host function to check cuda functions don't return errors */
__host__ inline void cudaErrCheck(hipError_t status) {
  if (status != hipSuccess) {
    printf("%s\n", hipGetErrorString(status));
  }
}

/* Task function to execute a force task. Uses naive n^2 algorithm without
 * symmetry*/
/* To do force between Cell i and cell j this needs to be called twice. */
__device__ void dopair_force(struct cell_cuda *ci, struct cell_cuda *cj) {

  /* Are these cells active? */
  if (!cuda_cell_is_active(ci) && !cuda_cell_is_active(cj)) return;

  const int count_i = ci->part_count;
  const int count_j = cj->part_count;
  int part_i = ci->first_part;
  int part_j = cj->first_part;

  float3 a_hydro;
  float h_dt, v_sig_stor, entropy_dt;

  double shift[3] = {0.0, 0.0, 0.0};
  /* Deal with periodicity concerns. */
  for (int k = 0; k < 3; k++) {
    if (cj->loc[k] - ci->loc[k] < -dim[k] / 2)
      shift[k] = dim[k];
    else if (cj->loc[k] - ci->loc[k] > dim[k] / 2)
      shift[k] = -dim[k];
  }

  /* Loop over the parts in cell ci */
  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {

    const float hi = cuda_parts.h[pid];
    if (!cuda_part_is_active(pid)) continue;
    /* Reset the values. */
    a_hydro.x = 0.0f;
    a_hydro.y = 0.0f;
    a_hydro.z = 0.0f;
    h_dt = 0.0f;
    v_sig_stor = cuda_parts.v_sig[pid];
    entropy_dt = 0.0f;

    double pix[3];
    pix[0] = cuda_parts.x_x[pid] - shift[0];
    pix[1] = cuda_parts.x_y[pid] - shift[1];
    pix[2] = cuda_parts.x_z[pid] - shift[2];

    const float hig2 = hi * hi * kernel_gamma2;

    /* Loop over the particles in cj. */
    for (int pjd = part_j; pjd < part_j + count_j; pjd++) {

      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];

      const float hj = cuda_parts.h[pjd];
      if (r2 < hig2 || r2 < hj * hj * kernel_gamma2) {
        float wi, wj, wi_dx, wj_dx;
        const float fac_mu = 1.f;

        const float r = sqrtf(r2);
        const float r_inv = 1.0f / r;

        /* Load some data.*/
        const float mj = cuda_parts.mass[pjd];
        const float rhoi = cuda_parts.rho[pid];
        const float rhoj = cuda_parts.rho[pjd];

        /* Get the kernel for hi. */
        const float hi_inv = 1.0f / hi;
        const float hid_inv = cuda_pow_dimension_plus_one(hi_inv);
        const float ui = r * hi_inv;
        cuda_kernel_deval(ui, &wi, &wi_dx);
        const float wi_dr = hid_inv * wi_dx;

        /* Get the kernel for hj. */
        const float hj_inv = 1.0f / hj;
        const float hjd_inv = cuda_pow_dimension_plus_one(hj_inv);
        const float xj = r * hj_inv;
        cuda_kernel_deval(xj, &wj, &wj_dx);
        const float wj_dr = hjd_inv * wj_dx;

        /* Compute h-gradient terms */
        const float f_i = cuda_parts.f[pid];
        const float f_j = cuda_parts.f[pjd];

        /* Compute pressure terms */
        const float P_over_rho2_i = cuda_parts.P_over_rho2[pid];
        const float P_over_rho2_j = cuda_parts.P_over_rho2[pjd];

        /* Compute sound speeds*/
        const float ci = cuda_parts.soundspeed[pid];
        const float cj = cuda_parts.soundspeed[pjd];

        /* Compute dv dot r. */
        const float dvdr = (cuda_parts.v[pid].x - cuda_parts.v[pjd].x) * dx[0] +
                           (cuda_parts.v[pid].y - cuda_parts.v[pjd].y) * dx[1] +
                           (cuda_parts.v[pid].z - cuda_parts.v[pjd].z) * dx[2];

        /* Balsara term */
        const float balsara_i = cuda_parts.balsara[pid];
        const float balsara_j = cuda_parts.balsara[pjd];

        /* Are the particles moving towards each other? */
        const float omega_ij = (dvdr < 0.f) ? dvdr : 0.f;
        const float mu_ij = fac_mu * r_inv * omega_ij;

        /* Signal velocity */
        const float v_sig = ci + cj - 3.f * mu_ij;

        /* Now construct the full viscosity term */
        const float rho_ij = 0.5f * (rhoi + rhoj);
        const float visc = -0.25 * const_viscosity_alpha * v_sig * mu_ij *
                           (balsara_i + balsara_j) / rho_ij;

        /* Now convolce with the kernel */
        const float visc_term = 0.5f * visc * (wi_dr + wj_dr) * r_inv;
        const float sph_term =
            (f_i * P_over_rho2_i * wi_dr + f_j * P_over_rho2_j * wj_dr) * r_inv;

        /* Compute the acceleration */
        const float acc = visc_term + sph_term;

        /* Compute the force */
        a_hydro.x -= mj * acc * dx[0];
        a_hydro.y -= mj * acc * dx[1];
        a_hydro.z -= mj * acc * dx[2];

        /* Get the time derivative for h. */
        h_dt -= mj * dvdr * r_inv / rhoj * wi_dr;

        /* Update the signal velocity. */
        v_sig_stor = (v_sig_stor > v_sig) ? v_sig_stor : v_sig;

        /* Change in entropy */
        entropy_dt += mj * visc_term * dvdr;
        // INTERACT
      }

    }  // Loop over cell cj.

    /* Flush to global stores.*/
    atomicAdd(&cuda_parts.a_hydro[pid].x, a_hydro.x);
    atomicAdd(&cuda_parts.a_hydro[pid].y, a_hydro.y);
    atomicAdd(&cuda_parts.a_hydro[pid].z, a_hydro.z);
    atomicAdd(&cuda_parts.h_dt[pid], h_dt);
    atomicAdd(&cuda_parts.entropy_dt[pid], entropy_dt);

    /* Update the signal velocity */
    float global_vsig = cuda_parts.v_sig[pid];
    int *address_as_int = (int *)&cuda_parts.v_sig[pid];
    int old = *address_as_int;
    int assumed;
    do {
      global_vsig = cuda_parts.v_sig[pid];
      assumed = old;
      if (v_sig_stor > global_vsig)
        old = atomicCAS(address_as_int, assumed, __float_as_int(v_sig_stor));
    } while (assumed != old && v_sig_stor > global_vsig);

  }  // Loop over cell ci.
}

/**
 * @brief Compute the interactions between a cell pair (non-symmetric).
 *
 * @param r The #runner.
 * @param ci The first #cell.
 * @param cj The second #cell.
 * @param sid The direction of the pair
 */
__device__ void dopair_density_sorted(struct cell_cuda *ci, struct cell_cuda *cj,
				      const int sid, const int swap) {

  /* Pick-out the sorted lists. */
  const long long int f_i = ci->first_part;
  const long long int f_j = cj->first_part;
  const struct entry *sort_i = &cuda_parts.sort[sid][f_i];
  const struct entry *sort_j = &cuda_parts.sort[sid][f_j];

  /* Get some other useful values. */
  const double hi_max = ci->h_max * kernel_gamma;
  const int count_i = ci->part_count;
  const int count_j = cj->part_count;
  const double dj_min = sort_j->d;
  float rho, rho_dh, div_v, wcount, wcount_dh;
  float3 rot_v;

  if (cuda_cell_is_active(ci)) {

    int pid = count_i - 1 - threadIdx.x;
    /* Loop over the parts in ci. */
    while (pid >= 0 && sort_i[pid].d + hi_max > dj_min) {
      /* Get a hold of the ith part in ci. */
      int pi;
      float hi;
      double di;
      while (1) {
	pi = sort_i[pid].i + f_i;
	hi = cuda_parts.h[pi];
	di = sort_i[pid].d + hi * kernel_gamma;
	if (cuda_part_is_active(pi) && di < dj_min) {
	  break;
	}
	pid -= blockDim.x;
	if (pid < 0 || sort_i[pid].d + hi_max < dj_min) {
	  break;
	}
	continue;
      }
      __syncthreads();

      if (pid < 0 || sort_i[pid].d + hi_max < dj_min)
	break;
      
      double pix[3], piv[3];
      pix[0] = cuda_parts.x_x[pi];
      pix[1] = cuda_parts.x_y[pi];
      pix[2] = cuda_parts.x_z[pi];
      
      piv[0] = cuda_parts.v[pi].x;
      piv[1] = cuda_parts.v[pi].y;
      piv[2] = cuda_parts.v[pi].z;
      const float hig2 = hi * hi * kernel_gamma2;
      rho = 0.0f;
      rho_dh = 0.0f;
      div_v = 0.0f;
      wcount = 0.0f;
      wcount_dh = 0.0f;
      rot_v.x = 0.0f;
      rot_v.y = 0.0f;
      rot_v.z = 0.0f;


      /* Loop over the parts in cj. */
      for (int pjd = 0; pjd < count_j && sort_j[pjd].d < di; pjd++) {
	//printf("pjd : %lli\n", pjd);
        /* Get a pointer to the jth particle. */
        const int pj = sort_j[pjd].i + f_j;

        /* Compute the pairwise distance. */
        float r2 = 0.0f;
        float dx[3];
	dx[0] = pix[0] - cuda_parts.x_x[pj];
	dx[1] = pix[1] - cuda_parts.x_y[pj];
	dx[2] = pix[2] - cuda_parts.x_z[pj];
        for (int k = 0; k < 3; k++) {
          r2 += dx[k] * dx[k];
        }


        /* Hit or miss? */
        if (r2 < hig2) {
	  
	  float wi, wi_dx;
	  float dv[3], curlvr[3];
	    
	  /* Get the masses. */
	  const float mj = cuda_parts.mass[pj];
	    
	  /* Get r and r inverse. */
	  const float r = sqrtf(r2);
	  const float r_inv = 1.0f / r;

	  /* Compute the kernel function */
	  const float hi_inv = 1.0f / hi;
	  const float ui = r * hi_inv;
	  cuda_kernel_deval(ui, &wi, &wi_dx);

	  /* Compute contribution to the density */
	  rho += mj * wi;
	  rho_dh -= mj * (hydro_dimension * wi + ui * wi_dx);

	  /* Compute contribution to the number of neighbours */
	  wcount += wi;
	  wcount_dh -= (hydro_dimension * wi + ui * wi_dx);

	  const float fac = mj * wi_dx * r_inv;

	  /* Compute dv dot r */
	  dv[0] = piv[0] - cuda_parts.v[pj].x;
	  dv[1] = piv[1] - cuda_parts.v[pj].y;
	  dv[2] = piv[2] - cuda_parts.v[pj].z;
	  const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];
	  div_v -= fac * dvdr;

	  /* Compute dv cross r */
	  curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
	  curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
	  curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];
	  
	  rot_v.x += fac * curlvr[0];
	  rot_v.y += fac * curlvr[1];
	  rot_v.z += fac * curlvr[2];
	}
	
      }
      int pid_write = pid + f_i;
      atomicAdd(&cuda_parts.rho[pid_write], rho);
      atomicAdd(&cuda_parts.rho_dh[pid_write], rho_dh);
      atomicAdd(&cuda_parts.wcount[pid_write], wcount);
      atomicAdd(&cuda_parts.wcount_dh[pid_write], wcount_dh);
      atomicAdd(&cuda_parts.div_v[pid_write], div_v);
      atomicAdd(&cuda_parts.rot_v[pid_write].x, rot_v.x);
      atomicAdd(&cuda_parts.rot_v[pid_write].y, rot_v.y);
      atomicAdd(&cuda_parts.rot_v[pid_write].z, rot_v.z);

      pid-= blockDim.x;
    } /* loop over the parts in cj. */
    
  } /* loop over the parts in ci. */
  
} /* Cell ci is active */

/* Task function to execute a density task. Uses naive n^2 algorithm without
 * symmetry*/
/* To do density between Cell i and cell j this needs to be called twice. */
__device__ void dopair_density(struct cell_cuda *ci, struct cell_cuda *cj) {

  /* Are these cells active? */
  if (!cuda_cell_is_active(ci) && !cuda_cell_is_active(cj)) return;

  const int count_i = ci->part_count;
  const int count_j = cj->part_count;
  int part_i = ci->first_part;
  int part_j = cj->first_part;
  float rho, rho_dh, div_v, wcount, wcount_dh;
  float3 rot_v;
  double shift[3] = {0.0, 0.0, 0.0};

  /* Deal with periodicity concerns. */
  for (int k = 0; k < 3; k++) {
    if (cj->loc[k] - ci->loc[k] < -dim[k] / 2)
      shift[k] = dim[k];
    else if (cj->loc[k] - ci->loc[k] > dim[k] / 2)
      shift[k] = -dim[k];
  }

  /* Loop over the parts in cell ci */
  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {

    const float hi = cuda_parts.h[pid];

    double pix[3];
    pix[0] = cuda_parts.x_x[pid] - shift[0];
    pix[1] = cuda_parts.x_y[pid] - shift[1];
    pix[2] = cuda_parts.x_z[pid] - shift[2];
    const float hig2 = hi * hi * kernel_gamma2;

    if (!cuda_part_is_active(pid)) continue;

    /* Reset local values. */
    rho = 0.0f;
    rho_dh = 0.0f;
    div_v = 0.0f;
    wcount = 0.0f;
    wcount_dh = 0.0f;
    rot_v.x = 0.0f;
    rot_v.y = 0.0f;
    rot_v.z = 0.0f;

    /* Loop over the parts in cj. */
    /* TODO May be possible to optimize this loop ordering.*/
    for (int pjd = part_j; pjd < part_j + count_j; pjd++) {

      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];
      /* If in range then interact. */
      if (r2 < hig2) {
        float w, dw_dx;
        float dv[3], curlvr[3];

        /* Load mass on particle pj. */
        const float mj = cuda_parts.mass[pjd];

        /* Get r and 1/r */
        const float r = sqrtf(r2);
        const float ri = 1.0f / r;

        /* Compute the kernel function */
        const float hi_inv = 1.0f / hi;
        const float ui = r * hi_inv;

        cuda_kernel_deval(ui, &w, &dw_dx);

        /* Compute contribution to the density. */
        rho += mj * w;
        rho_dh -= mj * (hydro_dimension * w + ui * dw_dx);

        /* Compute contribution to the number of neighbours */
        wcount += w;
        wcount_dh -= (hydro_dimension * w + ui * dw_dx);

        const float fac = mj * dw_dx * ri;

        /* Compute dv dot r */
        float3 piv, pjv;
        piv = cuda_parts.v[pid];
        pjv = cuda_parts.v[pjd];
        dv[0] = piv.x - pjv.x;
        dv[1] = piv.y - pjv.y;
        dv[2] = piv.z - pjv.z;
        const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

        div_v -= fac * dvdr;

        curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
        curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
        curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];
        
        rot_v.x += fac * curlvr[0];
        rot_v.y += fac * curlvr[1];
        rot_v.z += fac * curlvr[2];
        // w);
      }
    }  // Loop over cj.
    /* Write data for particle pid back to global stores. */
    atomicAdd(&cuda_parts.rho[pid], rho);
    atomicAdd(&cuda_parts.rho_dh[pid], rho_dh);
    atomicAdd(&cuda_parts.wcount[pid], wcount);
    atomicAdd(&cuda_parts.wcount_dh[pid], wcount_dh);
    atomicAdd(&cuda_parts.div_v[pid], div_v);
    atomicAdd(&cuda_parts.rot_v[pid].x, rot_v.x);
    atomicAdd(&cuda_parts.rot_v[pid].y, rot_v.y);
    atomicAdd(&cuda_parts.rot_v[pid].z, rot_v.z);

  }  // Loop over ci.
}

/* Task function to perform a self force task. No symmetry */
__device__ void doself_force(struct cell_cuda *ci) {
  /* Is the cell active? */
  if (!cuda_cell_is_active(ci)) return;

  const int count_i = ci->part_count;
  int part_i = ci->first_part;

  float3 a_hydro;
  float h_dt, v_sig_stor, entropy_dt;

  /* Loop over the particles */
  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {

    const float hi = cuda_parts.h[pid];
    if (!cuda_part_is_active(pid)) continue;
    /* Reset the values. */
    a_hydro.x = 0.0f;
    a_hydro.y = 0.0f;
    a_hydro.z = 0.0f;
    h_dt = 0.0f;
    v_sig_stor = cuda_parts.v_sig[pid];
    entropy_dt = 0.0f;

    double pix[3];
    pix[0] = cuda_parts.x_x[pid];
    pix[1] = cuda_parts.x_y[pid];
    pix[2] = cuda_parts.x_z[pid];

    const float hig2 = hi * hi * kernel_gamma2;

    /* Loop over the particles in cj. */
    for (int pjd = part_i; pjd < part_i + count_i; pjd++) {

      if (pid == pjd) continue;
      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];
      const float hj = cuda_parts.h[pjd];
      if (r2 < hig2 || r2 < hj * hj * kernel_gamma2) {
        float wi, wj, wi_dx, wj_dx;
        const float fac_mu = 1.f;

        const float r = sqrtf(r2);
        const float r_inv = 1.0f / r;

        /* Load some data.*/
        const float mj = cuda_parts.mass[pjd];
        const float rhoi = cuda_parts.rho[pid];
        const float rhoj = cuda_parts.rho[pjd];

        /* Get the kernel for hi. */
        const float hi_inv = 1.0f / hi;
        const float hid_inv = cuda_pow_dimension_plus_one(hi_inv);
        const float ui = r * hi_inv;
        cuda_kernel_deval(ui, &wi, &wi_dx);
        const float wi_dr = hid_inv * wi_dx;

        /* Get the kernel for hj. */
        const float hj_inv = 1.0f / hj;
        const float hjd_inv = cuda_pow_dimension_plus_one(hj_inv);
        const float xj = r * hj_inv;
        cuda_kernel_deval(xj, &wj, &wj_dx);
        const float wj_dr = hjd_inv * wj_dx;

        /* Compute h-gradient terms */
        const float f_i = cuda_parts.f[pid];
        const float f_j = cuda_parts.f[pjd];

        /* Compute pressure terms */
        const float P_over_rho2_i = cuda_parts.P_over_rho2[pid];
        const float P_over_rho2_j = cuda_parts.P_over_rho2[pjd];

        /* Compute sound speeds*/
        const float ci = cuda_parts.soundspeed[pid];
        const float cj = cuda_parts.soundspeed[pjd];

        /* Compute dv dot r. */
        const float dvdr = (cuda_parts.v[pid].x - cuda_parts.v[pjd].x) * dx[0] +
                           (cuda_parts.v[pid].y - cuda_parts.v[pjd].y) * dx[1] +
                           (cuda_parts.v[pid].z - cuda_parts.v[pjd].z) * dx[2];

        /* Balsara term */
        const float balsara_i = cuda_parts.balsara[pid];
        const float balsara_j = cuda_parts.balsara[pjd];

        /* Are the particles moving towards each other? */
        const float omega_ij = (dvdr < 0.f) ? dvdr : 0.f;
        const float mu_ij = fac_mu * r_inv * omega_ij;

        /* Signal velocity */
        const float v_sig = ci + cj - 3.f * mu_ij;
        const float rho_ij = 0.5f * (rhoi + rhoj);
        const float visc = -0.25 * const_viscosity_alpha * v_sig * mu_ij *
                           (balsara_i + balsara_j) / rho_ij;

        /* Now convolce with the kernel */
        const float visc_term = 0.5f * visc * (wi_dr + wj_dr) * r_inv;
        const float sph_term =
            (f_i * P_over_rho2_i * wi_dr + f_j * P_over_rho2_j * wj_dr) * r_inv;

        /* Compute the acceleration */
        const float acc = visc_term + sph_term;

        /* Compute the force */
        a_hydro.x -= mj * acc * dx[0];
        a_hydro.y -= mj * acc * dx[1];
        a_hydro.z -= mj * acc * dx[2];


        /* Get the time derivative for h. */
        h_dt -= mj * dvdr * r_inv / rhoj * wi_dr;

        /* Update the signal velocity. */
        v_sig_stor = (v_sig_stor > v_sig) ? v_sig_stor : v_sig;

        /* Change in entropy */
        entropy_dt += mj * visc_term * dvdr;
      }

    }  // Inner loop

    /* Flush to global stores.*/
    atomicAdd(&cuda_parts.a_hydro[pid].x, a_hydro.x);
    atomicAdd(&cuda_parts.a_hydro[pid].y, a_hydro.y);
    atomicAdd(&cuda_parts.a_hydro[pid].z, a_hydro.z);
    atomicAdd(&cuda_parts.h_dt[pid], h_dt);
    atomicAdd(&cuda_parts.entropy_dt[pid], entropy_dt);

    /* Update the signal velocity */
    float global_vsig = cuda_parts.v_sig[pid];
    int *address_as_int = (int *)&cuda_parts.v_sig[pid];
    int old = *address_as_int;
    int assumed;
    do {
      global_vsig = cuda_parts.v_sig[pid];  // Scary line.
      assumed = old;
      if (v_sig_stor > global_vsig)
        old = atomicCAS(address_as_int, assumed, __float_as_int(v_sig_stor));
    } while (assumed != old && v_sig_stor > global_vsig);

  }  // Outer loop
}

/* Task function to execute a self-density task. */
__device__ void doself_density_symmetric(struct cell_cuda *ci) {

  /* Is the cell active? */
  if (!cuda_cell_is_active(ci)) {
    printf(
        "Cell isn't active..., ti_end_min=%i, ti_current=%i, "
        "max_active_bin=%i\n",
        ci->ti_end_min, ti_current, max_active_bin);
    return;
  }

  const int count_i = ci->part_count;
  int part_i = ci->first_part;
  float rho, rho_dh, div_v, wcount, wcount_dh;
  float3 rot_v;

  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {
    double pix[3];
    pix[0] = cuda_parts.x_x[pid];
    pix[1] = cuda_parts.x_y[pid];
    pix[2] = cuda_parts.x_z[pid];
    const float hi = cuda_parts.h[pid];
    const float hig2 = hi * hi * kernel_gamma2;

    /* Reset local values. */
    rho = 0.0f;
    rho_dh = 0.0f;
    div_v = 0.0f;
    wcount = 0.0f;
    wcount_dh = 0.0f;
    rot_v.x = 0.0f;
    rot_v.y = 0.0f;
    rot_v.z = 0.0f;

    /* Search for the neighbours! */
    for (int pjd = pid + 1; pjd < part_i + count_i; pjd++) {
      /* Particles don't interact with themselves */
      float dx[3], r2 = 0.0f;
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];

      const float hj = cuda_parts.h[pjd];
      const float hjg2 = hj * hj * kernel_gamma2;
      /* If in range then interact. */
      if (r2 < hig2 || r2 < hjg2) {
        float w, dw_dx;
        float dv[3], curlvr[3];
        /* Get r and 1/r */
        const float r = sqrtf(r2);
        const float ri = 1.0f / r;

	if (r2 < hig2 && cuda_part_is_active(pid)) {
	  /* Load mass on particle pj. */
	  const float mj = cuda_parts.mass[pjd];


	  /* Compute the kernel function */
	  const float hi_inv = 1.0f / hi;
	  const float ui = r * hi_inv;

	  cuda_kernel_deval(ui, &w, &dw_dx);
	  /* Compute contribution to the density. */
	  rho = mj * w;
	  rho_dh = - mj * (hydro_dimension * w + ui * dw_dx);

	  /* Compute contribution to the number of neighbours */
	  wcount = w;
	  wcount_dh = -(hydro_dimension * w + ui * dw_dx);

	  const float fac = mj * dw_dx * ri;

	  /* Compute dv dot r */
	  float3 piv, pjv;
	  piv = cuda_parts.v[pid];
	  pjv = cuda_parts.v[pjd];
	  dv[0] = piv.x - pjv.x;
	  dv[1] = piv.y - pjv.y;
	  dv[2] = piv.z - pjv.z;
	  const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

	  div_v = - fac * dvdr;

	  curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
	  curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
	  curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];
	  
	  rot_v.x = fac * curlvr[0];
	  rot_v.y = fac * curlvr[1];
	  rot_v.z = fac * curlvr[2];

	  atomicAdd(&cuda_parts.rho[pid], rho);
	  atomicAdd(&cuda_parts.rho_dh[pid], rho_dh);
	  atomicAdd(&cuda_parts.wcount[pid], wcount);
	  atomicAdd(&cuda_parts.wcount_dh[pid], wcount_dh);
	  atomicAdd(&cuda_parts.div_v[pid], div_v);
	  atomicAdd(&cuda_parts.rot_v[pid].x, rot_v.x);
	  atomicAdd(&cuda_parts.rot_v[pid].y, rot_v.y);
	  atomicAdd(&cuda_parts.rot_v[pid].z, rot_v.z);
	}
	
	if (r2 < hjg2 && cuda_part_is_active(pjd)) {
	  /* Load mass on particle pj. */
	  const float mi = cuda_parts.mass[pid];


	  /* Compute the kernel function */
	  const float hj_inv = 1.0f / hj;
	  const float uj = r * hj_inv;

	  cuda_kernel_deval(uj, &w, &dw_dx);
	  /* Compute contribution to the density. */
	  rho = mi * w;
	  rho_dh = - mi * (hydro_dimension * w + uj * dw_dx);

	  /* Compute contribution to the number of neighbours */
	  wcount = w;
	  wcount_dh = -(hydro_dimension * w + uj * dw_dx);

	  const float fac = mi * dw_dx * ri;

	  /* Compute dv dot r */
	  float3 piv, pjv;
	  piv = cuda_parts.v[pid];
	  pjv = cuda_parts.v[pjd];
	  dv[0] = pjv.x - piv.x;
	  dv[1] = pjv.y - piv.y;
	  dv[2] = pjv.z - piv.z;
	  const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

	  div_v = - fac * dvdr;

	  curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
	  curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
	  curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];
	  
	  rot_v.x = fac * curlvr[0];
	  rot_v.y = fac * curlvr[1];
	  rot_v.z = fac * curlvr[2];

	  atomicAdd(&cuda_parts.rho[pjd], rho);
	  atomicAdd(&cuda_parts.rho_dh[pjd], rho_dh);
	  atomicAdd(&cuda_parts.wcount[pjd], wcount);
	  atomicAdd(&cuda_parts.wcount_dh[pjd], wcount_dh);
	  atomicAdd(&cuda_parts.div_v[pjd], div_v);
	  atomicAdd(&cuda_parts.rot_v[pjd].x, rot_v.x);
	  atomicAdd(&cuda_parts.rot_v[pjd].y, rot_v.y);
	  atomicAdd(&cuda_parts.rot_v[pjd].z, rot_v.z);
	}
      }
    }  // Loop over cj.
    /* Write data for particle pid back to global stores. */
  }
}

/* Task function to execute a self-density task. */
__device__ void doself_density(struct cell_cuda *ci) {

  /* Is the cell active? */
  if (!cuda_cell_is_active(ci)) {
    printf(
        "Cell isn't active..., ti_end_min=%i, ti_current=%i, "
        "max_active_bin=%i\n",
        ci->ti_end_min, ti_current, max_active_bin);
    return;
  }

  const int count_i = ci->part_count;
  int part_i = ci->first_part;
  float rho, rho_dh, div_v, wcount, wcount_dh;
  float3 rot_v;

  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {
    double pix[3];
    pix[0] = cuda_parts.x_x[pid];
    pix[1] = cuda_parts.x_y[pid];
    pix[2] = cuda_parts.x_z[pid];
    const float hi = cuda_parts.h[pid];
    const float hig2 = hi * hi * kernel_gamma2;

    /* Reset local values. */
    rho = 0.0f;
    rho_dh = 0.0f;
    div_v = 0.0f;
    wcount = 0.0f;
    wcount_dh = 0.0f;
    rot_v.x = 0.0f;
    rot_v.y = 0.0f;
    rot_v.z = 0.0f;

    /* If the particle isn't active skip it. */
    if (!cuda_part_is_active(pid)) {
      continue;
    }

    /* Search for the neighbours! */
    for (int pjd = part_i; pjd < part_i + count_i; pjd++) {
      /* Particles don't interact with themselves */
      if (pid == pjd) continue;
      float dx[3], r2 = 0.0f;
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];

      /* If in range then interact. */
      if (r2 < hig2) {
        float w, dw_dx;
        float dv[3], curlvr[3];
        /* Load mass on particle pj. */
        const float mj = cuda_parts.mass[pjd];

        /* Get r and 1/r */
        const float r = sqrtf(r2);
        const float ri = 1.0f / r;

        /* Compute the kernel function */
        const float hi_inv = 1.0f / hi;
        const float ui = r * hi_inv;

        cuda_kernel_deval(ui, &w, &dw_dx);
        /* Compute contribution to the density. */
        rho += mj * w;
        rho_dh -= mj * (hydro_dimension * w + ui * dw_dx);

        /* Compute contribution to the number of neighbours */
        wcount += w;
        wcount_dh -= (hydro_dimension * w + ui * dw_dx);

        const float fac = mj * dw_dx * ri;

        /* Compute dv dot r */
        float3 piv, pjv;
        piv = cuda_parts.v[pid];
        pjv = cuda_parts.v[pjd];
        dv[0] = piv.x - pjv.x;
        dv[1] = piv.y - pjv.y;
        dv[2] = piv.z - pjv.z;
        const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

        div_v -= fac * dvdr;

        curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
        curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
        curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];

        rot_v.x += fac * curlvr[0];
        rot_v.y += fac * curlvr[1];
        rot_v.z += fac * curlvr[2];
      }
    }  // Loop over cj.
    /* Write data for particle pid back to global stores. */
    atomicAdd(&cuda_parts.rho[pid], rho);
    atomicAdd(&cuda_parts.rho_dh[pid], rho_dh);
    atomicAdd(&cuda_parts.wcount[pid], wcount);
    atomicAdd(&cuda_parts.wcount_dh[pid], wcount_dh);
    atomicAdd(&cuda_parts.div_v[pid], div_v);
    atomicAdd(&cuda_parts.rot_v[pid].x, rot_v.x);
    atomicAdd(&cuda_parts.rot_v[pid].y, rot_v.y);
    atomicAdd(&cuda_parts.rot_v[pid].z, rot_v.z);
  }
}


__global__ void do_test_self_density_symmetric(struct cell_cuda *ci) {

  doself_density_symmetric(ci);

}

__global__ void do_test_self_density(struct cell_cuda *ci) {

  doself_density(ci);

}

__global__ void do_test_self_force(struct cell_cuda *ci) {

  doself_force(ci);

}
__global__ void do_test_pair_density(struct cell_cuda *ci, struct cell_cuda *cj) {

  dopair_density(ci,cj);
  dopair_density(cj,ci);

}

__global__ void do_test_pair_density_sorted(struct cell_cuda *ci, struct cell_cuda *cj, const int sid, const int swap) {

  dopair_density_sorted(ci, cj, sid, swap);
  dopair_density_sorted(cj, ci, sid, -swap);

}

__global__ void do_test_pair_force(struct cell_cuda *ci, struct cell_cuda *cj) {

  dopair_force(ci,cj);
  dopair_force(cj,ci);
}

void allocate_parts(struct particle_arrays *p, size_t num_part) { 
  p->id = (long long int*) malloc (sizeof(long long int) * num_part);
  p->x_x = (double*) malloc(sizeof(double) * num_part);
  p->x_y = (double*) malloc(sizeof(double) * num_part);
  p->x_z = (double*) malloc(sizeof(double) * num_part);
  p->v = (float3*) malloc(sizeof(float3) * num_part);
  p->a_hydro = (float3*) malloc(sizeof(float3) * num_part);
  p->h = (float*) malloc(sizeof(float) * num_part);
  p->mass = (float*) malloc(sizeof(float) * num_part);
  p->rho = (float*) malloc(sizeof(float) * num_part);
  p->entropy = (float*) malloc(sizeof(float) * num_part);
  p->entropy_dt = (float*) malloc(sizeof(float) * num_part);

  // density
  p->wcount = (float*) malloc(sizeof(float) * num_part);
  p->wcount_dh = (float*) malloc(sizeof(float) * num_part);
  p->rho_dh = (float*) malloc(sizeof(float) * num_part);
  p->rot_v = (float3*) malloc(sizeof(float3) * num_part);
  p->div_v = (float*) malloc(sizeof(float) * num_part);

  // force
  p->balsara = (float*) malloc(sizeof(float) * num_part);
  p->f = (float*) malloc(sizeof(float) * num_part);
  p->P_over_rho2 = (float*) malloc(sizeof(float) * num_part);
  p->soundspeed = (float*) malloc(sizeof(float) * num_part);
  p->v_sig = (float*) malloc(sizeof(float) * num_part);
  p->h_dt = (float*) malloc(sizeof(float) * num_part);
  
  p->time_bin = (timebin_t*) malloc(sizeof(timebin_t) * num_part);

  for (int i=0; i<NBRE_DIR; i++)
    p->sort[i] = (struct entry *)malloc(sizeof(struct entry)*(num_part + 1));

}

void allocate_device_parts(size_t num_part) {
  struct particle_arrays c_parts;
  
  cudaErrCheck(hipMalloc(&c_parts.id, sizeof(long long int) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.x_x, sizeof(double) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.x_y, sizeof(double) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.x_z, sizeof(double) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.v, sizeof(float3) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.a_hydro, sizeof(float3) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.h, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.mass, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.rho, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.entropy, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.entropy_dt, sizeof(float) * num_part));

  // density
  cudaErrCheck(hipMalloc(&c_parts.wcount, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.wcount_dh, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.rho_dh, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.rot_v, sizeof(float3) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.div_v, sizeof(float) * num_part));

  // force
  cudaErrCheck(hipMalloc(&c_parts.balsara, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.f, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.P_over_rho2, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.soundspeed, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.v_sig, sizeof(volatile float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.h_dt, sizeof(float) * num_part));

  cudaErrCheck(hipMalloc(&c_parts.time_bin, sizeof(timebin_t) * num_part));

  for (int i=0; i<NBRE_DIR; i++)
    cudaErrCheck(hipMalloc(&c_parts.sort[i], sizeof(struct entry) * num_part));

   cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_parts), &c_parts, sizeof(struct particle_arrays)));

}

void free_parts(struct particle_arrays p) {
  free(p.id); p.id = NULL;
  free(p.x_x); p.x_x = NULL;
  free(p.x_y); p.x_y = NULL;
  free(p.x_z); p.x_z = NULL;
  free(p.v); p.v = NULL;
  free(p.a_hydro); p.a_hydro = NULL;
  free(p.h); p.h = NULL;
  free(p.mass); p.mass = NULL;
  free(p.rho); p.rho = NULL;
  free(p.entropy); p.entropy = NULL;
  free(p.entropy_dt); p.entropy_dt = NULL;

  // density
  free(p.wcount); p.wcount = NULL;
  free(p.wcount_dh); p.wcount_dh = NULL;
  free(p.rho_dh); p.rho_dh = NULL;
  free(p.rot_v); p.rot_v = NULL;
  free(p.div_v); p.div_v = NULL;

  // force
  free(p.balsara); p.balsara = NULL;
  free(p.f); p.f = NULL;
  free(p.P_over_rho2); p.P_over_rho2 = NULL;
  free(p.soundspeed); p.soundspeed = NULL;
  free((void *)p.v_sig); p.v_sig = NULL;
  free(p.h_dt); p.h_dt = NULL;
  
  free(p.time_bin); p.time_bin = NULL;
}

void free_device_parts() {
  struct particle_arrays parts;
  cudaErrCheck(hipMemcpyFromSymbol(&parts, HIP_SYMBOL(cuda_parts),
				    sizeof(struct particle_arrays)));

  cudaErrCheck(hipFree(parts.id));
  cudaErrCheck(hipFree(parts.x_x));
  cudaErrCheck(hipFree(parts.x_y));
  cudaErrCheck(hipFree(parts.x_z));
  cudaErrCheck(hipFree(parts.v));
  cudaErrCheck(hipFree(parts.a_hydro));
  cudaErrCheck(hipFree(parts.h));
  cudaErrCheck(hipFree(parts.mass));
  cudaErrCheck(hipFree(parts.rho));
  cudaErrCheck(hipFree(parts.entropy));
  cudaErrCheck(hipFree(parts.entropy_dt));

  // density
  cudaErrCheck(hipFree(parts.wcount));
  cudaErrCheck(hipFree(parts.wcount_dh));
  cudaErrCheck(hipFree(parts.rho_dh));
  cudaErrCheck(hipFree(parts.rot_v));
  cudaErrCheck(hipFree(parts.div_v));

  // force
  cudaErrCheck(hipFree(parts.balsara));
  cudaErrCheck(hipFree(parts.f));
  cudaErrCheck(hipFree(parts.P_over_rho2));
  cudaErrCheck(hipFree(parts.soundspeed));
  cudaErrCheck(hipFree((void*)parts.v_sig));
  cudaErrCheck(hipFree(parts.h_dt));
  
  for (int i=0; i<NBRE_DIR; i++)
    cudaErrCheck(hipFree(parts.sort[i]));

  cudaErrCheck(hipFree(parts.time_bin));
}


void copy_to_device_array(struct cell *ci, int offset) {

  struct particle_arrays h_p;
    
  int num_part = ci->count;
  
  allocate_parts(&h_p, num_part);
  
  //copy particles data
  for(int i=0;i<num_part;i++) {
    struct part p = ci->parts[i];
    h_p.id[i] = p.id;
    h_p.x_x[i] = p.x[0];
    h_p.x_y[i] = p.x[1];
    h_p.x_z[i] = p.x[2];
    h_p.v[i].x = p.v[0];
    h_p.v[i].y = p.v[1];
    h_p.v[i].z = p.v[2];
    h_p.a_hydro[i].x = p.a_hydro[0];
    h_p.a_hydro[i].y = p.a_hydro[1];
    h_p.a_hydro[i].z = p.a_hydro[2];
    h_p.h[i] = p.h;
    h_p.mass[i] = p.mass;
    h_p.rho[i] = p.rho;
    h_p.entropy[i] = p.entropy;
    h_p.entropy_dt[i] = p.entropy_dt;

    // density
    h_p.wcount[i] = p.density.wcount;
    h_p.wcount_dh[i] = p.density.wcount_dh;
    h_p.rho_dh[i] = p.density.rho_dh;
    h_p.rot_v[i].x = p.density.rot_v[0];
    h_p.rot_v[i].y = p.density.rot_v[1];
    h_p.rot_v[i].z = p.density.rot_v[2];
    h_p.div_v[i] = p.density.div_v;

    // force
    h_p.balsara[i] = p.force.balsara;
    h_p.f[i] = p.force.f;
    h_p.P_over_rho2[i] = p.force.P_over_rho2;
    h_p.soundspeed[i] = p.force.soundspeed;
    h_p.v_sig[i] = p.force.v_sig;
    h_p.h_dt[i] = p.force.h_dt;

    for (int j=0; j<NBRE_DIR; j++) {
      h_p.sort[j][i].d = ci->sort[j][i].d;
      h_p.sort[j][i].i = ci->sort[j][i].i;
    }
      
    h_p.time_bin[i] = p.time_bin;
  }

  struct particle_arrays c_parts;
  cudaErrCheck(hipMemcpyFromSymbol(&c_parts, HIP_SYMBOL(cuda_parts),
                                    sizeof(struct particle_arrays)));

  void *p_data = c_parts.id + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.id, sizeof(long long int) * num_part,hipMemcpyHostToDevice));

  p_data = c_parts.x_x + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.x_x, sizeof(double) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.x_y + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.x_y, sizeof(double) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.x_z + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.x_z, sizeof(double) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.v + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.v, sizeof(float3) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.a_hydro + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.a_hydro,sizeof(float3) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.h + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.h,sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.mass + offset;
  cudaErrCheck(hipMemcpy(p_data,h_p.mass, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.rho + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.rho,sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.entropy + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.entropy,sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.entropy_dt + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.entropy_dt,sizeof(float) * num_part, hipMemcpyHostToDevice));

  // density
  p_data = c_parts.wcount + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.wcount, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.wcount_dh + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.wcount_dh, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.rho_dh + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.rho_dh, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.rot_v + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.rot_v, sizeof(float3) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.div_v + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.div_v, sizeof(float) * num_part, hipMemcpyHostToDevice));

  // force
  p_data = c_parts.balsara + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.balsara, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.f + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.f, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.P_over_rho2 + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.P_over_rho2, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.soundspeed + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.soundspeed, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = (void*)(c_parts.v_sig + offset);
  cudaErrCheck(hipMemcpy(p_data, (void*)h_p.v_sig, sizeof(volatile float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.h_dt + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.h_dt, sizeof(float) * num_part, hipMemcpyHostToDevice));

  
  for (int j=0; j<NBRE_DIR; j++) {
    p_data = c_parts.sort[j] + offset;
    cudaErrCheck(hipMemcpy(p_data, h_p.sort[j], sizeof(struct entry) * num_part, hipMemcpyHostToDevice));
  }
  
  p_data = c_parts.time_bin + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.time_bin,sizeof(timebin_t) * num_part, hipMemcpyHostToDevice));

  free_parts(h_p);
}

void copy_from_device_array(struct particle_arrays *h_p, int offset, size_t num_part) {

  struct particle_arrays c_parts;
  cudaErrCheck(hipMemcpyFromSymbol(&c_parts, HIP_SYMBOL(cuda_parts),
                                    sizeof(struct particle_arrays)));

  void *p_data = c_parts.id + offset;
  cudaErrCheck(hipMemcpy(h_p->id, p_data, sizeof(long long int) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.x_x + offset;
  cudaErrCheck(hipMemcpy(h_p->x_x, p_data, sizeof(double) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.x_y + offset;
  cudaErrCheck(hipMemcpy(h_p->x_y, p_data, sizeof(double) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.x_z + offset;
  cudaErrCheck(hipMemcpy(h_p->x_z, p_data, sizeof(double) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.v + offset;
  cudaErrCheck(hipMemcpy(h_p->v, p_data, sizeof(float3) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.a_hydro + offset;
  cudaErrCheck(hipMemcpy(h_p->a_hydro, p_data, sizeof(float3) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.h + offset;
  cudaErrCheck(hipMemcpy(h_p->h, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.mass + offset;
  cudaErrCheck(hipMemcpy(h_p->mass, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.rho + offset;
  cudaErrCheck(hipMemcpy(h_p->rho, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.entropy + offset;
  cudaErrCheck(hipMemcpy(h_p->entropy, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.entropy_dt + offset;
  cudaErrCheck(hipMemcpy(h_p->entropy_dt, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  // density
  p_data = c_parts.wcount + offset;
  cudaErrCheck(hipMemcpy(h_p->wcount, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.wcount_dh + offset;
  cudaErrCheck(hipMemcpy(h_p->wcount_dh, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.rho_dh + offset;
  cudaErrCheck(hipMemcpy(h_p->rho_dh, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.rot_v + offset;
  cudaErrCheck(hipMemcpy(h_p->rot_v, p_data, sizeof(float3) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.div_v + offset;
  cudaErrCheck(hipMemcpy(h_p->div_v, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  

  // force
  p_data = c_parts.balsara + offset;
  cudaErrCheck(hipMemcpy(h_p->balsara, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.f + offset;
  cudaErrCheck(hipMemcpy(h_p->f, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.P_over_rho2 + offset;
  cudaErrCheck(hipMemcpy(h_p->P_over_rho2, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.soundspeed + offset;
  cudaErrCheck(hipMemcpy(h_p->soundspeed, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = (void*)(c_parts.v_sig + offset);
  cudaErrCheck(hipMemcpy((void*)h_p->v_sig, p_data, sizeof(volatile float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.h_dt + offset;
  cudaErrCheck(hipMemcpy(h_p->h_dt, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  

  for (int j=0; j<NBRE_DIR; j++) {
    p_data = c_parts.sort[j] + offset;
    cudaErrCheck(hipMemcpy(h_p->sort[j], p_data, sizeof(struct entry) * num_part, hipMemcpyDeviceToHost));
  }

  p_data = c_parts.time_bin + offset;
  cudaErrCheck(hipMemcpy(h_p->time_bin, p_data, sizeof(timebin_t) * num_part, hipMemcpyDeviceToHost));
}

struct cell_cuda* copy_from_host(struct cell *ci, int offset) {
  copy_to_device_array(ci, offset);
  /* Set the host pointer. */
  struct cell_cuda c2;

  for (int k=0; k<DIM; k++) {
    c2.loc[k] = ci->loc[k];
    c2.width[k] = ci->width[k];
  
  }
  c2.h_max = ci->h_max;
  c2.first_part = offset;
  c2.part_count = ci->count;
  if (ci->parent != NULL) {
    c2.parent = ci->parent->cuda_ID;
  } else {
    c2.parent = -1;
  }
  if (ci->super != NULL) {
    c2.super = ci->super->cuda_ID;
  } else {
    c2.super = -1;
  }
  c2.ti_end_min = ci->ti_end_min;
  c2.ti_end_max = ci->ti_end_max;
  c2.dmin = ci->dmin;
  c2.nr_links = 0;
  c2.split = ci->split;

  struct cell_cuda *ret;
  cudaErrCheck( hipMalloc(&ret, sizeof(struct cell_cuda)) );
  cudaErrCheck( hipMemcpy(ret, &c2, sizeof(struct cell_cuda), hipMemcpyHostToDevice) );
  return ret;
}

void copy_to_host(struct cell_cuda *cuda_c, struct cell *c) {
  struct cell_cuda *h_cuda_cell = (struct cell_cuda*) malloc(sizeof(struct cell_cuda));

  hipMemcpy(h_cuda_cell, cuda_c, sizeof(struct cell_cuda), hipMemcpyDeviceToHost);
  int N = h_cuda_cell->part_count;
  int p_i = h_cuda_cell->first_part;

  struct particle_arrays h_cuda_part;
  allocate_parts(&h_cuda_part, N);
  copy_from_device_array(&h_cuda_part, p_i, N);

  for (int i=0; i<N; i++) {
    struct part *p = &c->parts[i];

    p->id = h_cuda_part.id[p_i];
    p->h = h_cuda_part.h[p_i];
    p->rho = h_cuda_part.rho[p_i];
    p->entropy = h_cuda_part.entropy[p_i];
    p->entropy_dt = h_cuda_part.entropy_dt[p_i];
    //density
    p->density.wcount = h_cuda_part.wcount[p_i];
    p->density.wcount_dh = h_cuda_part.wcount_dh[p_i];
    p->density.rho_dh = h_cuda_part.rho_dh[p_i];
    p->density.div_v = h_cuda_part.div_v[p_i];

    //force
    p->force.balsara = h_cuda_part.balsara[p_i];
    p->force.f = h_cuda_part.f[p_i];
    p->force.P_over_rho2 = h_cuda_part.P_over_rho2[p_i];
    p->force.soundspeed = h_cuda_part.soundspeed[p_i];
    p->force.v_sig = h_cuda_part.v_sig[p_i];
    p->force.h_dt = h_cuda_part.h_dt[p_i];
    
    p->x[0] = h_cuda_part.x_x[p_i];
    p->v[0] = h_cuda_part.v[p_i].x;
    p->a_hydro[0] = h_cuda_part.a_hydro[p_i].x;
    p->density.rot_v[0] = h_cuda_part.rot_v[p_i].x;
#if DIM > 1
    p->x[1] = h_cuda_part.x_y[p_i];
    p->v[1] = h_cuda_part.v[p_i].y;
    p->a_hydro[1] = h_cuda_part.a_hydro[p_i].y;
    p->density.rot_v[1] = h_cuda_part.rot_v[p_i].y;
#if DIM > 2
    p->x[2] = h_cuda_part.x_z[p_i];
    p->v[2] = h_cuda_part.v[p_i].z;
    p->a_hydro[2] = h_cuda_part.a_hydro[p_i].z;
    p->density.rot_v[2] = h_cuda_part.rot_v[p_i].z;
#endif
#endif

    for (int j=0; j<NBRE_DIR; j++)
      c->sort[j][i] = h_cuda_part.sort[j][i];

    p_i++;
  }
}


/* n is both particles per axis and box size:
 * particles are generated on a mesh with unit spacing
 */
struct cell *make_cell(size_t n, double *offset, double size, double h,
                       double density, unsigned long long *partId,
                       double pert) {
  const size_t count = n * n * n;
  const double volume = size * size * size;
  struct cell *cell = (struct cell*)malloc(sizeof(struct cell));

  bzero(cell, sizeof(struct cell));

  if (posix_memalign((void **)&cell->parts, part_align,
                     count * sizeof(struct part)) != 0) {
    error("couldn't allocate particles, no. of particles: %d", (int)count);
  }
  bzero(cell->parts, count * sizeof(struct part));

  /* Construct the parts */
  struct part *part = cell->parts;
  for (size_t x = 0; x < n; ++x) {
    for (size_t y = 0; y < n; ++y) {
      for (size_t z = 0; z < n; ++z) {
        part->x[0] = offset[0] +
	  size * (x + 0.5 + random_uniform(-0.5, 0.5) * pert) / (float)n;
        part->x[1] = offset[1] +
	  size * (y + 0.5 + random_uniform(-0.5, 0.5) * pert) / (float)n;
        part->x[2] = offset[2] +
	  size * (z + 0.5 + random_uniform(-0.5, 0.5) * pert) / (float)n;

	part->v[0] = random_uniform(-0.05, 0.05);
        part->v[1] = random_uniform(-0.05, 0.05);
        part->v[2] = random_uniform(-0.05, 0.05);
        part->h = size * h / (float)n;
        part->id = ++(*partId);
        part->mass = density * volume / count;
        part->time_bin = (int) random_uniform(1, 500);

	part->density.wcount = 0.;
	part->density.wcount_dh = 0.;
	part->density.rho_dh = 0.;
	part->density.rot_v[0] = 0.;
	part->density.rot_v[1] = 0.;
	part->density.rot_v[2] = 0.;
	part->density.div_v = 0.;

	part->force.balsara = 0.;
	part->force.f = 0.;
	part->force.P_over_rho2 = 0.;
	part->force.soundspeed = 0.;
	part->force.v_sig = 0.;
	part->force.h_dt = 0.;

        ++part;
      }
    }
  }

  /* Cell properties */
  cell->split = 0;
  cell->h_max = h;
  cell->count = count;
  cell->dx_max_part = 0.;
  cell->dx_max_sort = 0.;
  cell->width[0] = n;
  cell->width[1] = n;
  cell->width[2] = n;
  cell->loc[0] = offset[0];
  cell->loc[1] = offset[1];
  cell->loc[2] = offset[2];

  cell->ti_old_part = 8;
  cell->ti_end_min = 8;
  cell->ti_end_max = 8;

  //shuffle_particles(cell->parts, cell->count);

  return cell;
}

void clean_up(struct cell *ci) {
  free(ci->parts);
  //for (int k = 0; k < 13; k++)
  //if (ci->sort[k] != NULL) free(ci->sort[k]);
  free(ci);
}

/**
 * @brief Initializes all particles field to be ready for a density calculation
 */
void zero_particle_fields(struct cell *c) {
  for (int pid = 0; pid < c->count; pid++) {
    struct part* p = &c->parts[pid];
    p->rho = 0.f;

    // density
    p->density.wcount = 0.f;
    p->density.wcount_dh = 0.f;
    p->density.rho_dh = 0.f;
    p->density.div_v = 0.f;
    p->density.rot_v[0] = 0.f;
    p->density.rot_v[1] = 0.f;
    p->density.rot_v[2] = 0.f;

    // force
    p->force.balsara = 0.5f;
    p->force.f = 0.9f;
    p->force.P_over_rho2 = 1.2f;
    p->force.soundspeed = 0.f;
    p->force.v_sig = 0.f;
    p->force.h_dt = 0.f;
  }
}

/**
 * @brief Dump all the particles to a file
 */
void dump_particle_fields(char *fileName, struct cell *ci, struct cell *cj) {
  FILE *file = fopen(fileName, "w");

  /* Write header */
#ifdef FULL_COMP
  fprintf(file,
          "# %4s %10s %10s %10s %10s %10s %10s %13s %13s %13s %13s %13s "
          "%13s %13s %13s %13s %13s %13s %13s %13s\n",
          "ID", "pos_x", "pos_y", "pos_z", "v_x", "v_y", "v_z", "rho", "rho_dh",
          "wcount", "wcount_dh", "div_v", "curl_vx", "curl_vy", "curl_vz",
	  "a_hydro.x","a_hydro.y","a_hydro.z","h_dt","entropy_dt");
#else
  fprintf(file,
          "# %4s %10s %10s %10s %10s %10s %10s %13s\n",
          "ID", "pos_x", "pos_y", "pos_z", "v_x", "v_y", "v_z", "rho");
#endif

  fprintf(file, "# ci --------------------------------------------\n");

  for (int pid = 0; pid < ci->count; pid++) {
#ifdef FULL_COMP
    fprintf(file,
            "%6llu %10f %10f %10f %10f %10f %13e %13e %13e %13e %13e "
            "%13e %13e %13e %13e %13e %13e %13e %13e %13e\n",
            ci->parts[pid].id, ci->parts[pid].x[0], ci->parts[pid].x[1],
            ci->parts[pid].x[2], ci->parts[pid].v[0], ci->parts[pid].v[1],
            ci->parts[pid].v[2], ci->parts[pid].rho,
            ci->parts[pid].density.rho_dh,
            ci->parts[pid].density.wcount, ci->parts[pid].density.wcount_dh,
            ci->parts[pid].density.div_v, ci->parts[pid].density.rot_v[0],
            ci->parts[pid].density.rot_v[1], ci->parts[pid].density.rot_v[2],
            ci->parts[pid].a_hydro[0], ci->parts[pid].a_hydro[1], ci->parts[pid].a_hydro[2],
            ci->parts[pid].force.h_dt, ci->parts[pid].entropy_dt
            );
#else
    fprintf(file,
            "%6llu %10f %10f %10f %10f %10f %13e %13e\n",
            ci->parts[pid].id, ci->parts[pid].x[0], ci->parts[pid].x[1],
            ci->parts[pid].x[2], ci->parts[pid].v[0], ci->parts[pid].v[1],
            ci->parts[pid].v[2], ci->parts[pid].rho
            );
    
#endif
  }

  fprintf(file, "# cj --------------------------------------------\n");

  for (int pjd = 0; pjd < cj->count; pjd++) {
#ifdef FULL_COMP
    fprintf(file,
            "%6llu %10f %10f %10f %10f %10f %13e %13e %13e %13e %13e "
            "%13e %13e %13e %13e %13e %13e %13e %13e %13e\n",
            cj->parts[pjd].id, cj->parts[pjd].x[0], cj->parts[pjd].x[1],
            cj->parts[pjd].x[2], cj->parts[pjd].v[0], cj->parts[pjd].v[1],
            cj->parts[pjd].v[2], cj->parts[pjd].rho,
            cj->parts[pjd].density.rho_dh,
            cj->parts[pjd].density.wcount, cj->parts[pjd].density.wcount_dh,
            cj->parts[pjd].density.div_v, cj->parts[pjd].density.rot_v[0],
            cj->parts[pjd].density.rot_v[1], cj->parts[pjd].density.rot_v[2],
            cj->parts[pjd].a_hydro[0], cj->parts[pjd].a_hydro[1], cj->parts[pjd].a_hydro[2],
            cj->parts[pjd].force.h_dt, cj->parts[pjd].entropy_dt
            );
#else
    fprintf(file,
            "%6llu %10f %10f %10f %10f %10f %13e %13e\n",
            cj->parts[pjd].id, cj->parts[pjd].x[0], cj->parts[pjd].x[1],
            cj->parts[pjd].x[2], cj->parts[pjd].v[0], cj->parts[pjd].v[1],
            cj->parts[pjd].v[2], cj->parts[pjd].rho
            );
#endif
  }
  fclose(file);
}


int main(int argc, char *argv[]) {


  float host_kernel_coeffs[(kernel_degree + 1) * (kernel_ivals + 1)];
  for (int a = 0; a < (kernel_degree + 1) * (kernel_ivals + 1); a++) {
    host_kernel_coeffs[a] = kernel_coeffs[a];
  }
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
    cuda_kernel_coeffs), &host_kernel_coeffs,
    sizeof(float) * (kernel_degree + 1) * (kernel_ivals + 1)));

  size_t particles = 0, runs = 0, volume, type = 0;
  double offset[3] = {0, 0, 0}, h = 1.1255, size = 1., rho = 1.;
  double perturbation = 0.;
  struct cell *ci, *cj;

  char c;
  static unsigned long long partId = 0;
  char outputFileNameExtension[200] = "";
  char outputFileName[200] = "";

  //  ticks tic, toc, time;

  /* Initialize CPU frequency, this also starts time. */
  //unsigned long long cpufreq = 0;
  //clocks_set_cpufreq(cpufreq);

  /* Choke on FP-exceptions */
  //feenableexcept(FE_DIVBYZERO | FE_INVALID | FE_OVERFLOW);

  srand(0);

  while ((c = getopt(argc, argv, "h:p:r:t:d:f:")) != -1) {
    switch (c) {
      case 'h':
        sscanf(optarg, "%lf", &h);
        break;
      case 'p':
        sscanf(optarg, "%zu", &particles);
        break;
      case 'r':
        sscanf(optarg, "%zu", &runs);
        break;
      case 't':
        sscanf(optarg, "%zu", &type);
        break;
      case 'd':
        sscanf(optarg, "%lf", &perturbation);
        break;
      case 'f':
        strcpy(outputFileNameExtension, optarg);
        break;
      case '?':
        error("Unknown option.");
        break;
    }
  }

  if (h < 0 || particles == 0 || runs == 0 || type > 2) {
    printf(
        "\nUsage: %s -p PARTICLES_PER_AXIS -r NUMBER_OF_RUNS [OPTIONS...]\n"
        "\nGenerates a cell pair, filled with particles on a Cartesian grid."
        "\nThese are then interacted using runner_dopair1_density."
        "\n\nOptions:"
        "\n-t TYPE=0          - cells share face (0), edge (1) or corner (2)"
        "\n-h DISTANCE=1.1255 - smoothing length"
        "\n-d pert            - perturbation to apply to the particles [0,1["
        "\n-f fileName        - part of the file name used to save the dumps\n",
        argv[0]);
    exit(1);
  }

  volume = particles * particles * particles;
  message("particles: %zu B\npositions: 0 B", 2 * volume * sizeof(struct part));

  ci = make_cell(particles, offset, size, h, rho, &partId, perturbation);
  for (size_t i = 0; i < type + 1; ++i) offset[i] = 1.;
  cj = make_cell(particles, offset, size, h, rho, &partId, perturbation);

  sprintf(outputFileName, "swift_gpu_init_%s.dat", outputFileNameExtension);
  dump_particle_fields(outputFileName, ci, cj);

  allocate_device_parts(2*volume);

  struct cell_cuda *cuda_ci;
  struct cell_cuda *cuda_cj;
  
  //  time = 0;
  for (size_t i = 0; i < runs; ++i) {
    /* Zero the fields */
    zero_particle_fields(ci);
    zero_particle_fields(cj);

    do_sort(ci);
    do_sort(cj);
    
    cuda_ci = copy_from_host(ci, 0);
    cuda_cj = copy_from_host(cj, ci->count);
    //exit(1);
    //tic = getticks();

    /* Run the test */
    do_test_pair_density<<<1,CUDA_THREADS>>>(cuda_ci, cuda_cj);
    //do_test_pair_density_sorted<<<1,CUDA_THREADS>>>(cuda_ci, cuda_cj, 0, 1);
    cudaErrCheck( hipPeekAtLastError() );
    cudaErrCheck( hipDeviceSynchronize() );
    
    do_test_self_density<<<1,CUDA_THREADS>>>(cuda_ci);
    //do_test_self_density_symmetric<<<1,CUDA_THREADS>>>(cuda_ci);
    cudaErrCheck( hipPeekAtLastError() );
    cudaErrCheck( hipDeviceSynchronize() );

    do_test_pair_force<<<1,CUDA_THREADS>>>(cuda_ci, cuda_cj);
    cudaErrCheck( hipPeekAtLastError() );
    cudaErrCheck( hipDeviceSynchronize() );
    
    do_test_self_force<<<1,CUDA_THREADS>>>(cuda_ci);
    cudaErrCheck( hipPeekAtLastError() );
    cudaErrCheck( hipDeviceSynchronize() );
    
    //toc = getticks();
    //time += toc - tic;

    copy_to_host(cuda_ci, ci);
    copy_to_host(cuda_cj, cj);
    /* Dump if necessary */
    if (i % 50 == 0) {
      sprintf(outputFileName, "swift_gpu_%s.dat", outputFileNameExtension);
      dump_particle_fields(outputFileName, ci, cj);
    }
  }

  printf("rho %g\n", ci->parts[0].rho);
  /* Output timing */
//  message("SWIFT calculation took       %lli ticks.", time / runs);

  /* Now perform a brute-force version for accuracy tests */

  /* Zero the fields */
  //zero_particle_fields(ci);
  //zero_particle_fields(cj);

//  tic = getticks();

  /* Run the brute-force test */
  //pairs_all_density(&runner, ci, cj);

//  toc = getticks();

  /* Dump */
  sprintf(outputFileName, "swift_gpu_end_%s.dat", outputFileNameExtension);
  dump_particle_fields(outputFileName, ci, cj);

  /* Output timing */
//  message("Brute force calculation took %lli ticks.", toc - tic);

  /* Clean things to make the sanitizer happy ... */
  clean_up(ci);
  clean_up(cj);

  return 0;
}
