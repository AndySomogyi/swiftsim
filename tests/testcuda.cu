#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of SWIFT.
 * Copyright (C) 2015 Matthieu Schaller (matthieu.schaller@durham.ac.uk).
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ******************************************************************************/

// NEED TO USE
//   gpuErrchk( hipPeekAtLastError() );
//  gpuErrchk( hipDeviceSynchronize() );

#ifndef static
#define static
#endif
#ifndef restrict
#define restrict __restrict__
#endif
#define FULL_COMP
extern "C" {
#include "testcuda.h"
/* Some standard headers. */
#include <fenv.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

}

#include <hip/hip_runtime.h>


/* Host function to check cuda functions don't return errors */
__host__ inline void cudaErrCheck(hipError_t status) {
  if (status != hipSuccess) {
    printf("%s\n", hipGetErrorString(status));
  }
}

/* Task function to perform a self force task. No symmetry */
__device__ void doself_force(struct cell_cuda *ci) {
  /* Is the cell active? */
  if (!cuda_cell_is_active(ci)) return;

  const int count_i = ci->part_count;
  int part_i = ci->first_part;

  float3 a_hydro;
  float h_dt, v_sig_stor, entropy_dt;

  /* Loop over the particles */
  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {

    const float hi = cuda_parts.h[pid];
    if (!cuda_part_is_active(pid)) continue;
    /* Reset the values. */
    a_hydro.x = 0.0f;
    a_hydro.y = 0.0f;
    a_hydro.z = 0.0f;
    h_dt = 0.0f;
    v_sig_stor = cuda_parts.v_sig[pid];
    entropy_dt = 0.0f;

    double pix[3];
    pix[0] = cuda_parts.x_x[pid];
    pix[1] = cuda_parts.x_y[pid];
    pix[2] = cuda_parts.x_z[pid];

    const float hig2 = hi * hi * kernel_gamma2;

    /* Loop over the particles in cj. */
    for (int pjd = part_i; pjd < part_i + count_i; pjd++) {

      if (pid == pjd) continue;
      /* Compute the pairwise distance. */
      float r2 = 0.0f;
      float dx[3];
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];
      const float hj = cuda_parts.h[pjd];
      if (r2 < hig2 || r2 < hj * hj * kernel_gamma2) {
        float wi, wj, wi_dx, wj_dx;
        const float fac_mu = 1.f;

        const float r = sqrtf(r2);
        const float r_inv = 1.0f / r;

        /* Load some data.*/
        const float mj = cuda_parts.mass[pjd];
        const float rhoi = cuda_parts.rho[pid];
        const float rhoj = cuda_parts.rho[pjd];

        /* Get the kernel for hi. */
        const float hi_inv = 1.0f / hi;
        const float hid_inv = cuda_pow_dimension_plus_one(hi_inv);
        const float ui = r * hi_inv;
        cuda_kernel_deval(ui, &wi, &wi_dx);
        const float wi_dr = hid_inv * wi_dx;

        /* Get the kernel for hj. */
        const float hj_inv = 1.0f / hj;
        const float hjd_inv = cuda_pow_dimension_plus_one(hj_inv);
        const float xj = r * hj_inv;
        cuda_kernel_deval(xj, &wj, &wj_dx);
        const float wj_dr = hjd_inv * wj_dx;

        /* Compute h-gradient terms */
        const float f_i = cuda_parts.f[pid];
        const float f_j = cuda_parts.f[pjd];

        /* Compute pressure terms */
        const float P_over_rho2_i = cuda_parts.P_over_rho2[pid];
        const float P_over_rho2_j = cuda_parts.P_over_rho2[pjd];

        /* Compute sound speeds*/
        const float ci = cuda_parts.soundspeed[pid];
        const float cj = cuda_parts.soundspeed[pjd];

        /* Compute dv dot r. */
        const float dvdr = (cuda_parts.v[pid].x - cuda_parts.v[pjd].x) * dx[0] +
                           (cuda_parts.v[pid].y - cuda_parts.v[pjd].y) * dx[1] +
                           (cuda_parts.v[pid].z - cuda_parts.v[pjd].z) * dx[2];

        /* Balsara term */
        const float balsara_i = cuda_parts.balsara[pid];
        const float balsara_j = cuda_parts.balsara[pjd];

        /* Are the particles moving towards each other? */
        const float omega_ij = (dvdr < 0.f) ? dvdr : 0.f;
        const float mu_ij = fac_mu * r_inv * omega_ij;

        /* Signal velocity */
        const float v_sig = ci + cj - 3.f * mu_ij;
        const float rho_ij = 0.5f * (rhoi + rhoj);
        const float visc = -0.25 * const_viscosity_alpha * v_sig * mu_ij *
                           (balsara_i + balsara_j) / rho_ij;

        /* Now convolce with the kernel */
        const float visc_term = 0.5f * visc * (wi_dr + wj_dr) * r_inv;
        const float sph_term =
            (f_i * P_over_rho2_i * wi_dr + f_j * P_over_rho2_j * wj_dr) * r_inv;

        /* Compute the acceleration */
        const float acc = visc_term + sph_term;

        /* Compute the force */
        a_hydro.x -= mj * acc * dx[0];
        a_hydro.y -= mj * acc * dx[1];
        a_hydro.z -= mj * acc * dx[2];


        /* Get the time derivative for h. */
        h_dt -= mj * dvdr * r_inv / rhoj * wi_dr;

        /* Update the signal velocity. */
        v_sig_stor = (v_sig_stor > v_sig) ? v_sig_stor : v_sig;

        /* Change in entropy */
        entropy_dt += mj * visc_term * dvdr;
      }

    }  // Inner loop

    /* Flush to global stores.*/
    atomicAdd(&cuda_parts.a_hydro[pid].x, a_hydro.x);
    atomicAdd(&cuda_parts.a_hydro[pid].y, a_hydro.y);
    atomicAdd(&cuda_parts.a_hydro[pid].z, a_hydro.z);
    atomicAdd(&cuda_parts.h_dt[pid], h_dt);
    atomicAdd(&cuda_parts.entropy_dt[pid], entropy_dt);

    /* Update the signal velocity */
    float global_vsig = cuda_parts.v_sig[pid];
    int *address_as_int = (int *)&cuda_parts.v_sig[pid];
    int old = *address_as_int;
    int assumed;
    do {
      global_vsig = cuda_parts.v_sig[pid];  // Scary line.
      assumed = old;
      if (v_sig_stor > global_vsig)
        old = atomicCAS(address_as_int, assumed, __float_as_int(v_sig_stor));
    } while (assumed != old && v_sig_stor > global_vsig);

  }  // Outer loop
}

/* Task function to execute a self-density task. */
__device__ void doself_density(struct cell_cuda *ci) {

  /* Is the cell active? */
  if (!cuda_cell_is_active(ci)) {
    printf(
        "Cell isn't active..., ti_end_min=%i, ti_current=%i, "
        "max_active_bin=%i\n",
        ci->ti_end_min, ti_current, max_active_bin);
    return;
  }

  const int count_i = ci->part_count;
  int part_i = ci->first_part;
  float rho, rho_dh, div_v, wcount, wcount_dh;
  float3 rot_v;

  for (int pid = part_i + threadIdx.x; pid < part_i + count_i;
       pid += blockDim.x) {
    double pix[3];
    pix[0] = cuda_parts.x_x[pid];
    pix[1] = cuda_parts.x_y[pid];
    pix[2] = cuda_parts.x_z[pid];
    const float hi = cuda_parts.h[pid];
    const float hig2 = hi * hi * kernel_gamma2;

    /* Reset local values. */
    rho = 0.0f;
    rho_dh = 0.0f;
    div_v = 0.0f;
    wcount = 0.0f;
    wcount_dh = 0.0f;
    rot_v.x = 0.0f;
    rot_v.y = 0.0f;
    rot_v.z = 0.0f;

    /* If the particle isn't active skip it. */
    if (!cuda_part_is_active(pid)) {
      continue;
    }

    /* Search for the neighbours! */
    for (int pjd = part_i; pjd < part_i + count_i; pjd++) {
      /* Particles don't interact with themselves */
      if (pid == pjd) continue;
      float dx[3], r2 = 0.0f;
      dx[0] = pix[0] - cuda_parts.x_x[pjd];
      r2 += dx[0] * dx[0];
      dx[1] = pix[1] - cuda_parts.x_y[pjd];
      r2 += dx[1] * dx[1];
      dx[2] = pix[2] - cuda_parts.x_z[pjd];
      r2 += dx[2] * dx[2];

      /* If in range then interact. */
      if (r2 < hig2) {
        float w, dw_dx;
        float dv[3], curlvr[3];
        /* Load mass on particle pj. */
        const float mj = cuda_parts.mass[pjd];

        /* Get r and 1/r */
        const float r = sqrtf(r2);
        const float ri = 1.0f / r;

        /* Compute the kernel function */
        const float hi_inv = 1.0f / hi;
        const float ui = r * hi_inv;

        cuda_kernel_deval(ui, &w, &dw_dx);
        /* Compute contribution to the density. */
        rho += mj * w;
        rho_dh -= mj * (hydro_dimension * w + ui * dw_dx);

        /* Compute contribution to the number of neighbours */
        wcount += w;
        wcount_dh -= (hydro_dimension * w + ui * dw_dx);

        const float fac = mj * dw_dx * ri;

        /* Compute dv dot r */
        float3 piv, pjv;
        piv = cuda_parts.v[pid];
        pjv = cuda_parts.v[pjd];
        dv[0] = piv.x - pjv.x;
        dv[1] = piv.y - pjv.y;
        dv[2] = piv.z - pjv.z;
        const float dvdr = dv[0] * dx[0] + dv[1] * dx[1] + dv[2] * dx[2];

        div_v -= fac * dvdr;

        curlvr[0] = dv[1] * dx[2] - dv[2] * dx[1];
        curlvr[1] = dv[2] * dx[0] - dv[0] * dx[2];
        curlvr[2] = dv[0] * dx[1] - dv[1] * dx[0];

        rot_v.x += fac * curlvr[0];
        rot_v.y += fac * curlvr[1];
        rot_v.z += fac * curlvr[2];
      }
    }  // Loop over cj.
    /* Write data for particle pid back to global stores. */
    atomicAdd(&cuda_parts.rho[pid], rho);
    atomicAdd(&cuda_parts.rho_dh[pid], rho_dh);
    atomicAdd(&cuda_parts.wcount[pid], wcount);
    atomicAdd(&cuda_parts.wcount_dh[pid], wcount_dh);
    atomicAdd(&cuda_parts.div_v[pid], div_v);
    atomicAdd(&cuda_parts.rot_v[pid].x, rot_v.x);
    atomicAdd(&cuda_parts.rot_v[pid].y, rot_v.y);
    atomicAdd(&cuda_parts.rot_v[pid].z, rot_v.z);
  }
}


__global__ void do_test(struct cell_cuda *ci, struct cell_cuda *cj) {

  doself_density(ci);
  doself_force(ci);
  //dopair_density(ci, cj);
}

void allocate_parts(struct particle_arrays *p, size_t num_part) { 
  p->id = (long long int*) malloc (sizeof(long long int) * num_part);
  p->x_x = (double*) malloc(sizeof(double) * num_part);
  p->x_y = (double*) malloc(sizeof(double) * num_part);
  p->x_z = (double*) malloc(sizeof(double) * num_part);
  p->v = (float3*) malloc(sizeof(float3) * num_part);
  p->a_hydro = (float3*) malloc(sizeof(float3) * num_part);
  p->h = (float*) malloc(sizeof(float) * num_part);
  p->mass = (float*) malloc(sizeof(float) * num_part);
  p->rho = (float*) malloc(sizeof(float) * num_part);
  p->entropy = (float*) malloc(sizeof(float) * num_part);
  p->entropy_dt = (float*) malloc(sizeof(float) * num_part);

  // density
  p->wcount = (float*) malloc(sizeof(float) * num_part);
  p->wcount_dh = (float*) malloc(sizeof(float) * num_part);
  p->rho_dh = (float*) malloc(sizeof(float) * num_part);
  p->rot_v = (float3*) malloc(sizeof(float3) * num_part);
  p->div_v = (float*) malloc(sizeof(float) * num_part);

  // force
  p->balsara = (float*) malloc(sizeof(float) * num_part);
  p->f = (float*) malloc(sizeof(float) * num_part);
  p->P_over_rho2 = (float*) malloc(sizeof(float) * num_part);
  p->soundspeed = (float*) malloc(sizeof(float) * num_part);
  p->v_sig = (float*) malloc(sizeof(float) * num_part);
  p->h_dt = (float*) malloc(sizeof(float) * num_part);
  
  p->time_bin = (timebin_t*) malloc(sizeof(timebin_t) * num_part);
}

void allocate_device_parts(size_t num_part) {
  struct particle_arrays c_parts;
  
  cudaErrCheck(hipMalloc(&c_parts.id, sizeof(long long int) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.x_x, sizeof(double) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.x_y, sizeof(double) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.x_z, sizeof(double) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.v, sizeof(float3) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.a_hydro, sizeof(float3) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.h, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.mass, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.rho, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.entropy, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.entropy_dt, sizeof(float) * num_part));

  // density
  cudaErrCheck(hipMalloc(&c_parts.wcount, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.wcount_dh, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.rho_dh, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.rot_v, sizeof(float3) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.div_v, sizeof(float) * num_part));

  // force
  cudaErrCheck(hipMalloc(&c_parts.balsara, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.f, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.P_over_rho2, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.soundspeed, sizeof(float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.v_sig, sizeof(volatile float) * num_part));
  cudaErrCheck(hipMalloc(&c_parts.h_dt, sizeof(float) * num_part));

  cudaErrCheck(hipMalloc(&c_parts.time_bin, sizeof(timebin_t) * num_part));

  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(cuda_parts), &c_parts, sizeof(struct particle_arrays)));
}

void free_parts(struct particle_arrays p) {
  free(p.id); p.id = NULL;
  free(p.x_x); p.x_x = NULL;
  free(p.x_y); p.x_y = NULL;
  free(p.x_z); p.x_z = NULL;
  free(p.v); p.v = NULL;
  free(p.a_hydro); p.a_hydro = NULL;
  free(p.h); p.h = NULL;
  free(p.mass); p.mass = NULL;
  free(p.rho); p.rho = NULL;
  free(p.entropy); p.entropy = NULL;
  free(p.entropy_dt); p.entropy_dt = NULL;

  // density
  free(p.wcount); p.wcount = NULL;
  free(p.wcount_dh); p.wcount_dh = NULL;
  free(p.rho_dh); p.rho_dh = NULL;
  free(p.rot_v); p.rot_v = NULL;
  free(p.div_v); p.div_v = NULL;

  // force
  free(p.balsara); p.balsara = NULL;
  free(p.f); p.f = NULL;
  free(p.P_over_rho2); p.P_over_rho2 = NULL;
  free(p.soundspeed); p.soundspeed = NULL;
  free((void *)p.v_sig); p.v_sig = NULL;
  free(p.h_dt); p.h_dt = NULL;
  
  free(p.time_bin); p.time_bin = NULL;
}

void free_device_parts() {
  struct particle_arrays parts;
  cudaErrCheck(hipMemcpyFromSymbol(&parts, HIP_SYMBOL(cuda_parts),
				    sizeof(struct particle_arrays)));

  cudaErrCheck(hipFree(parts.id));
  cudaErrCheck(hipFree(parts.x_x));
  cudaErrCheck(hipFree(parts.x_y));
  cudaErrCheck(hipFree(parts.x_z));
  cudaErrCheck(hipFree(parts.v));
  cudaErrCheck(hipFree(parts.a_hydro));
  cudaErrCheck(hipFree(parts.h));
  cudaErrCheck(hipFree(parts.mass));
  cudaErrCheck(hipFree(parts.rho));
  cudaErrCheck(hipFree(parts.entropy));
  cudaErrCheck(hipFree(parts.entropy_dt));

  // density
  cudaErrCheck(hipFree(parts.wcount));
  cudaErrCheck(hipFree(parts.wcount_dh));
  cudaErrCheck(hipFree(parts.rho_dh));
  cudaErrCheck(hipFree(parts.rot_v));
  cudaErrCheck(hipFree(parts.div_v));

  // force
  cudaErrCheck(hipFree(parts.balsara));
  cudaErrCheck(hipFree(parts.f));
  cudaErrCheck(hipFree(parts.P_over_rho2));
  cudaErrCheck(hipFree(parts.soundspeed));
  cudaErrCheck(hipFree((void*)parts.v_sig));
  cudaErrCheck(hipFree(parts.h_dt));

  
  cudaErrCheck(hipFree(parts.time_bin));
}


void copy_to_device_array(struct cell *ci, int offset) {

  struct particle_arrays h_p;
    
  int num_part = ci->count;
  
  allocate_parts(&h_p, num_part);
  
  //copy particles data
  for(int i=0;i<num_part;i++) {
    int p_i = i + offset;
    struct part p = ci->parts[i];
    h_p.id[p_i] = p.id;
    h_p.x_x[p_i] = p.x[0];
    h_p.x_y[p_i] = p.x[1];
    h_p.x_z[p_i] = p.x[2];
    h_p.v[p_i].x = p.v[0];
    h_p.v[p_i].y = p.v[1];
    h_p.v[p_i].z = p.v[2];
    h_p.a_hydro[p_i].x = p.a_hydro[0];
    h_p.a_hydro[p_i].y = p.a_hydro[1];
    h_p.a_hydro[p_i].z = p.a_hydro[2];
    h_p.h[p_i] = p.h;
    h_p.mass[p_i] = p.mass;
    h_p.rho[p_i] = p.rho;
    h_p.entropy[p_i] = p.entropy;
    h_p.entropy_dt[p_i] = p.entropy_dt;

    // density
    h_p.wcount[p_i] = p.density.wcount;
    h_p.wcount_dh[p_i] = p.density.wcount_dh;
    h_p.rho_dh[p_i] = p.density.rho_dh;
    h_p.rot_v[p_i].x = p.density.rot_v[0];
    h_p.rot_v[p_i].y = p.density.rot_v[1];
    h_p.rot_v[p_i].z = p.density.rot_v[2];
    h_p.div_v[p_i] = p.density.div_v;

    // force
    h_p.balsara[p_i] = p.force.balsara;
    h_p.f[p_i] = p.force.f;
    h_p.P_over_rho2[p_i] = p.force.P_over_rho2;
    h_p.soundspeed[p_i] = p.force.soundspeed;
    h_p.v_sig[p_i] = p.force.v_sig;
    h_p.h_dt[p_i] = p.force.h_dt;
    
    h_p.time_bin[p_i] = p.time_bin;
  }

  struct particle_arrays c_parts;
  cudaErrCheck(hipMemcpyFromSymbol(&c_parts, HIP_SYMBOL(cuda_parts),
                                    sizeof(struct particle_arrays)));

  void *p_data = c_parts.id + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.id, sizeof(long long int) * num_part,hipMemcpyHostToDevice));

  p_data = c_parts.x_x + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.x_x, sizeof(double) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.x_y + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.x_y, sizeof(double) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.x_z + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.x_z, sizeof(double) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.v + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.v, sizeof(float3) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.a_hydro + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.a_hydro,sizeof(float3) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.h + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.h,sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.mass + offset;
  cudaErrCheck(hipMemcpy(p_data,h_p.mass, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.rho + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.rho,sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.entropy + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.entropy,sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.entropy_dt + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.entropy_dt,sizeof(float) * num_part, hipMemcpyHostToDevice));

  // density
  p_data = c_parts.wcount + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.wcount, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.wcount_dh + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.wcount_dh, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.rho_dh + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.rho_dh, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.rot_v + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.rot_v, sizeof(float3) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.div_v + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.div_v, sizeof(float) * num_part, hipMemcpyHostToDevice));

  // force
  p_data = c_parts.balsara + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.balsara, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.f + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.f, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.P_over_rho2 + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.P_over_rho2, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.soundspeed + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.soundspeed, sizeof(float) * num_part, hipMemcpyHostToDevice));

  p_data = (void*)(c_parts.v_sig + offset);
  cudaErrCheck(hipMemcpy(p_data, (void*)h_p.v_sig, sizeof(volatile float) * num_part, hipMemcpyHostToDevice));

  p_data = c_parts.h_dt + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.h_dt, sizeof(float) * num_part, hipMemcpyHostToDevice));

  
  p_data = c_parts.time_bin + offset;
  cudaErrCheck(hipMemcpy(p_data, h_p.time_bin,sizeof(timebin_t) * num_part, hipMemcpyHostToDevice));

  free_parts(h_p);
}

void copy_from_device_array(struct particle_arrays *h_p, int offset, size_t num_part) {

  struct particle_arrays c_parts;
  cudaErrCheck(hipMemcpyFromSymbol(&c_parts, HIP_SYMBOL(cuda_parts),
                                    sizeof(struct particle_arrays)));

  void *p_data = c_parts.id + offset;
  cudaErrCheck(hipMemcpy(h_p->id, p_data, sizeof(long long int) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.x_x + offset;
  cudaErrCheck(hipMemcpy(h_p->x_x, p_data, sizeof(double) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.x_y + offset;
  cudaErrCheck(hipMemcpy(h_p->x_y, p_data, sizeof(double) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.x_z + offset;
  cudaErrCheck(hipMemcpy(h_p->x_z, p_data, sizeof(double) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.v + offset;
  cudaErrCheck(hipMemcpy(h_p->v, p_data, sizeof(float3) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.a_hydro + offset;
  cudaErrCheck(hipMemcpy(h_p->a_hydro, p_data, sizeof(float3) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.h + offset;
  cudaErrCheck(hipMemcpy(h_p->h, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.mass + offset;
  cudaErrCheck(hipMemcpy(h_p->mass, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.rho + offset;
  cudaErrCheck(hipMemcpy(h_p->rho, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.entropy + offset;
  cudaErrCheck(hipMemcpy(h_p->entropy, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  p_data = c_parts.entropy_dt + offset;
  cudaErrCheck(hipMemcpy(h_p->entropy_dt, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));

  // density
  p_data = c_parts.wcount + offset;
  cudaErrCheck(hipMemcpy(h_p->wcount, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.wcount_dh + offset;
  cudaErrCheck(hipMemcpy(h_p->wcount_dh, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.rho_dh + offset;
  cudaErrCheck(hipMemcpy(h_p->rho_dh, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.rot_v + offset;
  cudaErrCheck(hipMemcpy(h_p->rot_v, p_data, sizeof(float3) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.div_v + offset;
  cudaErrCheck(hipMemcpy(h_p->div_v, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  

  // force
  p_data = c_parts.balsara + offset;
  cudaErrCheck(hipMemcpy(h_p->balsara, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.f + offset;
  cudaErrCheck(hipMemcpy(h_p->f, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.P_over_rho2 + offset;
  cudaErrCheck(hipMemcpy(h_p->P_over_rho2, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.soundspeed + offset;
  cudaErrCheck(hipMemcpy(h_p->soundspeed, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = (void*)(c_parts.v_sig + offset);
  cudaErrCheck(hipMemcpy((void*)h_p->v_sig, p_data, sizeof(volatile float) * num_part, hipMemcpyDeviceToHost));
  
  p_data = c_parts.h_dt + offset;
  cudaErrCheck(hipMemcpy(h_p->h_dt, p_data, sizeof(float) * num_part, hipMemcpyDeviceToHost));
  

  p_data = c_parts.time_bin + offset;
  cudaErrCheck(hipMemcpy(h_p->time_bin, p_data, sizeof(timebin_t) * num_part, hipMemcpyDeviceToHost));
}

struct cell_cuda* copy_from_host(struct cell *ci, int offset) {
  copy_to_device_array(ci, offset);
  /* Set the host pointer. */
  struct cell_cuda c2;

  for (int k=0; k<DIM; k++) {
    c2.loc[k] = ci->loc[k];
    c2.width[k] = ci->width[k];
  
  }
  c2.h_max = ci->h_max;
  c2.first_part = offset;
  c2.part_count = ci->count;
  if (ci->parent != NULL) {
    c2.parent = ci->parent->cuda_ID;
  } else {
    c2.parent = -1;
  }
  if (ci->super != NULL) {
    c2.super = ci->super->cuda_ID;
  } else {
    c2.super = -1;
  }
  c2.ti_end_min = ci->ti_end_min;
  c2.ti_end_max = ci->ti_end_max;
  c2.dmin = ci->dmin;
  c2.nr_links = 0;
  c2.split = ci->split;

  struct cell_cuda *ret;
  cudaErrCheck( hipMalloc(&ret, sizeof(struct cell_cuda)) );
  cudaErrCheck( hipMemcpy(ret, &c2, sizeof(struct cell_cuda), hipMemcpyHostToDevice) );
  return ret;
}

void copy_to_host(struct cell_cuda *cuda_c, struct cell *c) {
  struct cell_cuda *h_cuda_cell = (struct cell_cuda*) malloc(sizeof(struct cell_cuda));

  hipMemcpy(h_cuda_cell, cuda_c, sizeof(struct cell_cuda), hipMemcpyDeviceToHost);
  int N = h_cuda_cell->part_count;
  int p_i = h_cuda_cell->first_part;

  struct particle_arrays h_cuda_part;
  allocate_parts(&h_cuda_part, N);
  copy_from_device_array(&h_cuda_part, p_i, N);

  for (int i=0; i<N; i++) {
    struct part *p = &c->parts[i];

    p->id = h_cuda_part.id[p_i];
    p->h = h_cuda_part.h[p_i];
    p->rho = h_cuda_part.rho[p_i];
    p->entropy = h_cuda_part.entropy[p_i];
    p->entropy_dt = h_cuda_part.entropy_dt[p_i];
    //density
    p->density.wcount = h_cuda_part.wcount[p_i];
    p->density.wcount_dh = h_cuda_part.wcount_dh[p_i];
    p->density.rho_dh = h_cuda_part.rho_dh[p_i];
    p->density.div_v = h_cuda_part.div_v[p_i];

    //force
    p->force.balsara = h_cuda_part.balsara[p_i];
    p->force.f = h_cuda_part.f[p_i];
    p->force.P_over_rho2 = h_cuda_part.P_over_rho2[p_i];
    p->force.soundspeed = h_cuda_part.soundspeed[p_i];
    p->force.v_sig = h_cuda_part.v_sig[p_i];
    p->force.h_dt = h_cuda_part.h_dt[p_i];
    
    p->x[0] = h_cuda_part.x_x[p_i];
    p->v[0] = h_cuda_part.v[p_i].x;
    p->a_hydro[0] = h_cuda_part.a_hydro[p_i].x;
    p->density.rot_v[0] = h_cuda_part.rot_v[p_i].x;
#if DIM > 1
    p->x[1] = h_cuda_part.x_y[p_i];
    p->v[1] = h_cuda_part.v[p_i].y;
    p->a_hydro[1] = h_cuda_part.a_hydro[p_i].y;
    p->density.rot_v[1] = h_cuda_part.rot_v[p_i].y;
#if DIM > 2
    p->x[2] = h_cuda_part.x_z[p_i];
    p->v[2] = h_cuda_part.v[p_i].z;
    p->a_hydro[2] = h_cuda_part.a_hydro[p_i].z;
    p->density.rot_v[2] = h_cuda_part.rot_v[p_i].z;
#endif
#endif

    p_i++;
  }
}


/* n is both particles per axis and box size:
 * particles are generated on a mesh with unit spacing
 */
struct cell *make_cell(size_t n, double *offset, double size, double h,
                       double density, unsigned long long *partId,
                       double pert) {
  const size_t count = n * n * n;
  const double volume = size * size * size;
  struct cell *cell = (struct cell*)malloc(sizeof(struct cell));

  bzero(cell, sizeof(struct cell));

  if (posix_memalign((void **)&cell->parts, part_align,
                     count * sizeof(struct part)) != 0) {
    error("couldn't allocate particles, no. of particles: %d", (int)count);
  }
  bzero(cell->parts, count * sizeof(struct part));

  /* Construct the parts */
  struct part *part = cell->parts;
  for (size_t x = 0; x < n; ++x) {
    for (size_t y = 0; y < n; ++y) {
      for (size_t z = 0; z < n; ++z) {
        part->x[0] = offset[0] +
	  size * (x + 0.5 + random_uniform(-0.5, 0.5) * pert) / (float)n;
        part->x[1] = offset[1] +
	  size * (y + 0.5 + random_uniform(-0.5, 0.5) * pert) / (float)n;
        part->x[2] = offset[2] +
	  size * (z + 0.5 + random_uniform(-0.5, 0.5) * pert) / (float)n;

	part->v[0] = random_uniform(-0.05, 0.05);
        part->v[1] = random_uniform(-0.05, 0.05);
        part->v[2] = random_uniform(-0.05, 0.05);
        part->h = size * h / (float)n;
        part->id = ++(*partId);
        part->mass = density * volume / count;
        part->time_bin = 1;

	part->density.wcount = 0.;
	part->density.wcount_dh = 0.;
	part->density.rho_dh = 0.;
	part->density.rot_v[0] = 0.;
	part->density.rot_v[1] = 0.;
	part->density.rot_v[2] = 0.;
	part->density.div_v = 0.;

	part->force.balsara = 0.;
	part->force.f = 0.;
	part->force.P_over_rho2 = 0.;
	part->force.soundspeed = 0.;
	part->force.v_sig = 0.;
	part->force.h_dt = 0.;

        ++part;
      }
    }
  }

  /* Cell properties */
  cell->split = 0;
  cell->h_max = h;
  cell->count = count;
  cell->dx_max_part = 0.;
  cell->dx_max_sort = 0.;
  cell->width[0] = n;
  cell->width[1] = n;
  cell->width[2] = n;
  cell->loc[0] = offset[0];
  cell->loc[1] = offset[1];
  cell->loc[2] = offset[2];

  cell->ti_old_part = 8;
  cell->ti_end_min = 8;
  cell->ti_end_max = 8;

  //shuffle_particles(cell->parts, cell->count);

  return cell;
}

void clean_up(struct cell *ci) {
  free(ci->parts);
  //for (int k = 0; k < 13; k++)
  //if (ci->sort[k] != NULL) free(ci->sort[k]);
  free(ci);
}

/**
 * @brief Initializes all particles field to be ready for a density calculation
 */
void zero_particle_fields(struct cell *c) {
  for (int pid = 0; pid < c->count; pid++) {
    struct part* p = &c->parts[pid];
    p->rho = 0.f;

    // density
    p->density.wcount = 0.f;
    p->density.wcount_dh = 0.f;
    p->density.rho_dh = 0.f;
    p->density.div_v = 0.f;
    p->density.rot_v[0] = 0.f;
    p->density.rot_v[1] = 0.f;
    p->density.rot_v[2] = 0.f;

    // force
    p->force.balsara = 0.5f;
    p->force.f = 0.9f;
    p->force.P_over_rho2 = 1.2f;
    p->force.soundspeed = 0.f;
    p->force.v_sig = 0.f;
    p->force.h_dt = 0.f;
  }
}

/**
 * @brief Dump all the particles to a file
 */
void dump_particle_fields(char *fileName, struct cell *ci, struct cell *cj) {
  FILE *file = fopen(fileName, "w");

  /* Write header */
#ifdef FULL_COMP
  fprintf(file,
          "# %4s %10s %10s %10s %10s %10s %10s %13s %13s %13s %13s %13s "
          "%13s %13s %13s %13s %13s %13s %13s %13s\n",
          "ID", "pos_x", "pos_y", "pos_z", "v_x", "v_y", "v_z", "rho", "rho_dh",
          "wcount", "wcount_dh", "div_v", "curl_vx", "curl_vy", "curl_vz",
	  "a_hydro.x","a_hydro.y","a_hydro.z","h_dt","entropy_dt");
#else
  fprintf(file,
          "# %4s %10s %10s %10s %10s %10s %10s %13s\n",
          "ID", "pos_x", "pos_y", "pos_z", "v_x", "v_y", "v_z", "rho");
#endif

  fprintf(file, "# ci --------------------------------------------\n");

  for (int pid = 0; pid < ci->count; pid++) {
#ifdef FULL_COMP
    fprintf(file,
            "%6llu %10f %10f %10f %10f %10f %13e %13e %13e %13e %13e "
            "%13e %13e %13e %13e %13e %13e %13e %13e %13e\n",
            ci->parts[pid].id, ci->parts[pid].x[0], ci->parts[pid].x[1],
            ci->parts[pid].x[2], ci->parts[pid].v[0], ci->parts[pid].v[1],
            ci->parts[pid].v[2], ci->parts[pid].rho,
            ci->parts[pid].density.rho_dh,
            ci->parts[pid].density.wcount, ci->parts[pid].density.wcount_dh,
            ci->parts[pid].density.div_v, ci->parts[pid].density.rot_v[0],
            ci->parts[pid].density.rot_v[1], ci->parts[pid].density.rot_v[2],
            ci->parts[pid].a_hydro[0], ci->parts[pid].a_hydro[1], ci->parts[pid].a_hydro[2],
            ci->parts[pid].force.h_dt, ci->parts[pid].entropy_dt
            );
#else
    fprintf(file,
            "%6llu %10f %10f %10f %10f %10f %13e %13e\n",
            ci->parts[pid].id, ci->parts[pid].x[0], ci->parts[pid].x[1],
            ci->parts[pid].x[2], ci->parts[pid].v[0], ci->parts[pid].v[1],
            ci->parts[pid].v[2], ci->parts[pid].rho
            );
    
#endif
  }

  fprintf(file, "# cj --------------------------------------------\n");

  for (int pjd = 0; pjd < cj->count; pjd++) {
#ifdef FULL_COMP
    fprintf(file,
            "%6llu %10f %10f %10f %10f %10f %13e %13e %13e %13e %13e "
            "%13e %13e %13e %13e %13e %13e %13e %13e %13e\n",
            cj->parts[pjd].id, cj->parts[pjd].x[0], cj->parts[pjd].x[1],
            cj->parts[pjd].x[2], cj->parts[pjd].v[0], cj->parts[pjd].v[1],
            cj->parts[pjd].v[2], cj->parts[pjd].rho,
            cj->parts[pjd].density.rho_dh,
            cj->parts[pjd].density.wcount, cj->parts[pjd].density.wcount_dh,
            cj->parts[pjd].density.div_v, cj->parts[pjd].density.rot_v[0],
            cj->parts[pjd].density.rot_v[1], cj->parts[pjd].density.rot_v[2],
            cj->parts[pjd].a_hydro[0], cj->parts[pjd].a_hydro[1], cj->parts[pjd].a_hydro[2],
            cj->parts[pjd].force.h_dt, cj->parts[pjd].entropy_dt
            );
#else
    fprintf(file,
            "%6llu %10f %10f %10f %10f %10f %13e %13e\n",
            cj->parts[pjd].id, cj->parts[pjd].x[0], cj->parts[pjd].x[1],
            cj->parts[pjd].x[2], cj->parts[pjd].v[0], cj->parts[pjd].v[1],
            cj->parts[pjd].v[2], cj->parts[pjd].rho
            );
#endif
  }
  fclose(file);
}


int main(int argc, char *argv[]) {

  int tmp = 8;
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(ti_current), &tmp, sizeof(int)));
  tmp = 1000;
  cudaErrCheck(
      hipMemcpyToSymbol(HIP_SYMBOL(max_active_bin), &tmp, sizeof(int)));

  float host_kernel_coeffs[(kernel_degree + 1) * (kernel_ivals + 1)];
  for (int a = 0; a < (kernel_degree + 1) * (kernel_ivals + 1); a++) {
    host_kernel_coeffs[a] = kernel_coeffs[a];
  }
  cudaErrCheck(hipMemcpyToSymbol(HIP_SYMBOL(
    cuda_kernel_coeffs), &host_kernel_coeffs,
    sizeof(float) * (kernel_degree + 1) * (kernel_ivals + 1)));

  size_t particles = 0, runs = 0, volume, type = 0;
  double offset[3] = {0, 0, 0}, h = 1.1255, size = 1., rho = 1.;
  double perturbation = 0.;
  struct cell *ci, *cj;

  char c;
  static unsigned long long partId = 0;
  char outputFileNameExtension[200] = "";
  char outputFileName[200] = "";

  //  ticks tic, toc, time;

  /* Initialize CPU frequency, this also starts time. */
  //unsigned long long cpufreq = 0;
  //clocks_set_cpufreq(cpufreq);

  /* Choke on FP-exceptions */
  //feenableexcept(FE_DIVBYZERO | FE_INVALID | FE_OVERFLOW);

  srand(0);

  while ((c = getopt(argc, argv, "h:p:r:t:d:f:")) != -1) {
    switch (c) {
      case 'h':
        sscanf(optarg, "%lf", &h);
        break;
      case 'p':
        sscanf(optarg, "%zu", &particles);
        break;
      case 'r':
        sscanf(optarg, "%zu", &runs);
        break;
      case 't':
        sscanf(optarg, "%zu", &type);
        break;
      case 'd':
        sscanf(optarg, "%lf", &perturbation);
        break;
      case 'f':
        strcpy(outputFileNameExtension, optarg);
        break;
      case '?':
        error("Unknown option.");
        break;
    }
  }

  if (h < 0 || particles == 0 || runs == 0 || type > 2) {
    printf(
        "\nUsage: %s -p PARTICLES_PER_AXIS -r NUMBER_OF_RUNS [OPTIONS...]\n"
        "\nGenerates a cell pair, filled with particles on a Cartesian grid."
        "\nThese are then interacted using runner_dopair1_density."
        "\n\nOptions:"
        "\n-t TYPE=0          - cells share face (0), edge (1) or corner (2)"
        "\n-h DISTANCE=1.1255 - smoothing length"
        "\n-d pert            - perturbation to apply to the particles [0,1["
        "\n-f fileName        - part of the file name used to save the dumps\n",
        argv[0]);
    exit(1);
  }

  volume = particles * particles * particles;
  message("particles: %zu B\npositions: 0 B", 2 * volume * sizeof(struct part));

  ci = make_cell(particles, offset, size, h, rho, &partId, perturbation);
  for (size_t i = 0; i < type + 1; ++i) offset[i] = 1.;
  cj = make_cell(particles, offset, size, h, rho, &partId, perturbation);

  sprintf(outputFileName, "swift_gpu_init_%s.dat", outputFileNameExtension);
  dump_particle_fields(outputFileName, ci, cj);

  allocate_device_parts(2*volume);

  struct cell_cuda *cuda_ci;
  struct cell_cuda *cuda_cj;
  
  //  time = 0;
  for (size_t i = 0; i < runs; ++i) {
    /* Zero the fields */
    zero_particle_fields(ci);
    //zero_particle_fields(cj);

    cuda_ci = copy_from_host(ci, 0);
    //cuda_cj = copy_from_host(cj, ci->count);
    //exit(1);
    //tic = getticks();

    /* Run the test */
    do_test<<<volume/CUDA_THREADS + 1,CUDA_THREADS>>>(cuda_ci, cuda_cj);
    cudaErrCheck( hipPeekAtLastError() );
    cudaErrCheck( hipDeviceSynchronize() );

    //toc = getticks();
    //time += toc - tic;

    copy_to_host(cuda_ci, ci);
    //copy_to_host(cuda_cj, cj);
    /* Dump if necessary */
    if (i % 50 == 0) {
      sprintf(outputFileName, "swift_gpu_%s.dat", outputFileNameExtension);
      dump_particle_fields(outputFileName, ci, cj);
    }
  }

  printf("rho %g\n", ci->parts[0].rho);
  /* Output timing */
//  message("SWIFT calculation took       %lli ticks.", time / runs);

  /* Now perform a brute-force version for accuracy tests */

  /* Zero the fields */
  //zero_particle_fields(ci);
  //zero_particle_fields(cj);

//  tic = getticks();

  /* Run the brute-force test */
  //pairs_all_density(&runner, ci, cj);

//  toc = getticks();

  /* Dump */
  sprintf(outputFileName, "swift_gpu_end_%s.dat", outputFileNameExtension);
  dump_particle_fields(outputFileName, ci, cj);

  /* Output timing */
//  message("Brute force calculation took %lli ticks.", toc - tic);

  /* Clean things to make the sanitizer happy ... */
  clean_up(ci);
  clean_up(cj);

  return 0;
}
